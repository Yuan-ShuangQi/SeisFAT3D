#include "hip/hip_runtime.h"
# include "adjoint_state.cuh"

int Adjoint_State::iDivUp(int a, int b) 
{ 
    return ( (a % b) != 0 ) ? (a / b + 1) : (a / b); 
}

void Adjoint_State::set_parameters()
{
    set_general_parameters();
    
    set_forward_modeling();

    set_main_components();

    nSweeps = 8;
    meshDim = 3;

	totalLevels = (modeling->nxx - 1) + (modeling->nyy - 1) + (modeling->nzz - 1);

    inversion_method = "[1] - Adjoint State first arrival tomography";

    source = new float[modeling->volsize]();
    adjoint = new float[modeling->volsize]();

    hipMalloc((void**)&(d_T), modeling->volsize*sizeof(float));
    hipMalloc((void**)&(d_source), modeling->volsize*sizeof(float));
    hipMalloc((void**)&(d_adjoint), modeling->volsize*sizeof(float));
}

void Adjoint_State::forward_modeling()
{
    initial_setup();

    for (int shot = 0; shot < modeling->total_shots; shot++)
    {
        modeling->shot_id = shot;
    
        modeling->info_message();

        tomography_message();

        modeling->initial_setup();
        modeling->forward_solver();
        modeling->build_outputs();

        extract_calculated_data();
        
        adjoint_state_solver();
    }

    export_gradient();
}

void Adjoint_State::initial_setup()
{
    for (int index = 0; index < modeling->nPoints; index++)
    {    
        int k = (int) (index / (modeling->nx*modeling->nz));        
        int j = (int) (index - k*modeling->nx*modeling->nz) / modeling->nz;    
        int i = (int) (index - j*modeling->nz - k*modeling->nx*modeling->nz);          

        int indB = (i+modeling->nbzu) + (j+modeling->nbxl)*modeling->nzz + (k+modeling->nbyl)*modeling->nxx*modeling->nzz;

        modeling->S[indB] = model[index];

        gradient[index] = 0.0f;
    }

    for (int i = 0; i < n_data; i++) dcal[i] = 0.0f; 
}

void Adjoint_State::adjoint_state_solver()
{
    Tmax = 0.0f;

    cell_volume = modeling->dx * modeling->dy * modeling->dz;

    # pragma omp parallel for
    for (int index = 0; index < modeling->volsize; index++) 
    {
        source[index] = 0.0f;    
        adjoint[index] = 1e6f;

        int k = (int) (index / (modeling->nxx*modeling->nzz));        
        int j = (int) (index - k*modeling->nxx*modeling->nzz) / modeling->nzz;    
        int i = (int) (index - j*modeling->nzz - k*modeling->nxx*modeling->nzz);  

        if ((i == 0) || (i == modeling->nzz-1) || 
            (j == 0) || (j == modeling->nxx-1) || 
            (k == 0) || (k == modeling->nyy-1))  
        {    
            adjoint[index] = 0.0f;        
        }

        if (Tmax < modeling->T[index]) Tmax = modeling->T[index];
    }

    modeling->expand_boundary(modeling->wavefield_output, modeling->T);

    # pragma omp parallel for
    for (int node = 0; node < modeling->total_nodes; node++)
    {
        int current_node = node + modeling->shot_id*modeling->total_nodes;

        int i = (int)(modeling->geometry->nodes.z[node] / modeling->dz) + modeling->nbzu;
        int j = (int)(modeling->geometry->nodes.x[node] / modeling->dx) + modeling->nbxl;
        int k = (int)(modeling->geometry->nodes.y[node] / modeling->dy) + modeling->nbyl;

        int index = i + j*modeling->nzz + k*modeling->nxx*modeling->nzz;

        source[index] += (dobs[current_node] - modeling->T[index]) / cell_volume; 
        source[index + 1] += (dobs[current_node] - modeling->T[index + 1]) / cell_volume; 
        source[index + modeling->nzz] += (dobs[current_node] - modeling->T[index + modeling->nzz]) / cell_volume;         
        source[index + 1 + modeling->nzz] += (dobs[current_node] - modeling->T[index + 1 + modeling->nzz]) / cell_volume; 
        source[index + modeling->nxx*modeling->nzz] += (dobs[current_node] - modeling->T[index + modeling->nxx*modeling->nzz]) / cell_volume; 
        source[index + 1 + modeling->nxx*modeling->nzz] += (dobs[current_node] - modeling->T[index + 1 + modeling->nxx*modeling->nzz]) / cell_volume; 
        source[index + modeling->nzz + modeling->nxx*modeling->nzz] += (dobs[current_node] - modeling->T[index + modeling->nzz + modeling->nxx*modeling->nzz]) / cell_volume; 
        source[index + 1 + modeling->nzz + modeling->nxx*modeling->nzz] += (dobs[current_node] - modeling->T[index + 1 + modeling->nzz + modeling->nxx*modeling->nzz]) / cell_volume;     
    }

	hipMemcpy(d_T, modeling->T, modeling->volsize*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_source, source, modeling->volsize*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_adjoint, adjoint, modeling->volsize*sizeof(float), hipMemcpyHostToDevice);

    for (int sweep = 0; sweep < nSweeps; sweep++)
	{ 
		int start = (sweep == 3 || sweep == 5 || sweep == 6 || sweep == 7) ? totalLevels : meshDim;
		int end = (start == meshDim) ? totalLevels + 1 : meshDim - 1;
		int incr = (start == meshDim) ? true : false;

		int xSweepOff = (sweep == 3 || sweep == 4) ? modeling->nxx + 1 : 0;
		int ySweepOff = (sweep == 2 || sweep == 5) ? modeling->nyy + 1 : 0;
		int zSweepOff = (sweep == 1 || sweep == 6) ? modeling->nzz + 1 : 0;
		
		for (int level = start; level != end; level = (incr) ? level + 1 : level - 1)
		{			
			int xs = max(1, level - (modeling->nyy + modeling->nzz));	
			int ys = max(1, level - (modeling->nxx + modeling->nzz));

			int xe = min(modeling->nxx, level - (meshDim - 1));
			int ye = min(modeling->nyy, level - (meshDim - 1));	
		
			int xr = xe - xs + 1;
			int yr = ye - ys + 1;

			int nThreads = xr * yr;
				
			dim3 bs(16, 16, 1);

			if (nThreads < modeling->threadsPerBlock) { bs.x = xr; bs.y = yr; } 

			dim3 gs(iDivUp(xr, bs.x), iDivUp(yr , bs.y), 1);

            adjoint_state_kernel<<<gs,bs>>>(d_adjoint, d_source, d_T, level, xs, ys, xSweepOff, ySweepOff, zSweepOff, 
                                            modeling->nxx, modeling->nyy, modeling->nzz, modeling->dx, modeling->dy, modeling->dz);

            hipDeviceSynchronize();
		}
	}

    hipMemcpy(adjoint, d_adjoint, modeling->volsize*sizeof(float), hipMemcpyDeviceToHost);

    adjoint_conditioning();

    # pragma omp parallel for
    for (int index = 0; index < modeling->nPoints; index++) 
    {
        int k = (int) (index / (modeling->nx*modeling->nz));        
        int j = (int) (index - k*modeling->nx*modeling->nz) / modeling->nz;    
        int i = (int) (index - j*modeling->nz - k*modeling->nx*modeling->nz);  

        int indp = (i+modeling->nbzu) + (j+modeling->nbxl)*modeling->nzz + (k+modeling->nbyl)*modeling->nxx*modeling->nzz;

        gradient[index] += adjoint[indp]*modeling->S[indp]*modeling->S[indp]*cell_volume;
    }
}

void Adjoint_State::adjoint_conditioning() 
{
    int sidx = (int)(modeling->geometry->shots.x[modeling->shot_id] / modeling->dx) + modeling->nbxl;
    int sidy = (int)(modeling->geometry->shots.y[modeling->shot_id] / modeling->dy) + modeling->nbyl;

    # pragma omp parallel for
    for (int i = 0; i < modeling->nzz; i++)
    {
        for (int j = 0; j < modeling->nxx; j++)
        {
            adjoint[i + j*modeling->nzz + sidy*modeling->nxx*modeling->nzz] = 0.5f*(adjoint[i + j*modeling->nzz + (sidy+1)*modeling->nxx*modeling->nzz] + adjoint[i + j*modeling->nzz + (sidy-1)*modeling->nxx*modeling->nzz]);
        }

        for (int k = 0; k < modeling->nyy; k++)    
        {
            adjoint[i + sidx*modeling->nzz + k*modeling->nxx*modeling->nzz] = 0.5f*(adjoint[i + (sidx+1)*modeling->nzz + k*modeling->nxx*modeling->nzz] + adjoint[i + (sidx-1)*modeling->nzz + k*modeling->nxx*modeling->nzz]);
        }
    }
    
    # pragma omp parallel for
    for (int index = 0; index < modeling->volsize; index++)
    {
        adjoint[index] *= modeling->T[index] / Tmax;
    }
}

void Adjoint_State::optimization() 
{ 



}

__global__ void adjoint_state_kernel(float * adjoint, float * source, float * T, int level, int xOffset, int yOffset, 
                                     int xSweepOffset, int ySweepOffset, int zSweepOffset, int nxx, int nyy, int nzz, 
                                     float dx, float dy, float dz)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x) + xOffset;
	int y = (blockIdx.y * blockDim.y + threadIdx.y) + yOffset;

	if ((x <= nxx) && (y <= nyy)) 
	{
		int z = level - (x + y);
		
		if ((z >= 0) && (z <= nzz))	
		{
			int i = (int)abs(z - zSweepOffset);
			int j = (int)abs(x - xSweepOffset);
			int k = (int)abs(y - ySweepOffset);

			if ((i > 0) && (i < nzz-1) && (j > 0) && (j < nxx-1) && (k > 0) && (k < nyy-1))
			{		
                float a1 = -1.0f*(T[i + j*nzz + k*nxx*nzz] - T[i + (j-1)*nzz + k*nxx*nzz]) / dx;
                float ap1 = (a1 + abs(a1)) / 2.0f;
                float am1 = (a1 - abs(a1)) / 2.0f;

                float a2 = -1.0f*(T[i + (j+1)*nzz + k*nxx*nzz] - T[i + j*nzz + k*nxx*nzz]) / dx;
                float ap2 = (a2 + abs(a2)) / 2.0f;
                float am2 = (a2 - abs(a2)) / 2.0f;

                float b1 = -1.0f*(T[i + j*nzz + k*nxx*nzz] - T[i + j*nzz + (k-1)*nxx*nzz]) / dy;
                float bp1 = (b1 + abs(b1)) / 2.0f;
                float bm1 = (b1 - abs(b1)) / 2.0f;

                float b2 = -1.0f*(T[i + j*nzz + (k+1)*nxx*nzz] - T[i + j*nzz + k*nxx*nzz]) / dy;
                float bp2 = (b2 + abs(b2)) / 2.0f;
                float bm2 = (b2 - abs(b2)) / 2.0f;

                float c1 = -1.0f*(T[i + j*nzz + k*nxx*nzz] - T[(i-1) + j*nzz + k*nxx*nzz]) / dz;
                float cp1 = (c1 + abs(c1)) / 2.0f;
                float cm1 = (c1 - abs(c1)) / 2.0f;

                float c2 = -1.0f*(T[(i+1) + j*nzz + k*nxx*nzz] - T[i + j*nzz + k*nxx*nzz]) / dz;
                float cp2 = (c2 + abs(c2)) / 2.0f;
                float cm2 = (c2 - abs(c2)) / 2.0f;

                float d = (ap2 - am1)/dx + (bp2 - bm1)/dy + (cp2 - cm1)/dz;

                if (abs(d) < 1e-6f)
                {
                    adjoint[i + j*nzz + k*nxx*nzz] = 0.0f;    
                }
                else
                {
                    float e = (ap1*adjoint[i + (j-1)*nzz + k*nxx*nzz] - am2*adjoint[i + (j+1)*nzz + k*nxx*nzz]) / dx +
                              (bp1*adjoint[i + j*nzz + (k-1)*nxx*nzz] - bm2*adjoint[i + j*nzz + (k+1)*nxx*nzz]) / dy +
                              (cp1*adjoint[(i-1) + j*nzz + k*nxx*nzz] - cm2*adjoint[(i+1) + j*nzz + k*nxx*nzz]) / dz;

                    float f = (e + source[i + j*nzz + k*nxx*nzz]) / d;
                    float g = adjoint[i + j*nzz + k*nxx*nzz];

                    if (g > f) adjoint[i + j*nzz + k*nxx*nzz] = f;
                }
            }
        }
    }
}