#include "hip/hip_runtime.h"
# include "least_squares.cuh"

void Least_Squares::set_parameters()
{
    set_general_parameters();
    
    set_forward_modeling();

    set_main_components();

    illumination_folder = catch_parameter("illumination_folder", file);
    write_illumination_per_iteration = str2bool(catch_parameter("export_illumination", file));

    dx_tomo = std::stof(catch_parameter("dx_tomo", file));
    dy_tomo = std::stof(catch_parameter("dy_tomo", file));
    dz_tomo = std::stof(catch_parameter("dz_tomo", file));

    nz_tomo = (int)((modeling->nz-1) * modeling->dz / dz_tomo) + 1;    
    nx_tomo = (int)((modeling->nx-1) * modeling->dx / dx_tomo) + 1;    
    ny_tomo = (int)((modeling->ny-1) * modeling->dy / dy_tomo) + 1;  

    lambda = std::stof(catch_parameter("tk_param", file));
    tk_order = std::stoi(catch_parameter("tk_order", file));

    n_model = nx_tomo * ny_tomo * nz_tomo;

    inversion_method = "[0] - Classical least squares first arrival tomography";

    ray_path_estimated_samples = 0;

    for (int shot = 0; shot < modeling->total_shots; shot++)
    {
        for (int node = 0; node < modeling->total_nodes; node++)
        {
            float dx = (modeling->geometry->shots.x[shot] - modeling->geometry->nodes.x[node]) / modeling->dx;
            float dy = (modeling->geometry->shots.y[shot] - modeling->geometry->nodes.y[node]) / modeling->dy;
            float dz = (modeling->geometry->shots.z[shot] - modeling->geometry->nodes.z[node]) / modeling->dz;
            
            ray_path_estimated_samples += (size_t)(3.0f*sqrtf(dx*dx + dy*dy + dz*dz));
        }
    }

    illumination = new float[modeling->nPoints]();

    iG.reserve(ray_path_estimated_samples);
    jG.reserve(ray_path_estimated_samples);
    vG.reserve(ray_path_estimated_samples);
}

void Least_Squares::forward_modeling()
{
    initial_setup();

    for (int shot = 0; shot < modeling->total_shots; shot++)
    {
        modeling->shot_id = shot;
    
        modeling->info_message();

        tomography_message();

        modeling->initial_setup();
        modeling->forward_solver();
        modeling->build_outputs();

        extract_calculated_data();
        gradient_ray_tracing();
    }

    compute_gradient();
    
    export_gradient();
    export_illumination();
}

void Least_Squares::initial_setup()
{
    for (int index = 0; index < modeling->nPoints; index++)
    {    
        int k = (int) (index / (modeling->nx*modeling->nz));        
        int j = (int) (index - k*modeling->nx*modeling->nz) / modeling->nz;    
        int i = (int) (index - j*modeling->nz - k*modeling->nx*modeling->nz);          

        int indB = (i+modeling->nbzu) + (j+modeling->nbxl)*modeling->nzz + (k+modeling->nbyl)*modeling->nxx*modeling->nzz;

        modeling->S[indB] = model[index];

        gradient[index] = 0.0f;
        illumination[index] = 0.0f;
    }

    for (int i = 0; i < n_data; i++) dcal[i] = 0.0f;    
}

void Least_Squares::gradient_ray_tracing()
{
    int nxx = modeling->nxx;
    int nzz = modeling->nzz;

    int sIdz = (int)(modeling->geometry->shots.z[modeling->shot_id] / dz_tomo);
    int sIdx = (int)(modeling->geometry->shots.x[modeling->shot_id] / dx_tomo);
    int sIdy = (int)(modeling->geometry->shots.y[modeling->shot_id] / dy_tomo);

    int sId = sIdz + sIdx*nz_tomo + sIdy*nx_tomo*nz_tomo;   

    float rayStep = 0.2f * modeling->dz;

    std::vector < int > ray_index;

    for (int ray_id = 0; ray_id < modeling->total_nodes; ray_id++)
    {
        float zi = modeling->geometry->nodes.z[ray_id];
        float xi = modeling->geometry->nodes.x[ray_id];
        float yi = modeling->geometry->nodes.y[ray_id];

        if ((modeling->geometry->shots.z[modeling->shot_id] == zi) && 
            (modeling->geometry->shots.x[modeling->shot_id] == xi) && 
            (modeling->geometry->shots.y[modeling->shot_id] == yi))
            continue;

        while (true)
        {
            int i = (int)(zi / modeling->dz) + modeling->nbzu;
            int j = (int)(xi / modeling->dx) + modeling->nbxl;
            int k = (int)(yi / modeling->dy) + modeling->nbyl;

            float dTz = (modeling->T[(i+1) + j*nzz + k*nxx*nzz] - modeling->T[(i-1) + j*nzz + k*nxx*nzz]) / (2.0f*modeling->dz);    
            float dTx = (modeling->T[i + (j+1)*nzz + k*nxx*nzz] - modeling->T[i + (j-1)*nzz + k*nxx*nzz]) / (2.0f*modeling->dx);    
            float dTy = (modeling->T[i + j*nzz + (k+1)*nxx*nzz] - modeling->T[i + j*nzz + (k-1)*nxx*nzz]) / (2.0f*modeling->dy);

            float norm = sqrtf(dTx*dTx + dTy*dTy + dTz*dTz);

            zi -= rayStep*dTz / norm;    
            xi -= rayStep*dTx / norm;   
            yi -= rayStep*dTy / norm;   

            int im = (int)(zi / dz_tomo); 
            int jm = (int)(xi / dx_tomo); 
            int km = (int)(yi / dy_tomo); 

            ray_index.push_back(im + jm*nz_tomo + km*nx_tomo*nz_tomo);

            int index = (i - modeling->nbzu) + (j - modeling->nbxl)*modeling->nz + (k - modeling->nbyl)*modeling->nx*modeling->nz;

            illumination[index] += rayStep;

            if (ray_index.back() == sId) break;
        }
   
        float final_distance = sqrtf(powf(zi - modeling->geometry->shots.z[modeling->shot_id],2.0f) + 
                                    powf(xi - modeling->geometry->shots.x[modeling->shot_id],2.0f) + 
                                    powf(yi - modeling->geometry->shots.y[modeling->shot_id],2.0f));

        std::sort(ray_index.begin(), ray_index.end());

        int current_voxel_index = ray_index[0];
        float distance_per_voxel = rayStep;

        for (int index = 0; index < ray_index.size(); index++)
        {
            if (ray_index[index] == current_voxel_index)
            {
                distance_per_voxel += rayStep;
            }
            else
            {
                vG.emplace_back(distance_per_voxel);
                jG.emplace_back(current_voxel_index);
                iG.emplace_back(ray_id + modeling->shot_id * modeling->total_nodes);

                if (current_voxel_index == sId) vG.back() = final_distance;

                distance_per_voxel = rayStep;
                current_voxel_index = ray_index[index];    
            }
        }

        if (current_voxel_index == sId)
        {
            vG.emplace_back(final_distance);
            jG.emplace_back(current_voxel_index);
            iG.emplace_back(ray_id + modeling->shot_id * modeling->total_nodes);
        }
        else 
        {
            vG.emplace_back(distance_per_voxel);
            jG.emplace_back(current_voxel_index);
            iG.emplace_back(ray_id + modeling->shot_id * modeling->total_nodes);
        }

        std::vector < int >().swap(ray_index);
    }
}

void Least_Squares::compute_gradient()
{
    float * grad = new float[n_model]();

    for (int index = 0; index < vG.size(); index++)
    {
        grad[jG[index]] += vG[index] * (dobs[iG[index]] - dcal[iG[index]]);
    }

    for (int index = 0; index < modeling->nPoints; index++)
    {
        int k = (int) (index / (modeling->nx*modeling->nz));        
        int j = (int) (index - k*modeling->nx*modeling->nz) / modeling->nz;    
        int i = (int) (index - j*modeling->nz - k*modeling->nx*modeling->nz);  

        float xp = j*modeling->dx; 
        float yp = k*modeling->dy; 
        float zp = i*modeling->dz; 

        float x0 = floorf(xp/dx_tomo)*dx_tomo;
        float y0 = floorf(yp/dy_tomo)*dy_tomo;
        float z0 = floorf(zp/dz_tomo)*dz_tomo;

        float x1 = floorf(xp/dx_tomo)*dx_tomo + dx_tomo;
        float y1 = floorf(yp/dy_tomo)*dy_tomo + dy_tomo;
        float z1 = floorf(zp/dz_tomo)*dz_tomo + dz_tomo;

        gradient[index] = 0.0f;

        int idz = (int)(zp/dz_tomo);
        int idx = (int)(xp/dx_tomo);
        int idy = (int)(yp/dy_tomo);

        int ind_m = (int)(idz + idx*nz_tomo + idy*nx_tomo*nz_tomo);

        float c000 = grad[ind_m];                  
        float c001 = grad[ind_m + 1];
        float c100 = grad[ind_m + nz_tomo];
        float c101 = grad[ind_m + 1 + nz_tomo];
        float c010 = grad[ind_m + nx_tomo*nz_tomo];
        float c011 = grad[ind_m + 1 + nx_tomo*nz_tomo];
        float c110 = grad[ind_m + nz_tomo + nx_tomo*nz_tomo];
        float c111 = grad[ind_m + 1 + nz_tomo + nx_tomo*nz_tomo];  

        float xd = (xp - x0) / (x1 - x0);
        float yd = (yp - y0) / (y1 - y0);
        float zd = (zp - z0) / (z1 - z0);

        float c00 = c000*(1 - xd) + c100*xd;    
        float c01 = c001*(1 - xd) + c101*xd;    
        float c10 = c010*(1 - xd) + c110*xd;    
        float c11 = c011*(1 - xd) + c111*xd;    

        float c0 = c00*(1 - yd) + c10*yd;
        float c1 = c01*(1 - yd) + c11*yd;

        float g_ijk = (c0*(1 - zd) + c1*zd);

        gradient[i + j*modeling->nz + k*modeling->nx*modeling->nz] = g_ijk;            
    }    

    delete[] grad;
}

void Least_Squares::export_illumination()
{
    if (write_illumination_per_iteration)
    {
        std::string illumination_path = illumination_folder + "illumination_iteration_" + std::to_string(iteration) + "_" + std::to_string(modeling->nz) + "x" + std::to_string(modeling->nx) + "x" + std::to_string(modeling->ny) + ".bin";

        export_binary_float(illumination_path, illumination, modeling->nPoints);
    }
}

void Least_Squares::optimization()
{
    std::cout<<"\nSolving linear system using Tikhonov regularization with order " + std::to_string(tk_order) + "\n\n";

    M = n_model;                                  
    N = n_data + n_model - tk_order;                    
    NNZ = vG.size() + (tk_order + 1) * (n_model - tk_order);

    iA = new int[NNZ]();
    jA = new int[NNZ]();
    vA = new float[NNZ]();

    B = new float[N]();
    x = new float[M]();

    for (int index = 0; index < n_data; index++) 
        B[index] = dobs[index] - dcal[index];

    for (int index = 0; index < vG.size(); index++)
    {
        iA[index] = iG[index];
        jA[index] = jG[index];
        vA[index] = vG[index];
    }

    std::vector< int >().swap(iG);
    std::vector< int >().swap(jG);
    std::vector<float>().swap(vG);

    apply_regularization();
    solve_linear_system_lscg();
    slowness_variation_rescaling();

    delete[] B;
    delete[] iA;
    delete[] jA;
    delete[] vA;
}

void Least_Squares::apply_regularization()
{
    int elements = tk_order + 1;
		
    int n = n_model - tk_order;
    int nnz = elements * n;	
    
    int * iL = new int[nnz]();
    int * jL = new int[nnz]();
    float * vL = new float[nnz]();

    if (tk_order <= 0)
	{
		for (int index = 0; index < nnz; index++)
		{
			iL[index] = index;
			jL[index] = index;
			vL[index] = 1.0f;
		}
	} 
    else
    {
        int * df = new int[elements]();	
        int * df1 = new int[elements + 1]();
        int * df2 = new int[elements + 1]();
        
        df[0] = -1; df[1] = 1;
        
        for (int index = 1; index < tk_order; index++)
        {
            for (int k = 0; k < elements; k++)
            {
                df2[k] = df[k];
                df1[k + 1] = df[k];

                df[k] = df1[k] - df2[k]; 
            }		 
        }
        
        for (int index = 0; index < n; index++)
        {
            for (int k = 0; k < elements; k++)
            {
                iL[elements*index + k] = index;	
                jL[elements*index + k] = index + k;
                vL[elements*index + k] = df[k];
            }	
        }

        delete[] df;
        delete[] df1;
        delete[] df2;
    }

    for (int index = NNZ - nnz; index < NNZ; index++) 
    {
        iA[index] = n_data + iL[index - (NNZ - nnz)];
        jA[index] = jL[index - (NNZ - nnz)];
        vA[index] = lambda * vL[index - (NNZ - nnz)];        
    }

    delete[] iL;
    delete[] jL;
    delete[] vL;
}

void Least_Squares::solve_linear_system_lscg()
{
    float a, b, qTq, rTr, rd;
    int cg_max_iteration = 10;

    float * s = new float[N]();
    float * q = new float[N]();
    float * r = new float[M]();
    float * p = new float[M]();

    // s = d - G * x, where d = dobs - dcal and x = slowness variation
    for (int i = 0; i < N; i++) 
        s[i] = B[i]; 

    // r = G' * s    
    for (int i = 0; i < NNZ; i++) 
        r[jA[i]] += vA[i] * s[iA[i]];        

    // p = r and x = 0;
    for (int i = 0; i < M; i++) 
    {
        p[i] = r[i]; 
        x[i] = 0.0f;
    }

    // q = G * p
    for (int i = 0; i < NNZ; i++) 
        q[iA[i]] += vA[i] * p[jA[i]];        

    for (int i = 0; i < cg_max_iteration; i++)
    {
        qTq = 0.0f;
        for (int k = 0; k < N; k++)           // q inner product
            qTq += q[k] * q[k];               // qTq = q' * q

        rTr = 0.0f;
        for (int k = 0; k < M; k++)           // r inner product
            rTr += r[k] * r[k];               // rTr = r' * r 

        a = rTr / qTq;                        // a = (r' * r) / (q' * q)                    

        for (int k = 0; k < M; k++)           // model atualization
            x[k] += a * p[k];                 // x = x + a * p

        for (int k = 0; k < N; k++)           // s atualization  
            s[k] -= a * q[k];                 // s = s - a * q 

        rd = 0.0f;
        for (int k = 0; k < M; k++)           // r inner product for division 
            rd += r[k] * r[k];                // rd = r' * r

        for (int k = 0; k < M; k++)           // Zeroing r 
            r[k] = 0.0f;                      // r = 0, for multiplication
        
        for (int k = 0; k < NNZ; k++)         // r atualization 
            r[jA[k]] += vA[k] * s[iA[k]];     // r = G' * s    

        rTr = 0.0f;                
        for (int k = 0; k < M; k++)           // r inner product
            rTr += r[k] * r[k];               // rTr = r' * r

        b = rTr / rd;                         // b = (r' * r) / rd

        for (int k = 0; k < M; k++)          
            p[k] = r[k] + b * p[k];           // p = r + b * p 

        for (int k = 0; k < N; k++) 
            q[k] = 0.0f;                      // q = 0, for multiplication

        for (int k = 0; k < NNZ; k++) 
            q[iA[k]] += vA[k] * p[jA[k]];     // q = G * p   
    }
}

void Least_Squares::slowness_variation_rescaling()
{
    for (int index = 0; index < modeling->nPoints; index++)
    {
        int k = (int) (index / (modeling->nx*modeling->nz));        
        int j = (int) (index - k*modeling->nx*modeling->nz) / modeling->nz;    
        int i = (int) (index - j*modeling->nz - k*modeling->nx*modeling->nz);  

        float xp = j*modeling->dx; 
        float yp = k*modeling->dy; 
        float zp = i*modeling->dz; 

        float x0 = floorf(xp/dx_tomo)*dx_tomo;
        float y0 = floorf(yp/dy_tomo)*dy_tomo;
        float z0 = floorf(zp/dz_tomo)*dz_tomo;

        float x1 = floorf(xp/dx_tomo)*dx_tomo + dx_tomo;
        float y1 = floorf(yp/dy_tomo)*dy_tomo + dy_tomo;
        float z1 = floorf(zp/dz_tomo)*dz_tomo + dz_tomo;

        dm[index] = 0.0f;

        int idz = (int)(zp/dz_tomo);
        int idx = (int)(xp/dx_tomo);
        int idy = (int)(yp/dy_tomo);

        int ind_m = (int)(idz + idx*nz_tomo + idy*nx_tomo*nz_tomo);

        float c000 = x[ind_m];                  
        float c001 = x[ind_m + 1];
        float c100 = x[ind_m + nz_tomo];
        float c101 = x[ind_m + 1 + nz_tomo];
        float c010 = x[ind_m + nx_tomo*nz_tomo];
        float c011 = x[ind_m + 1 + nx_tomo*nz_tomo];
        float c110 = x[ind_m + nz_tomo + nx_tomo*nz_tomo];
        float c111 = x[ind_m + 1 + nz_tomo + nx_tomo*nz_tomo];  

        float xd = (xp - x0) / (x1 - x0);
        float yd = (yp - y0) / (y1 - y0);
        float zd = (zp - z0) / (z1 - z0);

        float c00 = c000*(1 - xd) + c100*xd;    
        float c01 = c001*(1 - xd) + c101*xd;    
        float c10 = c010*(1 - xd) + c110*xd;    
        float c11 = c011*(1 - xd) + c111*xd;    

        float c0 = c00*(1 - yd) + c10*yd;
        float c1 = c01*(1 - yd) + c11*yd;

        float dm_ijk = (c0*(1 - zd) + c1*zd);

        dm[i + j*modeling->nz + k*modeling->nx*modeling->nz] = dm_ijk;            
    }
}