#include "hip/hip_runtime.h"
# include "elastic_iso.cuh"

void Elastic_ISO::set_conditions()
{
    modeling_type = "elastic_iso";
    modeling_name = "Modeling type: Elastic isotropic solver";

    eikonal = new Eikonal_ISO();
    eikonal->parameters = parameters;
    eikonal->set_parameters();

    M = new float[volsize]();
    L = new float[volsize]();
    B = new float[volsize]();

    for (int index = 0; index < volsize; index++)
    {
        M[index] = Ro[index]*Vs[index]*Vs[index];
        L[index] = Ro[index]*Vp[index]*Vp[index] - 2.0f*M[index];
        B[index] = 1.0f / Ro[index];
    }
    
    hipMalloc((void**)&(d_M), volsize*sizeof(float));
    hipMalloc((void**)&(d_L), volsize*sizeof(float));
    hipMalloc((void**)&(d_B), volsize*sizeof(float));

    hipMemcpy(d_M, M, volsize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_L, L, volsize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, volsize*sizeof(float), hipMemcpyHostToDevice);
}

void Elastic_ISO::propagation()
{
    for (int tId = 0; tId < nt + tlag; tId++)
    {
        compute_pressure<<<nBlocks, nThreads>>>(d_Vx, d_Vy, d_Vz, d_Txx, d_Tyy, d_Tzz, d_Txz, d_Tyz, d_Txy, d_P, d_M, d_L, d_T, wavelet, sIdx, sIdy, sIdz, tId, tlag, nt, dx, dy, dz, dt, nxx, nyy, nzz);
        hipDeviceSynchronize();

        compute_velocity<<<nBlocks, nThreads>>>(d_Vx, d_Vy, d_Vz, d_Txx, d_Tyy, d_Tzz, d_Txz, d_Tyz, d_Txy, d_B, d_T, d1D, d2D, d3D, dx, dy, dz, dt, tId, tlag, nxx, nyy, nzz, nb);
        hipDeviceSynchronize();

        compute_seismogram<<<sBlocks, nThreads>>>(d_P, rIdx, rIdy, rIdz, seismogram, geometry->spread[srcId], tId, tlag, nt, nxx, nzz);     
        hipDeviceSynchronize();
    }
}

__global__ void compute_pressure(float * Vx, float * Vy, float * Vz, float * Txx, float * Tyy, float * Tzz, float * Txz, float * Tyz, float * Txy, float * P, float * M, float * L, float * T, float * wavelet, int sIdx, int sIdy, int sIdz, int tId, int tlag, int nt, float dx, float dy, float dz, float dt, int nxx, int nyy, int nzz)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int k = (int) (index / (nxx*nzz));         
    int j = (int) (index - k*nxx*nzz) / nzz;   
    int i = (int) (index - j*nzz - k*nxx*nzz); 

    if ((index == 0) && (tId < nt))
    {
        Txx[sIdz + sIdx*nzz + sIdy*nxx*nzz] += wavelet[tId] / (dx*dy*dz);
        Tyy[sIdz + sIdx*nzz + sIdy*nxx*nzz] += wavelet[tId] / (dx*dy*dz);
        Tzz[sIdz + sIdx*nzz + sIdy*nxx*nzz] += wavelet[tId] / (dx*dy*dz);
    }

    if ((index < nxx*nyy*nzz) && (T[index] < (float)(tId + tlag)*dt))
    {
        if((i >= 3) && (i < nzz-4) && (j >= 3) && (j < nxx-4) && (k >= 3) && (k < nyy-4)) 
        {    
            float dVx_dx = (FDM1*(Vx[i + (j-3)*nzz + k*nxx*nzz] - Vx[i + (j+4)*nzz + k*nxx*nzz]) +
                            FDM2*(Vx[i + (j+3)*nzz + k*nxx*nzz] - Vx[i + (j-2)*nzz + k*nxx*nzz]) +
                            FDM3*(Vx[i + (j-1)*nzz + k*nxx*nzz] - Vx[i + (j+2)*nzz + k*nxx*nzz]) +
                            FDM4*(Vx[i + (j+1)*nzz + k*nxx*nzz] - Vx[i + j*nzz + k*nxx*nzz])) / dx;

            float dVy_dy = (FDM1*(Vy[i + j*nzz + (k-3)*nxx*nzz] - Vy[i + j*nzz + (k+4)*nxx*nzz]) +
                            FDM2*(Vy[i + j*nzz + (k+3)*nxx*nzz] - Vy[i + j*nzz + (k-2)*nxx*nzz]) +
                            FDM3*(Vy[i + j*nzz + (k-1)*nxx*nzz] - Vy[i + j*nzz + (k+2)*nxx*nzz]) +
                            FDM4*(Vy[i + j*nzz + (k+1)*nxx*nzz] - Vy[i + j*nzz + k*nxx*nzz])) / dy;

            float dVz_dz = (FDM1*(Vz[(i-3) + j*nzz + k*nxx*nzz] - Vz[(i+4) + j*nzz + k*nxx*nzz]) +
                            FDM2*(Vz[(i+3) + j*nzz + k*nxx*nzz] - Vz[(i-2) + j*nzz + k*nxx*nzz]) +
                            FDM3*(Vz[(i-1) + j*nzz + k*nxx*nzz] - Vz[(i+2) + j*nzz + k*nxx*nzz]) +
                            FDM4*(Vz[(i+1) + j*nzz + k*nxx*nzz] - Vz[i + j*nzz + k*nxx*nzz])) / dz;

            Txx[index] += dt*((L[index] + 2*M[index])*dVx_dx + L[index]*(dVy_dy + dVz_dz));
            Tyy[index] += dt*((L[index] + 2*M[index])*dVy_dy + L[index]*(dVx_dx + dVz_dz));
            Tzz[index] += dt*((L[index] + 2*M[index])*dVz_dz + L[index]*(dVx_dx + dVy_dy));                    
        }

        if((i >= 3) && (i < nzz-4) && (j > 3) && (j < nxx-3) && (k > 3) && (k < nyy-3)) 
        {
            float dVx_dy = (FDM1*(Vx[i + j*nzz + (k-4)*nxx*nzz] - Vx[i + j*nzz + (k+3)*nxx*nzz]) +
                            FDM2*(Vx[i + j*nzz + (k+2)*nxx*nzz] - Vx[i + j*nzz + (k-3)*nxx*nzz]) +
                            FDM3*(Vx[i + j*nzz + (k-2)*nxx*nzz] - Vx[i + j*nzz + (k+1)*nxx*nzz]) +
                            FDM4*(Vx[i + j*nzz + k*nxx*nzz]     - Vx[i + j*nzz + (k-1)*nxx*nzz])) / dy;

            float dVy_dx = (FDM1*(Vy[i + (j-4)*nzz + k*nxx*nzz] - Vy[i + (j+3)*nzz + k*nxx*nzz]) +
                            FDM2*(Vy[i + (j+2)*nzz + k*nxx*nzz] - Vy[i + (j-3)*nzz + k*nxx*nzz]) +
                            FDM3*(Vy[i + (j-2)*nzz + k*nxx*nzz] - Vy[i + (j+1)*nzz + k*nxx*nzz]) +
                            FDM4*(Vy[i + j*nzz + k*nxx*nzz]     - Vy[i + (j-1)*nzz + k*nxx*nzz])) / dx;

            float Mxy = powf(0.25f*(1.0f/M[i + (j+1)*nzz + (k+1)*nxx*nzz] + 1.0f/M[i + (j+1)*nzz + k*nxx*nzz] + 
                                    1.0f/M[i + j*nzz + (k+1)*nxx*nzz]     + 1.0f/M[i + j*nzz + k*nxx*nzz]), -1.0f);

            Txy[index] += dt*Mxy*(dVx_dy + dVy_dx);
        }

        if((i > 3) && (i < nzz-3) && (j > 3) && (j < nxx-3) && (k >= 3) && (k < nyy-4)) 
        {
            float dVx_dz = (FDM1*(Vx[(i-4) + j*nzz + k*nxx*nzz] - Vx[(i+3) + j*nzz + k*nxx*nzz]) +
                            FDM2*(Vx[(i+2) + j*nzz + k*nxx*nzz] - Vx[(i-3) + j*nzz + k*nxx*nzz]) +
                            FDM3*(Vx[(i-2) + j*nzz + k*nxx*nzz] - Vx[(i+1) + j*nzz + k*nxx*nzz]) +
                            FDM4*(Vx[i + j*nzz + k*nxx*nzz]     - Vx[(i-1) + j*nzz + k*nxx*nzz])) / dz;

            float dVz_dx = (FDM1*(Vz[i + (j-4)*nzz + k*nxx*nzz] - Vz[i + (j+3)*nzz + k*nxx*nzz]) +
                            FDM2*(Vz[i + (j+2)*nzz + k*nxx*nzz] - Vz[i + (j-3)*nzz + k*nxx*nzz]) +
                            FDM3*(Vz[i + (j-2)*nzz + k*nxx*nzz] - Vz[i + (j+1)*nzz + k*nxx*nzz]) +
                            FDM4*(Vz[i + j*nzz + k*nxx*nzz]     - Vz[i + (j-1)*nzz + k*nxx*nzz])) / dx;

            float Mxz = powf(0.25f*(1.0f/M[(i+1) + (j+1)*nzz + k*nxx*nzz] + 1.0f/M[i + (j+1)*nzz + k*nxx*nzz] + 
                                    1.0f/M[(i+1) + j*nzz + k*nxx*nzz]     + 1.0f/M[i + j*nzz + k*nxx*nzz]), -1.0f);

            Txz[index] += dt*Mxz*(dVx_dz + dVz_dx);
        }

        if((i > 3) && (i < nzz-3) && (j >= 3) && (j < nxx-4) && (k > 3) && (k < nyy-3)) 
        {
            float dVy_dz = (FDM1*(Vy[(i-4) + j*nzz + k*nxx*nzz] - Vy[(i+3) + j*nzz + k*nxx*nzz]) +
                            FDM2*(Vy[(i+2) + j*nzz + k*nxx*nzz] - Vy[(i-3) + j*nzz + k*nxx*nzz]) +
                            FDM3*(Vy[(i-2) + j*nzz + k*nxx*nzz] - Vy[(i+1) + j*nzz + k*nxx*nzz]) +
                            FDM4*(Vy[i + j*nzz + k*nxx*nzz]     - Vy[(i-1) + j*nzz + k*nxx*nzz])) / dz;

            float dVz_dy = (FDM1*(Vz[i + j*nzz + (k-4)*nxx*nzz] - Vz[i + j*nzz + (k+3)*nxx*nzz]) +
                            FDM2*(Vz[i + j*nzz + (k+2)*nxx*nzz] - Vz[i + j*nzz + (k-3)*nxx*nzz]) +
                            FDM3*(Vz[i + j*nzz + (k-2)*nxx*nzz] - Vz[i + j*nzz + (k+1)*nxx*nzz]) +
                            FDM4*(Vz[i + j*nzz + k*nxx*nzz]     - Vz[i + j*nzz + (k-1)*nxx*nzz])) / dy;

            float Myz = powf(0.25f*(1.0f/M[(i+1) + j*nzz + (k+1)*nxx*nzz] + 1.0f/M[i + j*nzz + (k+1)*nxx*nzz] + 
                                    1.0f/M[(i+1) + j*nzz + k*nxx*nzz] +     1.0f/M[i + j*nzz + k*nxx*nzz]), -1.0f);

            Tyz[index] += dt*Myz*(dVy_dz + dVz_dy);
        }

        if ((i > 3) && (i < nzz-4) && (j > 3) && (j < nxx-4) && (k > 3) && (k < nyy-4))
        {
            P[index] = (Txx[index] + Tyy[index] + Tzz[index]) / 3.0f;
        }
    }
}

