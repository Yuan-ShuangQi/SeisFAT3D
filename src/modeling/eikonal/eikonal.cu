#include "hip/hip_runtime.h"
# include "eikonal.cuh"

void Eikonal::set_parameters(std::string file)
{
    Modeling::set_parameters(file);

    eikonal_type = std::stoi(catch_parameter("eikonal_type", file));

    switch (eikonal_type)
    {
    case 0:
        PAL_parameters();    
        break;    
    case 1:
        FIM_parameters();
        break;
    case 2:
        FSM_parameters();
        break;
    default:
        eikonal_type = 1;
        FIM_parameters();
        break;
    }

    volsize = nxx * nyy * nzz;

    S = new float[volsize]();
    vp = new float[nPoints]();

    std::string vp_model_file = catch_parameter("vp_model_file", file);

    import_binary_float(vp_model_file, vp, nPoints);

    eikonal_type == true ? pad_expansion() : fdm_expansion();

    delete[] vp;
}

void Eikonal::set_components()
{
    switch (eikonal_type)
    {
    case 0:
        PAL_components();    
        break;    
    case 1:
        FIM_components();
        break;
    case 2:
        FSM_components();
        break;
    default:
        eikonal_type = 1;
        FIM_components();
        break;
    }
	
    T = new float[volsize]();

    wavefield_output_samples = nPoints;
    receiver_output_samples = geometry->nodes.total;

    receiver_output = new float[receiver_output_samples]();
    wavefield_output = new float[wavefield_output_samples]();
    
    get_RAM_usage();
    get_GPU_usage();
}

void Eikonal::initial_setup()
{
    switch (eikonal_type)
    {
    case 0:
        PAL_init();    
        break;    
    case 1:
        FIM_init();
        break;
    case 2:
        FSM_init();
        break;
    default:
        eikonal_type = 1;
        FIM_init();
        break;
    }
}

void Eikonal::forward_solver()
{
    switch (eikonal_type)
    {
    case 0:
        PAL_solver();    
        break;    
    case 1:
        FIM_solver();
        break;
    case 2:
        FSM_solver();
        break;
    default:
        eikonal_type = 1;
        FIM_solver();
        break;
    }
}

void Eikonal::build_outputs()
{
    get_travelTimes();
    get_firstArrivals();
}

void Eikonal::get_travelTimes()
{
    eikonal_type == true ? pad_reduction() : fdm_reduction();    

    wavefield_output_file = wavefield_output_folder + "travel_times_" + std::to_string(nz) + "x" + std::to_string(nx) + "x" + std::to_string(ny) + "_shot_" + std::to_string(shot_id+1) + ".bin";
}

void Eikonal::get_firstArrivals()
{
    for (int r = 0; r < total_nodes; r++)
    {
        float x = geometry->nodes.x[r];
        float y = geometry->nodes.y[r];
        float z = geometry->nodes.z[r];

        float x0 = floorf(x / dh) * dh;
        float y0 = floorf(y / dh) * dh;
        float z0 = floorf(z / dh) * dh;

        float x1 = floorf(x / dh) * dh + dh;
        float y1 = floorf(y / dh) * dh + dh;
        float z1 = floorf(z / dh) * dh + dh;

        int id = ((int)(z / dh)) + ((int)(x / dh))*nz + ((int)(y / dh))*nx*nz;

        float c000 = wavefield_output[id];
        float c001 = wavefield_output[id + 1];
        float c100 = wavefield_output[id + nz]; 
        float c101 = wavefield_output[id + 1 + nz]; 
        float c010 = wavefield_output[id + nx*nz]; 
        float c011 = wavefield_output[id + 1 + nx*nz]; 
        float c110 = wavefield_output[id + nz + nx*nz]; 
        float c111 = wavefield_output[id + 1 + nz + nx*nz];

        float xd = (x - x0) / (x1 - x0);
        float yd = (y - y0) / (y1 - y0);
        float zd = (z - z0) / (z1 - z0);

        float c00 = c000*(1 - xd) + c100*xd;    
        float c01 = c001*(1 - xd) + c101*xd;    
        float c10 = c010*(1 - xd) + c110*xd;    
        float c11 = c011*(1 - xd) + c111*xd;    

        float c0 = c00*(1 - yd) + c10*yd;
        float c1 = c01*(1 - yd) + c11*yd;

        receiver_output[r] = c0*(1 - zd) + c1*zd;
    }

    receiver_output_file = receiver_output_folder + "first_arrivals_" + std::to_string(geometry->nodes.total) + "_shot_" + std::to_string(shot_id+1) + ".bin";
}

void Eikonal::pad_expansion()
{
    for (int z = 0; z < nz; z++)
    {
        for (int y = 0; y < ny; y++)
        {
            for (int x = 0; x < nx; x++)
            {
                S[z + x*nzz + y*nxx*nzz] = 1.0f / vp[z + x*nz + y*nx*nz];
            }
        }
    }

    for (int z = 0; z < pdz; z++)
    {
        for (int y = 0; y < nyy - pdy; y++)
        {
            for (int x = 0; x < nxx - pdx; x++)
            {
                S[(nzz - z - 1) + x*nzz + y*nxx*nzz] = 1.0f / vp[(nz - 1) + x*nz + y*nx*nz];
            }
        }
    }

    for (int x = 0; x < pdx; x++)
    {
        for (int z = 0; z < nzz; z++)
        {
            for (int y = 0; y < nyy - pdy; y++)
            {
                S[z + (nxx - x - 1)*nzz + y*nxx*nzz] = S[z + (nxx - pdx - 1)*nzz + y*nxx*nzz];
            }
        }
    }

    for (int y = 0; y < pdy; y++)
    {
        for (int z = 0; z < nzz; z++)
        {
            for (int x = 0; x < nxx; x++)
            {
                S[z + x*nzz + (nyy - y - 1)*nxx*nzz] = S[z + x*nzz + (nyy - pdy - 1)*nxx*nzz];
            }
        }
    }
}

void Eikonal::fdm_expansion()
{
    // Centering
    for (int z = padb; z < nzz - padb; z++)
    {
        for (int y = padb; y < nyy - padb; y++)
        {
            for (int x = padb; x < nxx - padb; x++)
            {
                S[z + x*nzz + y*nxx*nzz] = 1.0f / vp[(z - padb) + (x - padb)*nz + (y - padb)*nx*nz];
            }
        }
    }

    // Z direction
    for (int z = 0; z < padb; z++)
    {
        for (int y = padb; y < nyy - padb; y++)
        {
            for (int x = padb; x < nxx - padb; x++)
            {
                S[z + x*nzz + y*nxx*nzz] = 1.0f / vp[0 + (x - padb)*nz + (y - padb)*nx*nz];
                S[(nzz - z - 1) + x*nzz + y*nxx*nzz] = 1.0f / vp[(nz - 1) + (x - padb)*nz + (y - padb)*nx*nz];
            }
        }
    }

    // X direction
    for (int x = 0; x < padb; x++)
    {
        for (int z = 0; z < nzz; z++)
        {
            for (int y = padb; y < nyy - padb; y++)
            {
                S[z + x*nzz + y*nxx*nzz] = S[z + padb*nzz + y*nxx*nzz];
                S[z + (nxx - x - 1)*nzz + y*nxx*nzz] = S[z + (nxx - padb - 1)*nzz + y*nxx*nzz];
            }
        }
    }

    // Y direction
    for (int y = 0; y < padb; y++)
    {
        for (int z = 0; z < nzz; z++)
        {
            for (int x = 0; x < nxx; x++)
            {
                S[z + x*nzz + y*nxx*nzz] = S[z + x*nzz + padb*nxx*nzz];
                S[z + x*nzz + (nyy - y - 1)*nxx*nzz] = S[z + x*nzz + (nyy - padb - 1)*nxx*nzz];
            }
        }
    }
}

void Eikonal::pad_reduction()
{
    for (int index = 0; index < nPoints; index++)
    {
        int y = (int) (index / (nx*nz));         
        int x = (int) (index - y*nx*nz) / nz;    
        int z = (int) (index - x*nz - y*nx*nz);  

        wavefield_output[z + x*nz + y*nx*nz] = T[z + x*nzz + y*nxx*nzz];
    }
}

void Eikonal::fdm_reduction()
{
    for (int index = 0; index < nPoints; index++)
    {
        int y = (int) (index / (nx*nz));         
        int x = (int) (index - y*nx*nz) / nz;    
        int z = (int) (index - x*nz - y*nx*nz);  

        wavefield_output[z + x*nz + y*nx*nz] = T[(z + padb) + (x + padb)*nzz + (y + padb)*nxx*nzz];
    }
}
