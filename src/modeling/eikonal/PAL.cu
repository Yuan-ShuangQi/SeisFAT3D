#include "hip/hip_runtime.h"
# include "PAL.cuh"

void Eikonal_pal::parameters()
{
    padb = 1;

    nxx = nx + 2*padb;
    nyy = ny + 2*padb;
    nzz = nz + 2*padb;

    title = "Eikonal solver for acoustic isotropic media\n\nSolving eikonal equation with the \033[32mPodvin & Lecomte (1991)\033[0;0m formulation\n";    
}

void Eikonal_pal::components() 
{ 
    K = new float[volsize]();

    hipMalloc((void**)&(d_S), volsize*sizeof(float));     
    hipMalloc((void**)&(d_T), volsize*sizeof(float));     
    hipMalloc((void**)&(d_K), volsize*sizeof(float));     
    hipMalloc((void**)&(d_nK), volsize*sizeof(float));     
    hipMalloc((void**)&(d_nT), volsize*sizeof(float));     
}

void Eikonal_pal::initial_setup()
{
    nit = 0;
    
    int sidx = (int)(geometry->shots.x[shot_id] / dh) + padb;
    int sidy = (int)(geometry->shots.y[shot_id] / dh) + padb;
    int sidz = (int)(geometry->shots.z[shot_id] / dh) + padb;

    int sId = sidz + sidx*nzz + sidy*nxx*nzz;

    float t0 = S[sId] * sqrtf(powf((float)((sidx-padb)*dh) - geometry->shots.x[shot_id], 2.0f) +
                              powf((float)((sidy-padb)*dh) - geometry->shots.y[shot_id], 2.0f) +
                              powf((float)((sidz-padb)*dh) - geometry->shots.z[shot_id], 2.0f));

    for (int index = 0; index < volsize; index++)
    {    
        T[index] = 1e6f;
        K[index] = 0.0f;
    }

    T[sId] = S[sId] * sqrtf(powf((sidx-padb)*dh - geometry->shots.x[shot_id], 2.0f) + powf((sidy-padb)*dh - geometry->shots.y[shot_id], 2.0f) + powf((sidz-padb)*dh - geometry->shots.z[shot_id], 2.0f));

    int aux = (int)sqrtf(powf(sidx, 2.0f) + powf(sidy,2.0f) + powf(sidz,2.0f)); 
    if (aux > nit) nit = aux;

    aux = (int)sqrtf(powf(nxx - sidx,2.0f) + powf(sidy,2.0f) + powf(sidz,2.0f));
    if (aux > nit) nit = aux;

    aux = (int)sqrtf(powf(sidx,2.0f) + powf(nyy - sidy,2.0f) + powf(sidz,2.0f)); 
    if (aux > nit) nit = aux;

    aux = (int)sqrtf(powf(sidx,2.0f) + powf(sidy,2.0f) + powf(nzz - sidz,2.0f)); 
    if (aux > nit) nit = aux;

    aux = (int)sqrtf(powf(sidx,2.0f) + powf(nyy - sidy,2.0f) + powf(nzz - sidz,2.0f));
    if (aux > nit) nit = aux;

    aux = (int)sqrtf(powf(nxx - sidx,2.0f) + powf(sidy,2.0f) + powf(nzz - sidz,2.0f));
    if (aux > nit) nit = aux;

    aux = (int)sqrtf(powf(nxx - sidx,2.0f) + powf(nyy - sidy,2.0f) + powf(sidz,2.0f));
    if (aux > nit) nit = aux;

    aux = (int)sqrtf(powf(nxx - sidx,2.0f) + powf(nyy - sidy,2.0f) + powf(nzz - sidz,2.0f));
    if (aux > nit) nit = aux;

    K[sId - 1] = 1.0f;
    K[sId + 1] = 1.0f;
    K[sId - nzz] = 1.0f;
    K[sId + nzz] = 1.0f;
    K[sId - nxx*nzz] = 1.0f;
    K[sId + nxx*nzz] = 1.0f;
    K[sId + 1 - nzz] = 1.0f;
    K[sId - 1 - nzz] = 1.0f;
    K[sId + 1 + nzz] = 1.0f;
    K[sId - 1 + nzz] = 1.0f;
    K[sId + 1 + nxx*nzz] = 1.0f;
    K[sId + 1 - nxx*nzz] = 1.0f;
    K[sId - 1 + nxx*nzz] = 1.0f;
    K[sId - 1 - nxx*nzz] = 1.0f;
    K[sId - nzz - nxx*nzz] = 1.0f;
    K[sId - nzz + nxx*nzz] = 1.0f;
    K[sId + nzz - nxx*nzz] = 1.0f;
    K[sId + nzz + nxx*nzz] = 1.0f;
    K[sId + 1 + nzz + nxx*nzz] = 1.0f;
    K[sId + 1 + nzz - nxx*nzz] = 1.0f;
    K[sId + 1 - nzz + nxx*nzz] = 1.0f;
    K[sId + 1 - nzz - nxx*nzz] = 1.0f;
    K[sId - 1 - nzz - nxx*nzz] = 1.0f;
    K[sId - 1 - nzz + nxx*nzz] = 1.0f;
    K[sId - 1 + nzz - nxx*nzz] = 1.0f;
    K[sId - 1 + nzz + nxx*nzz] = 1.0f;

    hipMemcpy(d_K, K, volsize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_S, S, volsize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_T, T, volsize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_nT, T, volsize*sizeof(float), hipMemcpyHostToDevice);
}

void Eikonal_pal::expansion()
{
    for (int z = padb; z < nzz - padb; z++)
    {
        for (int y = padb; y < nyy - padb; y++)
        {
            for (int x = padb; x < nxx - padb; x++)
            {
                S[z + x*nzz + y*nxx*nzz] = 1.0f / V[(z - padb) + (x - padb)*nz + (y - padb)*nx*nz];
            }
        }
    }

    for (int z = 0; z < padb; z++)
    {
        for (int y = padb; y < nyy - padb; y++)
        {
            for (int x = padb; x < nxx - padb; x++)
            {
                S[z + x*nzz + y*nxx*nzz] = 1.0f / V[0 + (x - padb)*nz + (y - padb)*nx*nz];
                S[(nzz - z - 1) + x*nzz + y*nxx*nzz] = 1.0f / V[(nz - 1) + (x - padb)*nz + (y - padb)*nx*nz];
            }
        }
    }

    for (int x = 0; x < padb; x++)
    {
        for (int z = 0; z < nzz; z++)
        {
            for (int y = padb; y < nyy - padb; y++)
            {
                S[z + x*nzz + y*nxx*nzz] = S[z + padb*nzz + y*nxx*nzz];
                S[z + (nxx - x - 1)*nzz + y*nxx*nzz] = S[z + (nxx - padb - 1)*nzz + y*nxx*nzz];
            }
        }
    }

    for (int y = 0; y < padb; y++)
    {
        for (int z = 0; z < nzz; z++)
        {
            for (int x = 0; x < nxx; x++)
            {
                S[z + x*nzz + y*nxx*nzz] = S[z + x*nzz + padb*nxx*nzz];
                S[z + x*nzz + (nyy - y - 1)*nxx*nzz] = S[z + x*nzz + (nyy - padb - 1)*nxx*nzz];
            }
        }
    }
}

void Eikonal_pal::reduction()
{
    for (int index = 0; index < nPoints; index++)
    {
        int y = (int) (index / (nx*nz));         
        int x = (int) (index - y*nx*nz) / nz;    
        int z = (int) (index - x*nz - y*nx*nz);  

        wavefield_output[z + x*nz + y*nx*nz] = T[(z + padb) + (x + padb)*nzz + (y + padb)*nxx*nzz];
    }
}

void Eikonal_pal::forward_solver()
{
    int nThreads = 256;
    int nBlocks = volsize / nThreads;

    for (int it = 0; it < nit; it++)
    {
        equations<<<nBlocks,nThreads>>>(d_S, d_T, d_K, d_nT, dh, nxx, nyy, nzz);        
        hipDeviceSynchronize();

        hipMemset(d_nK, 0.0f, volsize*sizeof(float));

        wavefront<<<nBlocks,nThreads>>>(d_K, d_nK, nxx, nyy, nzz);
        hipDeviceSynchronize();

        update<<<nBlocks,nThreads>>>(d_T, d_nT, d_K, d_nK, volsize);
        hipDeviceSynchronize();
    }

    hipMemcpy(T, d_T, volsize*sizeof(float), hipMemcpyDeviceToHost);
}

void Eikonal_pal::free_space()
{
    delete[] S;
    delete[] T;
    delete[] K;

    hipFree(d_K);
    hipFree(d_S);
    hipFree(d_T);
    hipFree(d_nK);
    hipFree(d_nT);
}

__global__ void equations(float * S, float * T, float * K, float * nT, float h, int nxx, int nyy, int nzz)
{
    float sqrt2 = sqrtf(2.0f);
    float sqrt3 = sqrtf(3.0f);

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if ((K[index] == 1.0f) && (index < nxx*nyy*nzz))
    {
        int k = (int) (index / (nxx*nzz));         // y direction
        int j = (int) (index - k*nxx*nzz) / nzz;   // x direction
        int i = (int) (index - j*nzz - k*nxx*nzz); // z direction

        if ((i > 0) && (i < nzz-1) && (j > 0) && (j < nxx-1) && (k > 0) && (k < nyy-1))
        {
            float lowest = T[index];
            float Tijk, T1, T2, Sref, M, N, P, Q, hs2;    

            /* 1D operator head wave: i,j-1,k -> i,j,k (x direction) */
            Tijk = T[index - nzz] + h * min(S[index - nzz], 
                                        min(S[index - 1 - nzz], 
                                        min(S[index - nzz - nxx*nzz], S[index - 1 - nzz - nxx*nzz]))); 
            
            if (Tijk < lowest) lowest = Tijk;

            /* 1D operator head wave: i,j+1,k -> i,j,k (x direction) */
            Tijk = T[index + nzz] + h * min(S[index], 
                                        min(S[index - 1], 
                                        min(S[index - nxx*nzz], S[index - 1 - nxx*nzz])));
            
            if (Tijk < lowest) lowest = Tijk;

            /* 1D operator head wave: i,j,k-1 -> i,j,k (y direction) */
            Tijk = T[index - nxx*nzz] + h * min(S[index - nxx*nzz], 
                                            min(S[index - nzz - nxx*nzz], 
                                            min(S[index - 1 - nxx*nzz], S[index - 1 - nzz - nxx*nzz]))); 
            
            if (Tijk < lowest) lowest = Tijk;

            /* 1D operator head wave: i,j,k+1 -> i,j,k (y direction) */
            Tijk = T[index + nxx*nzz] + h * min(S[index],
                                            min(S[index - 1], 
                                            min(S[index - nzz], S[index - 1 - nzz]))); 
            
            if (Tijk < lowest) lowest = Tijk;

            /* 1D operator head wave: i-1,j,k -> i,j,k (z direction) */
            Tijk = T[index - 1] + h * min(S[index - 1], 
                                      min(S[index - 1 - nzz], 
                                      min(S[index - 1 - nxx*nzz], S[index - 1 - nzz - nxx*nzz]))); 
            
            if (Tijk < lowest) lowest = Tijk;

            /* 1D operator head wave: i+1,j,k -> i,j,k (z direction) */
            Tijk = T[index + 1] + h * min(S[index], 
                                      min(S[index - nzz], 
                                      min(S[index - nxx*nzz], S[index - nzz - nxx*nzz]))); 
            
            if (Tijk < lowest) lowest = Tijk;
                
            /* 1D operator diffraction XZ plane */
            
            // i-1,j-1,k -> i,j,k
            Tijk = T[index - 1 - nzz] + h*sqrt2*min(S[index - 1 - nzz], S[index - 1 - nzz - nxx*nzz]); 
            if (Tijk < lowest) lowest = Tijk;

            // i-1,j+1,k -> i,j,k
            Tijk = T[index - 1 + nzz] + h*sqrt2*min(S[index - 1], S[index - 1 - nxx*nzz]); 
            if (Tijk < lowest) lowest = Tijk;
            
            // i+1,j-1,k -> i,j,k
            Tijk = T[index + 1 - nzz] + h*sqrt2*min(S[index - nzz], S[index - nzz - nxx*nzz]); 
            if (Tijk < lowest) lowest = Tijk;
            
            // i+1,j+1,k -> i,j,k
            Tijk = T[index + 1 + nzz] + h*sqrt2*min(S[index], S[index - nxx*nzz]); 
            if (Tijk < lowest) lowest = Tijk;

            /* 1D operator diffraction YZ plane */

            // i-1,j,k-1 -> i,j,k
            Tijk = T[index - 1 - nxx*nzz] + h*sqrt2*min(S[index - 1 - nxx*nzz], S[index - 1 - nzz - nxx*nzz]); 
            if (Tijk < lowest) lowest = Tijk;

            // i-1,j,k+1 -> i,j,k
            Tijk = T[index - 1 + nxx*nzz] + h*sqrt2*min(S[index - 1], S[index - 1 - nzz]); 
            if (Tijk < lowest) lowest = Tijk;
            
            // i+1,j,k-1 -> i,j,k
            Tijk = T[index + 1 - nxx*nzz] + h*sqrt2*min(S[index - nxx*nzz], S[index - nzz - nxx*nzz]); 
            if (Tijk < lowest) lowest = Tijk;
            
            // i+1,j,k+1 -> i,j,k
            Tijk = T[index + 1 + nxx*nzz] + h*sqrt2*min(S[index], S[index - nzz]); 
            if (Tijk < lowest) lowest = Tijk;

            /* 1D operator diffraction XY plane */
            
            // i,j-1,k-1 -> i,j,k
            Tijk = T[index - nzz - nxx*nzz] + h*sqrt2*min(S[index - nzz - nxx*nzz], S[index - 1 - nzz - nxx*nzz]); 
            if (Tijk < lowest) lowest = Tijk;

            // i,j-1,k+1 -> i,j,k
            Tijk = T[index - nzz + nxx*nzz] + h*sqrt2*min(S[index - nzz], S[index - 1 - nzz]); 
            if (Tijk < lowest) lowest = Tijk;

            // i,j+1,k-1 -> i,j,k
            Tijk = T[index + nzz - nxx*nzz] + h*sqrt2*min(S[index - nxx*nzz], S[index - 1 - nxx*nzz]); 
            if (Tijk < lowest) lowest = Tijk;

            // i,j+1,k+1 -> i,j,k
            Tijk = T[index + nzz + nxx*nzz] + h*sqrt2*min(S[index], S[index - 1]); 
            if (Tijk < lowest) lowest = Tijk;

            /* 1D operator corner diffractions */

            // i-1,j-1,k-1 -> i,j,k
            Tijk = T[index - 1 - nzz - nxx*nzz] + h*sqrt3*S[index - 1 - nzz - nxx*nzz]; 
            if (Tijk < lowest) lowest = Tijk;

            // i-1,j-1,k+1 -> i,j,k
            Tijk = T[index - 1 - nzz + nxx*nzz] + h*sqrt3*S[index - 1 - nzz]; 
            if (Tijk < lowest) lowest = Tijk;

            // i+1,j-1,k-1 -> i,j,k
            Tijk = T[index + 1 - nzz - nxx*nzz] + h*sqrt3*S[index - nzz - nxx*nzz]; 
            if (Tijk < lowest) lowest = Tijk;

            // i+1,j-1,k+1 -> i,j,k
            Tijk = T[index + 1 - nzz + nxx*nzz] + h*sqrt3*S[index - nzz]; 
            if (Tijk < lowest) lowest = Tijk;

            // i-1,j+1,k-1 -> i,j,k
            Tijk = T[index - 1 + nzz - nxx*nzz] + h*sqrt3*S[index - 1 - nxx*nzz]; 
            if (Tijk < lowest) lowest = Tijk;

            // i-1,j+1,k+1 -> i,j,k
            Tijk = T[index - 1 + nzz + nxx*nzz] + h*sqrt3*S[index - 1]; 
            if (Tijk < lowest) lowest = Tijk;

            // i+1,j+1,k-1 -> i,j,k
            Tijk = T[index + 1 + nzz - nxx*nzz] + h*sqrt3*S[index - nxx*nzz]; 
            if (Tijk < lowest) lowest = Tijk;

            // i+1,j+1,k+1 -> i,j,k
            Tijk = T[index + 1 + nzz + nxx*nzz] + h*sqrt3*S[index]; 
            if (Tijk < lowest) lowest = Tijk;

            /* 2D operator XZ plane: First Quadrant*/

            Sref = min(S[index - 1 - nzz], S[index - 1 - nzz - nxx*nzz]);

            // i,j-1,k - i-1,j-1,k -> i,j,k
            T1 = T[index - nzz];
            T2 = T[index - 1 - nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i-1,j,k - i-1,j-1,k -> i,j,k
            T1 = T[index - 1];
            T2 = T[index - 1 - nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator XZ plane: Second Quadrant*/                        

            Sref = min(S[index - nzz], S[index - nzz - nxx*nzz]);

            // i,j-1,k - i+1,j-1,k -> i,j,k
            T1 = T[index - nzz];
            T2 = T[index + 1 - nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i+1,j,k - i+1,j-1,k -> i,j,k
            T1 = T[index + 1];
            T2 = T[index + 1 - nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator XZ plane: Third Quadrant*/                        

            Sref = min(S[index], S[index - nxx*nzz]);

            // i+1,j,k - i+1,j+1,k -> i,j,k
            T1 = T[index + 1];
            T2 = T[index + 1 + nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i,j+1,k - i+1,j+1,k -> i,j,k
            T1 = T[index + nzz];
            T2 = T[index + 1 + nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator XZ plane: Fourth Quadrant*/                        

            Sref = min(S[index - 1], S[index - 1 - nxx*nzz]);

            // i,j+1,k - i-1,j+1,k -> i,j,k
            T1 = T[index + nzz];
            T2 = T[index - 1 + nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i-1,j,k - i-1,j+1,k -> i,j,k
            T1 = T[index - 1];
            T2 = T[index - 1 + nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator YZ plane: First Quadrant */                        

            Sref = min(S[index - 1 - nxx*nzz], S[index - 1 - nzz - nxx*nzz]);

            // i,j,k-1 - i-1,j,k-1 -> i,j,k
            T1 = T[index - nxx*nzz];
            T2 = T[index - 1 - nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i-1,j,k - i-1,j,k-1 -> i,j,k
            T1 = T[index - 1];
            T2 = T[index - 1 - nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator YZ plane: Second Quadrant */                        

            Sref = min(S[index - nxx*nzz], S[index - nzz - nxx*nzz]);

            // i,j,k-1 - i+1,j,k-1 -> i,j,k
            T1 = T[index - nxx*nzz];
            T2 = T[index + 1 - nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i+1,j,k - i+1,j,k-1 -> i,j,k
            T1 = T[index + 1];
            T2 = T[index + 1 - nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator YZ plane: Third Quadrant*/                        

            Sref = min(S[index], S[index - nzz]);

            // i+1,j,k - i+1,j,k+1 -> i,j,k
            T1 = T[index + 1];
            T2 = T[index + 1 + nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i,j,k+1 - i+1,j,k+1 -> i,j,k
            T1 = T[index + nxx*nzz];
            T2 = T[index + 1 + nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator YZ plane: Fourth Quadrant*/                        

            Sref = min(S[index - 1], S[index - 1 - nzz]);

            // i,j,k+1 - i-1,j,k+1 -> i,j,k
            T1 = T[index + nxx*nzz];
            T2 = T[index - 1 + nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i-1,j,k - i-1,j,k+1 -> i,j,k
            T1 = T[index - 1];
            T2 = T[index - 1 + nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator XY plane: First Quadrant*/                        

            Sref = min(S[index - nzz - nxx*nzz], S[index - 1 - nzz - nxx*nzz]);

            // i,j-1,k - i,j-1,k-1 -> i,j,k
            T1 = T[index - nzz];
            T2 = T[index - nzz - nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i,j,k-1 - i,j-1,k-1 -> i,j,k
            T1 = T[index - nxx*nzz];
            T2 = T[index - nzz - nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator XY plane: Second Quadrant*/                        

            Sref = min(S[index - nzz], S[index - 1 - nzz]);

            // i,j-1,k - i,j-1,k+1 -> i,j,k
            T1 = T[index - nzz];
            T2 = T[index - nzz + nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i,j,k+1 - i,j-1,k+1 -> i,j,k
            T1 = T[index + nxx*nzz];
            T2 = T[index - nzz + nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator XY plane: Third Quadrant*/                        

            Sref = min(S[index], S[index - 1]);

            // i,j,k+1 - i,j+1,k+1 -> i,j,k
            T1 = T[index + nxx*nzz];
            T2 = T[index + nzz + nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i,j+1,k - i,j+1,k+1 -> i,j,k
            T1 = T[index + nzz];
            T2 = T[index + nzz + nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator XY plane: Fourth Quadrant*/                        

            Sref = min(S[index - nxx*nzz], S[index - 1 - nxx*nzz]);

            // i,j+1,k - i,j+1,k-1 -> i,j,k
            T1 = T[index + nzz];
            T2 = T[index + nzz - nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i,j,k-1 - i,j+1,k-1 -> i,j,k
            T1 = T[index - nxx*nzz];
            T2 = T[index + nzz - nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 3D operator - First octant: XY plane */

            Sref = S[index - 1 - nzz - nxx*nzz];
            hs2 = h*h*Sref*Sref;

            M = T[index - 1 - nzz - nxx*nzz]; /* i-1,j-1,k-1 */  
            N = T[index - 1 - nzz];           /* i-1,j-1, k  */  
            P = T[index - 1 - nxx*nzz];       /* i-1, j ,k-1 */
            Q = T[index - 1];                 /* i-1, j , k  */

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - First octant: YZ plane */

            M = T[index - 1 - nzz - nxx*nzz]; /* i-1,j-1,k-1 */   
            N = T[index - 1 - nzz];           /* i-1,j-1, k  */   
            P = T[index - nzz - nxx*nzz];     /*  i ,j-1,k-1 */   
            Q = T[index - nzz];               /*  i ,j-1, k  */   

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - First octant: XZ plane */

            M = T[index - 1 - nzz - nxx*nzz]; /* i-1,j-1,k-1 */   
            N = T[index - nzz - nxx*nzz];     /*  i ,j-1,k-1 */         
            P = T[index - 1 - nxx*nzz];       /* i-1, j ,k-1 */ 
            Q = T[index - nxx*nzz];           /*  i , j ,k-1 */       

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
            ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
            ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
            ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
            ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Second octant: XY plane */

            Sref = S[index - 1 - nxx*nzz];
            hs2 = h*h*Sref*Sref;

            M = T[index - 1 + nzz - nxx*nzz];  /* i-1,j+1,k-1 */ 
            N = T[index - 1 - nxx*nzz];        /* i-1, j ,k-1 */     
            P = T[index - 1 + nzz];            /* i-1,j+1, k  */
            Q = T[index - 1];                  /* i-1, j , k  */

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Second octant: YZ plane */

            M = T[index - 1 + nzz - nxx*nzz]; /* i-1,j+1,k-1 */   
            N = T[index - 1 + nzz];           /* i-1,j+1, k  */   
            P = T[index + nzz - nxx*nzz];     /*  i ,j+1,k-1 */   
            Q = T[index + nzz];               /*  i ,j+1, k  */   

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Second octant: XZ plane */

            M = T[index - 1 + nzz - nxx*nzz]; /* i-1,j+1,k-1 */   
            N = T[index - 1 - nxx*nzz];       /* i-1, j ,k-1 */       
            P = T[index + nzz - nxx*nzz];     /*  i ,j+1,k-1 */   
            Q = T[index - nxx*nzz];           /*  i , j ,k-1 */       

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Third octant: XY plane */

            Sref = S[index - 1];
            hs2 = h*h*Sref*Sref;

            M = T[index - 1 + nzz + nxx*nzz]; /* i-1,j+1,k+1 */   
            N = T[index - 1 + nzz];           /* i-1,j+1, k  */   
            P = T[index - 1 + nxx*nzz];       /* i-1, j ,k+1 */ 
            Q = T[index - 1];                 /* i-1, j , k  */ 

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Third octant: YZ plane */

            M = T[index - 1 + nzz + nxx*nzz]; /* i-1,j+1,k+1 */   
            N = T[index + nzz + nxx*nzz];     /*  i ,j+1,k+1 */         
            P = T[index - 1 + nzz];           /* i-1,j+1, k  */ 
            Q = T[index + nzz];               /*  i ,j+1, k  */   

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Third octant: XZ plane */

            M = T[index - 1 + nzz + nxx*nzz]; /* i-1,j+1,k+1 */   
            N = T[index - 1 + nxx*nzz];       /* i-1, j ,k+1 */       
            P = T[index + nzz + nxx*nzz];     /*  i ,j+1,k+1 */   
            Q = T[index + nxx*nzz];           /*  i , j ,k+1 */       

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Fourth octant: XY plane */

            Sref = S[index - 1 - nzz];
            hs2 = h*h*Sref*Sref;

            M = T[index - 1 - nzz + nxx*nzz]; /* i-1,j-1,k+1 */  
            N = T[index - 1 + nxx*nzz];       /* i-1, j ,k+1 */      
            P = T[index - 1 - nzz];           /* i-1,j-1, k  */
            Q = T[index - 1];                 /* i-1, j , k  */

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Fourth octant: YZ plane */

            M = T[index - 1 - nzz + nxx*nzz]; /* i-1,j-1,k+1 */  
            N = T[index - 1 - nzz];           /* i-1,j-1, k  */  
            P = T[index - nzz + nxx*nzz];     /*  i ,j-1,k+1 */  
            Q = T[index - nzz];               /*  i ,j-1, k  */  

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Fourth octant: XZ plane */

            M = T[index - 1 - nzz + nxx*nzz]; /* i-1,j-1,k+1 */   
            N = T[index - nzz + nxx*nzz];     /*  i ,j-1,k+1 */         
            P = T[index - 1 + nxx*nzz];       /* i-1, j ,k+1 */ 
            Q = T[index + nxx*nzz];           /*  i , j ,k+1 */       

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Fifth octant: XY plane */

            Sref = S[index - nzz - nxx*nzz];
            hs2 = h*h*Sref*Sref;

            M = T[index + 1 - nzz - nxx*nzz]; /* i+1,j-1,k-1 */  
            N = T[index + 1 - nxx*nzz];       /* i+1, j ,k-1 */      
            P = T[index + 1 - nzz];           /* i+1,j-1, k  */
            Q = T[index + 1];                 /* i+1, j , k  */

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Fifth octant: YZ plane */

            M = T[index + 1 - nzz - nxx*nzz]; /* i+1,j-1,k-1 */   
            N = T[index + 1 - nzz];           /* i+1,j-1, k  */   
            P = T[index - nzz - nxx*nzz];     /*  i ,j-1,k-1 */   
            Q = T[index - nzz];               /*  i ,j-1, k  */   

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
            ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
            ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
            ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
            ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Fifth octant: XZ plane */

            M = T[index + 1 - nzz - nxx*nzz]; /* i+1,j-1,k-1 */   
            N = T[index - nzz - nxx*nzz];     /*  i ,j-1,k-1 */         
            P = T[index + 1 - nxx*nzz];       /* i+1, j ,k-1 */ 
            Q = T[index - nxx*nzz];           /*  i , j ,k-1 */       

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Sixth octant: XY plane */

            Sref = S[index - nxx*nzz];
            hs2 = h*h*Sref*Sref;

            M = T[index + 1 + nzz - nxx*nzz]; /* i+1,j+1,k-1 */   
            N = T[index + 1 + nzz];           /* i+1,j+1, k  */   
            P = T[index + 1 - nxx*nzz];       /* i+1, j ,k-1 */ 
            Q = T[index + 1];                 /* i+1, j , k  */ 

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Sixth octant: YZ plane */

            M = T[index + 1 + nzz - nxx*nzz]; /* i+1,j+1,k-1 */   
            N = T[index + nzz - nxx*nzz];     /*  i ,j+1,k-1 */        
            P = T[index + 1 + nzz];           /* i+1,j+1, k  */
            Q = T[index + nzz];               /*  i ,j+1, k  */  

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Sixth octant: XZ plane */

            M = T[index + 1 + nzz - nxx*nzz]; /* i+1,j+1,k-1 */   
            N = T[index + 1 - nxx*nzz];       /* i+1, j ,k-1 */       
            P = T[index + nzz - nxx*nzz];     /*  i ,j+1,k-1 */   
            Q = T[index - nxx*nzz];           /*  i , j ,k-1 */       

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Seventh octant: XY plane */
            
            Sref = S[index - nzz];
            hs2 = h*h*Sref*Sref;

            M = T[index + 1 - nzz + nxx*nzz]; /* i+1,j-1,k+1 */   
            N = T[index + 1 - nzz];           /* i+1,j-1, k  */   
            P = T[index + 1 + nxx*nzz];       /* i+1, j ,k+1 */ 
            Q = T[index + 1];                 /* i+1, j , k  */ 

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Seventh octant: YZ plane */

            M = T[index + 1 - nzz + nxx*nzz]; /* i+1,j-1,k+1 */  
            N = T[index - nzz + nxx*nzz];     /*  i ,j-1,k+1 */        
            P = T[index + 1 - nzz];           /* i+1,j-1, k  */
            Q = T[index - nzz];               /*  i ,j-1, k  */  

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Seventh octant: XZ plane */

            M = T[index + 1 - nzz + nxx*nzz]; /* i+1,j-1,k+1 */    
            N = T[index + 1 + nxx*nzz];       /* i+1, j ,k+1 */        
            P = T[index - nzz + nxx*nzz];     /*  i ,j-1,k+1 */    
            Q = T[index + nxx*nzz];           /*  i , j ,k+1 */        

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Eighth octant: XY plane */

            Sref = S[index];
            hs2 = h*h*Sref*Sref;

            M = T[index + 1 + nzz + nxx*nzz]; /* i+1,j+1,k+1 */  
            N = T[index + 1 + nxx*nzz];       /* i+1, j ,k+1 */      
            P = T[index + 1 + nzz];           /* i+1,j+1, k  */
            Q = T[index + 1];                 /* i+1, j , k  */

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Eighth octant: YZ plane */

            M = T[index + 1 + nzz + nxx*nzz]; /* i+1,j+1,k+1 */   
            N = T[index + 1 + nzz];           /* i+1,j+1, k  */   
            P = T[index + nzz + nxx*nzz];     /*  i ,j+1,k+1 */   
            Q = T[index + nzz];               /*  i ,j+1, k  */   

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Eighth octant: XZ plane */

            M = T[index + 1 + nzz + nxx*nzz]; /* i+1,j+1,k+1 */   
            N = T[index + nzz + nxx*nzz];     /*  i ,j+1,k+1 */         
            P = T[index + 1 + nxx*nzz];       /* i+1, j ,k+1 */ 
            Q = T[index + nxx*nzz];           /*  i , j ,k+1 */       

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* Time atualization */
            if (lowest == T[index]) K[index] = 0.0f;

            nT[index] = lowest;
        }
    }
}

__global__ void wavefront(float * K, float * nK, int nxx, int nyy, int nzz)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if ((K[index] == 1.0f) && (index < nxx*nyy*nzz))
    {
        int k = (int) (index / (nxx*nzz));         // y direction
        int j = (int) (index - k*nxx*nzz) / nzz;   // x direction
        int i = (int) (index - j*nzz - k*nxx*nzz); // z direction

        if ((i > 0) && (i < nzz-1) && (j > 0) && (j < nxx-1) && (k > 0) && (k < nyy-1))
        {
            nK[index - 1] = 1.0f;
            nK[index + 1] = 1.0f;
            nK[index - nzz] = 1.0f;
            nK[index + nzz] = 1.0f;
            nK[index - nxx*nzz] = 1.0f;
            nK[index + nxx*nzz] = 1.0f;
            nK[index + 1 - nzz] = 1.0f;
            nK[index - 1 - nzz] = 1.0f;
            nK[index + 1 + nzz] = 1.0f;
            nK[index - 1 + nzz] = 1.0f;
            nK[index + 1 + nxx*nzz] = 1.0f;
            nK[index + 1 - nxx*nzz] = 1.0f;
            nK[index - 1 + nxx*nzz] = 1.0f;
            nK[index - 1 - nxx*nzz] = 1.0f;
            nK[index - nzz - nxx*nzz] = 1.0f;
            nK[index - nzz + nxx*nzz] = 1.0f;
            nK[index + nzz - nxx*nzz] = 1.0f;
            nK[index + nzz + nxx*nzz] = 1.0f;
            nK[index + 1 + nzz + nxx*nzz] = 1.0f;
            nK[index + 1 + nzz - nxx*nzz] = 1.0f;
            nK[index + 1 - nzz + nxx*nzz] = 1.0f;
            nK[index + 1 - nzz - nxx*nzz] = 1.0f;
            nK[index - 1 - nzz - nxx*nzz] = 1.0f;
            nK[index - 1 - nzz + nxx*nzz] = 1.0f;
            nK[index - 1 + nzz - nxx*nzz] = 1.0f;
            nK[index - 1 + nzz + nxx*nzz] = 1.0f;
        }
    }
}

__global__ void update(float * T, float * nT, float * K, float * nK, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
    {
        T[index] = nT[index];
        K[index] = nK[index];
    }
}
