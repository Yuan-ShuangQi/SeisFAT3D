#include "hip/hip_runtime.h"
# include "elastic.cuh"

void Elastic::set_models()
{
    std::string vp_file = catch_parameter("vp_model_file", file);
    std::string vs_file = catch_parameter("vs_model_file", file);
    std::string rho_file = catch_parameter("rho_model_file", file);

    float * vp = new float[nPoints]();
    float * vs = new float[nPoints]();
    float * p = new float[nPoints]();

    import_binary_float(vp_file, vp, nPoints);
    import_binary_float(vs_file, vs, nPoints);
    import_binary_float(rho_file, p, nPoints);

    float * b = new float[volsize]();
    float * l = new float[volsize]();
    float * m = new float[volsize]();

    expand_boundary(vp, l);
    expand_boundary(vs, m);
    expand_boundary(p, b);

    for (int index = 0; index < volsize; index++)
    {
        m[index] = b[index]*m[index]*m[index];
        l[index] = b[index]*l[index]*l[index] - 2.0f*m[index];
        b[index] = 1.0f / b[index];
    }

    hipMalloc((void**)&(L), volsize*sizeof(float));
    hipMalloc((void**)&(M), volsize*sizeof(float));
    hipMalloc((void**)&(B), volsize*sizeof(float));
    
    hipMemcpy(L, l, volsize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(M, m, volsize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B, b, volsize*sizeof(float), hipMemcpyHostToDevice);

    delete[] vp;    
    delete[] vs;    
    delete[] p;

    delete[] l;    
    delete[] m;    
    delete[] b;    
}

void Elastic::set_volumes()
{
    type_name = std::string("elastic");
    type_message = std::string("[5] - Elastic isotropic media");

    define_staggered_wavelet();

    hipMalloc((void**)&(P), volsize*sizeof(float));
    hipMalloc((void**)&(Vx), volsize*sizeof(float));
    hipMalloc((void**)&(Vy), volsize*sizeof(float));
    hipMalloc((void**)&(Vz), volsize*sizeof(float));
    hipMalloc((void**)&(Txx), volsize*sizeof(float));
    hipMalloc((void**)&(Tyy), volsize*sizeof(float));
    hipMalloc((void**)&(Tzz), volsize*sizeof(float));
    hipMalloc((void**)&(Txy), volsize*sizeof(float));
    hipMalloc((void**)&(Txz), volsize*sizeof(float));
    hipMalloc((void**)&(Tyz), volsize*sizeof(float));
}

void Elastic::initialization()
{
    hipMemset(P, 0.0f, volsize*sizeof(float));
    hipMemset(Vx, 0.0f, volsize*sizeof(float));
    hipMemset(Vy, 0.0f, volsize*sizeof(float));
    hipMemset(Vz, 0.0f, volsize*sizeof(float));
    hipMemset(Txx, 0.0f, volsize*sizeof(float));
    hipMemset(Tyy, 0.0f, volsize*sizeof(float));
    hipMemset(Tzz, 0.0f, volsize*sizeof(float));
    hipMemset(Txy, 0.0f, volsize*sizeof(float));
    hipMemset(Txz, 0.0f, volsize*sizeof(float));
    hipMemset(Tyz, 0.0f, volsize*sizeof(float));

    snap_index = 0;
}

void Elastic::set_forward_solver()
{
    for (time_index = 0; time_index < nt; time_index++)
    {
        display_progress();

        compute_velocity<<<blocksPerGrid,threadsPerBlock>>>(Vx,Vy,Vz,Txx,Tyy,Tzz,Txz,Tyz,Txy,B,wavelet,source_index,time_index,dx,dy,dz,dt,nxx,nyy,nzz);
        hipDeviceSynchronize();

        compute_stress<<<blocksPerGrid,threadsPerBlock>>>(Vx,Vy,Vz,Txx,Tyy,Tzz,Txz,Tyz,Txy,P,M,L,damp1D,damp2D,damp3D,dx,dy,dz,dt,nxx,nyy,nzz,nabc);
        hipDeviceSynchronize();

        get_wavefield_output();
        get_seismogram();
    }   

    get_receiver_output();
}

void Elastic::free_space()
{
    hipFree(Vx);
    hipFree(Vy);
    hipFree(Vz);
    hipFree(Txx);
    hipFree(Tyy);
    hipFree(Tzz);
    hipFree(Txy);
    hipFree(Txz);
    hipFree(Tyz);
}

__global__ void compute_velocity(float * Vx, float * Vy, float * Vz, float * Txx, float * Tyy, float * Tzz, float * Txz, float * Tyz, float * Txy, float * B, float * wavelet, int sId, int tId, float dx, float dy, float dz, float dt, int nxx, int nyy, int nzz)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int k = (int) (index / (nxx*nzz));         // y direction
    int j = (int) (index - k*nxx*nzz) / nzz;   // x direction
    int i = (int) (index - j*nzz - k*nxx*nzz); // z direction

    if (index == 0)
    {
        Txx[sId] += wavelet[tId] / (dx*dy*dz);
        Tyy[sId] += wavelet[tId] / (dx*dy*dz);
        Tzz[sId] += wavelet[tId] / (dx*dy*dz);
    }

    if((i >= 3) && (i < nzz-4) && (j > 3) && (j < nxx-3) && (k >= 3) && (k < nyy-4)) 
    {
        float dTxx_dx = (75.0f*(Txx[i + (j-4)*nzz + k*nxx*nzz] - Txx[i + (j+3)*nzz + k*nxx*nzz]) +
                       1029.0f*(Txx[i + (j+2)*nzz + k*nxx*nzz] - Txx[i + (j-3)*nzz + k*nxx*nzz]) +
                       8575.0f*(Txx[i + (j-2)*nzz + k*nxx*nzz] - Txx[i + (j+1)*nzz + k*nxx*nzz]) +
                     128625.0f*(Txx[i + j*nzz + k*nxx*nzz]     - Txx[i + (j-1)*nzz + k*nxx*nzz])) / (107520.0f*dx);

        float dTxy_dy = (75.0f*(Txy[i + j*nzz + (k-3)*nxx*nzz] - Txy[i + j*nzz + (k+4)*nxx*nzz]) +
                       1029.0f*(Txy[i + j*nzz + (k+3)*nxx*nzz] - Txy[i + j*nzz + (k-2)*nxx*nzz]) +
                       8575.0f*(Txy[i + j*nzz + (k-1)*nxx*nzz] - Txy[i + j*nzz + (k+2)*nxx*nzz]) +
                     128625.0f*(Txy[i + j*nzz + (k+1)*nxx*nzz] - Txy[i + j*nzz + k*nxx*nzz])) / (107520.0f*dy);

        float dTxz_dz = (75.0f*(Txz[(i-3) + j*nzz + k*nxx*nzz] - Txz[(i+4) + j*nzz + k*nxx*nzz]) +
                       1029.0f*(Txz[(i+3) + j*nzz + k*nxx*nzz] - Txz[(i-2) + j*nzz + k*nxx*nzz]) +
                       8575.0f*(Txz[(i-1) + j*nzz + k*nxx*nzz] - Txz[(i+2) + j*nzz + k*nxx*nzz]) +
                     128625.0f*(Txz[(i+1) + j*nzz + k*nxx*nzz] - Txz[i + j*nzz + k*nxx*nzz])) / (107520.0f*dz);

        float Bx = 0.5f*(B[i + (j+1)*nzz + k*nxx*nzz] + B[i + j*nzz + k*nxx*nzz]);

        Vx[index] += dt*Bx*(dTxx_dx + dTxy_dy + dTxz_dz); 
    }

    if((i >= 3) && (i < nzz-3) && (j >= 3) && (j < nxx-4) && (k > 3) && (k < nyy-3)) 
    {
        float dTxy_dx = (75.0f*(Txy[i + (j-3)*nzz + k*nxx*nzz] - Txy[i + (j+4)*nzz + k*nxx*nzz]) +
                       1029.0f*(Txy[i + (j+3)*nzz + k*nxx*nzz] - Txy[i + (j-2)*nzz + k*nxx*nzz]) +
                       8575.0f*(Txy[i + (j-1)*nzz + k*nxx*nzz] - Txy[i + (j+2)*nzz + k*nxx*nzz]) +
                     128625.0f*(Txy[i + (j+1)*nzz + k*nxx*nzz] - Txy[i + j*nzz + k*nxx*nzz])) / (107520.0f*dx);

        float dTyy_dy = (75.0f*(Tyy[i + j*nzz + (k-4)*nxx*nzz] - Tyy[i + j*nzz + (k+3)*nxx*nzz]) +
                       1029.0f*(Tyy[i + j*nzz + (k+2)*nxx*nzz] - Tyy[i + j*nzz + (k-3)*nxx*nzz]) +
                       8575.0f*(Tyy[i + j*nzz + (k-2)*nxx*nzz] - Tyy[i + j*nzz + (k+1)*nxx*nzz]) +
                     128625.0f*(Tyy[i + j*nzz + k*nxx*nzz]     - Tyy[i + j*nzz + (k-1)*nxx*nzz])) / (107520.0f*dy);

        float dTyz_dz = (75.0f*(Tyz[(i-3) + j*nzz + k*nxx*nzz] - Tyz[(i+4) + j*nzz + k*nxx*nzz]) +
                       1029.0f*(Tyz[(i+3) + j*nzz + k*nxx*nzz] - Tyz[(i-2) + j*nzz + k*nxx*nzz]) +
                       8575.0f*(Tyz[(i-1) + j*nzz + k*nxx*nzz] - Tyz[(i+2) + j*nzz + k*nxx*nzz]) +
                     128625.0f*(Tyz[(i+1) + j*nzz + k*nxx*nzz] - Tyz[i + j*nzz + k*nxx*nzz])) / (107520.0f*dz);

        float By = 0.5f*(B[i + j*nzz + (k+1)*nxx*nzz] + B[i + j*nzz + k*nxx*nzz]);

        Vy[index] += dt*By*(dTxy_dx + dTyy_dy + dTyz_dz); 
    }    

    if((i > 3) && (i < nzz-3) && (j >= 3) && (j < nxx-4) && (k >= 3) && (k < nyy-4)) 
    {
        float dTxz_dx = (75.0f*(Txz[i + (j-3)*nzz + k*nxx*nzz] - Txz[i + (j+4)*nzz + k*nxx*nzz]) +
                       1029.0f*(Txz[i + (j+3)*nzz + k*nxx*nzz] - Txz[i + (j-2)*nzz + k*nxx*nzz]) +
                       8575.0f*(Txz[i + (j-1)*nzz + k*nxx*nzz] - Txz[i + (j+2)*nzz + k*nxx*nzz]) +
                     128625.0f*(Txz[i + (j+1)*nzz + k*nxx*nzz] - Txz[i + j*nzz + k*nxx*nzz])) / (107520.0f*dx);

        float dTyz_dy = (75.0f*(Tyz[i + j*nzz + (k-3)*nxx*nzz] - Tyz[i + j*nzz + (k+4)*nxx*nzz]) +
                       1029.0f*(Tyz[i + j*nzz + (k+3)*nxx*nzz] - Tyz[i + j*nzz + (k-2)*nxx*nzz]) +
                       8575.0f*(Tyz[i + j*nzz + (k-1)*nxx*nzz] - Tyz[i + j*nzz + (k+2)*nxx*nzz]) +
                     128625.0f*(Tyz[i + j*nzz + (k+1)*nxx*nzz] - Tyz[i + j*nzz + k*nxx*nzz])) / (107520.0f*dy);

        float dTzz_dz = (75.0f*(Tzz[(i-4) + j*nzz + k*nxx*nzz] - Tzz[(i+3) + j*nzz + k*nxx*nzz]) +
                       1029.0f*(Tzz[(i+2) + j*nzz + k*nxx*nzz] - Tzz[(i-3) + j*nzz + k*nxx*nzz]) +
                       8575.0f*(Tzz[(i-2) + j*nzz + k*nxx*nzz] - Tzz[(i+1) + j*nzz + k*nxx*nzz]) +
                     128625.0f*(Tzz[i + j*nzz + k*nxx*nzz]     - Tzz[(i-1) + j*nzz + k*nxx*nzz])) / (107520.0f*dz);

        float Bz = 0.5f*(B[(i+1) + j*nzz + k*nxx*nzz] + B[i + j*nzz + k*nxx*nzz]);

        Vz[index] += dt*Bz*(dTxz_dx + dTyz_dy + dTzz_dz); 
    }
}

__global__ void compute_stress(float * Vx, float * Vy, float * Vz, float * Txx, float * Tyy, float * Tzz, float * Txz, float * Tyz, float * Txy, float * P, float * M, float * L, float * damp1D, float * damp2D, float * damp3D, float dx, float dy, float dz, float dt, int nxx, int nyy, int nzz, int nabc)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int k = (int) (index / (nxx*nzz));         // y direction
    int j = (int) (index - k*nxx*nzz) / nzz;   // x direction
    int i = (int) (index - j*nzz - k*nxx*nzz); // z direction

    if((i >= 3) && (i < nzz-4) && (j >= 3) && (j < nxx-4) && (k >= 3) && (k < nyy-4)) 
    {    
        float dVx_dx = (75.0f*(Vx[i + (j-3)*nzz + k*nxx*nzz] - Vx[i + (j+4)*nzz + k*nxx*nzz]) +
                      1029.0f*(Vx[i + (j+3)*nzz + k*nxx*nzz] - Vx[i + (j-2)*nzz + k*nxx*nzz]) +
                      8575.0f*(Vx[i + (j-1)*nzz + k*nxx*nzz] - Vx[i + (j+2)*nzz + k*nxx*nzz]) +
                    128625.0f*(Vx[i + (j+1)*nzz + k*nxx*nzz] - Vx[i + j*nzz + k*nxx*nzz])) / (107520.0f*dx);

        float dVy_dy = (75.0f*(Vy[i + j*nzz + (k-3)*nxx*nzz] - Vy[i + j*nzz + (k+4)*nxx*nzz]) +
                      1029.0f*(Vy[i + j*nzz + (k+3)*nxx*nzz] - Vy[i + j*nzz + (k-2)*nxx*nzz]) +
                      8575.0f*(Vy[i + j*nzz + (k-1)*nxx*nzz] - Vy[i + j*nzz + (k+2)*nxx*nzz]) +
                    128625.0f*(Vy[i + j*nzz + (k+1)*nxx*nzz] - Vy[i + j*nzz + k*nxx*nzz])) / (107520.0f*dy);

        float dVz_dz = (75.0f*(Vz[(i-3) + j*nzz + k*nxx*nzz] - Vz[(i+4) + j*nzz + k*nxx*nzz]) +
                      1029.0f*(Vz[(i+3) + j*nzz + k*nxx*nzz] - Vz[(i-2) + j*nzz + k*nxx*nzz]) +
                      8575.0f*(Vz[(i-1) + j*nzz + k*nxx*nzz] - Vz[(i+2) + j*nzz + k*nxx*nzz]) +
                    128625.0f*(Vz[(i+1) + j*nzz + k*nxx*nzz] - Vz[i + j*nzz + k*nxx*nzz])) / (107520.0f*dz);

        Txx[index] += dt*((L[index] + 2*M[index])*dVx_dx + L[index]*(dVy_dy + dVz_dz));
        Tyy[index] += dt*((L[index] + 2*M[index])*dVy_dy + L[index]*(dVx_dx + dVz_dz));
        Tzz[index] += dt*((L[index] + 2*M[index])*dVz_dz + L[index]*(dVx_dx + dVy_dy));                    
    }

    if((i >= 0) && (i < nzz) && (j > 3) && (j < nxx-3) && (k > 3) && (k < nyy-3)) 
    {
        float dVx_dy = (75.0f*(Vx[i + j*nzz + (k-4)*nxx*nzz] - Vx[i + j*nzz + (k+3)*nxx*nzz]) +
                      1029.0f*(Vx[i + j*nzz + (k+2)*nxx*nzz] - Vx[i + j*nzz + (k-3)*nxx*nzz]) +
                      8575.0f*(Vx[i + j*nzz + (k-2)*nxx*nzz] - Vx[i + j*nzz + (k+1)*nxx*nzz]) +
                    128625.0f*(Vx[i + j*nzz + k*nxx*nzz]     - Vx[i + j*nzz + (k-1)*nxx*nzz])) / (107520.0f*dy);

        float dVy_dx = (75.0f*(Vy[i + (j-4)*nzz + k*nxx*nzz] - Vy[i + (j+3)*nzz + k*nxx*nzz]) +
                      1029.0f*(Vy[i + (j+2)*nzz + k*nxx*nzz] - Vy[i + (j-3)*nzz + k*nxx*nzz]) +
                      8575.0f*(Vy[i + (j-2)*nzz + k*nxx*nzz] - Vy[i + (j+1)*nzz + k*nxx*nzz]) +
                    128625.0f*(Vy[i + j*nzz + k*nxx*nzz]     - Vy[i + (j-1)*nzz + k*nxx*nzz])) / (107520.0f*dx);

        float Mxy = powf(0.25f*(1.0f/M[i + (j+1)*nzz + (k+1)*nxx*nzz] + 1.0f/M[i + (j+1)*nzz + k*nxx*nzz] + 
                                1.0f/M[i + j*nzz + (k+1)*nxx*nzz]     + 1.0f/M[i + j*nzz + k*nxx*nzz]), -1.0f);

        Txy[index] += dt*Mxy*(dVx_dy + dVy_dx);
    }

    if((i > 3) && (i < nzz-3) && (j > 3) && (j < nxx-3) && (k >= 0) && (k < nyy)) 
    {
        float dVx_dz = (75.0f*(Vx[(i-4) + j*nzz + k*nxx*nzz] - Vx[(i+3) + j*nzz + k*nxx*nzz]) +
                      1029.0f*(Vx[(i+2) + j*nzz + k*nxx*nzz] - Vx[(i-3) + j*nzz + k*nxx*nzz]) +
                      8575.0f*(Vx[(i-2) + j*nzz + k*nxx*nzz] - Vx[(i+1) + j*nzz + k*nxx*nzz]) +
                    128625.0f*(Vx[i + j*nzz + k*nxx*nzz]     - Vx[(i-1) + j*nzz + k*nxx*nzz])) / (107520.0f*dz);

        float dVz_dx = (75.0f*(Vz[i + (j-4)*nzz + k*nxx*nzz] - Vz[i + (j+3)*nzz + k*nxx*nzz]) +
                      1029.0f*(Vz[i + (j+2)*nzz + k*nxx*nzz] - Vz[i + (j-3)*nzz + k*nxx*nzz]) +
                      8575.0f*(Vz[i + (j-2)*nzz + k*nxx*nzz] - Vz[i + (j+1)*nzz + k*nxx*nzz]) +
                    128625.0f*(Vz[i + j*nzz + k*nxx*nzz]     - Vz[i + (j-1)*nzz + k*nxx*nzz])) / (107520.0f*dx);

        float Mxz = powf(0.25f*(1.0f/M[(i+1) + (j+1)*nzz + k*nxx*nzz] + 1.0f/M[i + (j+1)*nzz + k*nxx*nzz] + 
                                1.0f/M[(i+1) + j*nzz + k*nxx*nzz]     + 1.0f/M[i + j*nzz + k*nxx*nzz]), -1.0f);

        Txz[index] += dt*Mxz*(dVx_dz + dVz_dx);
    }

    if((i > 3) && (i < nzz-3) && (j >= 0) && (j < nxx) && (k > 3) && (k < nyy-3)) 
    {
        float dVy_dz = (75.0f*(Vy[(i-4) + j*nzz + k*nxx*nzz] - Vy[(i+3) + j*nzz + k*nxx*nzz]) +
                      1029.0f*(Vy[(i+2) + j*nzz + k*nxx*nzz] - Vy[(i-3) + j*nzz + k*nxx*nzz]) +
                      8575.0f*(Vy[(i-2) + j*nzz + k*nxx*nzz] - Vy[(i+1) + j*nzz + k*nxx*nzz]) +
                    128625.0f*(Vy[i + j*nzz + k*nxx*nzz]     - Vy[(i-1) + j*nzz + k*nxx*nzz])) / (107520.0f*dz);

        float dVz_dy = (75.0f*(Vz[i + j*nzz + (k-4)*nxx*nzz] - Vz[i + j*nzz + (k+3)*nxx*nzz]) +
                      1029.0f*(Vz[i + j*nzz + (k+2)*nxx*nzz] - Vz[i + j*nzz + (k-3)*nxx*nzz]) +
                      8575.0f*(Vz[i + j*nzz + (k-2)*nxx*nzz] - Vz[i + j*nzz + (k+1)*nxx*nzz]) +
                    128625.0f*(Vz[i + j*nzz + k*nxx*nzz]     - Vz[i + j*nzz + (k-1)*nxx*nzz])) / (107520.0f*dy);

        float Myz = powf(0.25f*(1.0f/M[(i+1) + j*nzz + (k+1)*nxx*nzz] + 1.0f/M[i + j*nzz + (k+1)*nxx*nzz] + 
                                1.0f/M[(i+1) + j*nzz + k*nxx*nzz] +     1.0f/M[i + j*nzz + k*nxx*nzz]), -1.0f);

        Tyz[index] += dt*Myz*(dVy_dz + dVz_dy);
    }

    float damper = get_boundary_damper(damp1D,damp2D,damp3D,i,j,k,nxx,nyy,nzz,nabc);

    if (index < nxx*nyy*nzz) 
    {
        Vx[index] *= damper;
        Vy[index] *= damper;
        Vz[index] *= damper;

        Txx[index] *= damper;
        Tyy[index] *= damper;
        Tzz[index] *= damper;
        Txz[index] *= damper;
        Tyz[index] *= damper;
        Txy[index] *= damper;    

        P[index] = (Txx[index] + Tyy[index] + Tzz[index]) / 3.0f;
    }
}
