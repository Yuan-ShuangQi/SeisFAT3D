# include "acoustic.cuh"

void Acoustic::set_models()
{
    std::string vp_file = catch_parameter("vp_model_file", file);
    std::string rho_file = catch_parameter("rho_model_file", file);

    float * v = new float[nPoints]();
    float * p = new float[nPoints]();

    float * k = new float[volsize]();
    float * b = new float[volsize]();

    import_binary_float(vp_file, v, nPoints);
    import_binary_float(rho_file, p, nPoints);

    expand_boundary(v, k);
    expand_boundary(p, b);

    for (int index = 0; index < volsize; index++)
    {
        k[index] = b[index]*k[index]*k[index];
        b[index] = 1.0f / b[index];
    }

    hipMalloc((void**)&(K), volsize*sizeof(float));
    hipMalloc((void**)&(B), volsize*sizeof(float));
    
    hipMemcpy(K, k, volsize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B, b, volsize*sizeof(float), hipMemcpyHostToDevice);

    delete[] v;    
    delete[] p;    
    delete[] k;    
    delete[] b;    
}

void Acoustic::set_volumes()
{
    type_name = std::string("acoustic");
    type_message = std::string("[4] - Acoustic isotropic media");

    define_staggered_wavelet();

    hipMalloc((void**)&(P), volsize*sizeof(float));
    hipMalloc((void**)&(Vx), volsize*sizeof(float));
    hipMalloc((void**)&(Vy), volsize*sizeof(float));
    hipMalloc((void**)&(Vz), volsize*sizeof(float));
}

void Acoustic::initialization()
{
    hipMemset(P, 0.0f, volsize*sizeof(float));
    hipMemset(Vx, 0.0f, volsize*sizeof(float));
    hipMemset(Vy, 0.0f, volsize*sizeof(float));
    hipMemset(Vz, 0.0f, volsize*sizeof(float));
}

void Acoustic::set_forward_solver()
{
    for (time_index = 0; time_index < nt; time_index++)
    {
        // display_progression();

        // apply_wavelet<<<1,1>>>();

        // fdm_8E2T_get_velocity<<<>>>();
        // fdm_8E2T_get_pressure<<<>>>();

        // get_snapshots<<<>>>();
        // get_seismogram<<<>>>();
    }   
}