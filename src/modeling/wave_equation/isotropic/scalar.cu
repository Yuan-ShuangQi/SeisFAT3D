# include "scalar.cuh"

void Scalar::set_models()
{
    std::string vp_file = catch_parameter("vp_model_file", file);

    float * vp = new float[nPoints]();
    float * v = new float[volsize]();

    import_binary_float(vp_file, vp, nPoints);

    expand_boundary(vp, v);

    hipMalloc((void**)&(V), volsize*sizeof(float));
    
    hipMemcpy(V, v, volsize*sizeof(float), hipMemcpyHostToDevice);

    delete[] v;
    delete[] vp;    
}

void Scalar::set_volumes()
{
    type_name = std::string("scalar");
    type_message = std::string("[3] - Scalar isotropic media");

    define_common_wavelet();

    hipMalloc((void**)&(P), volsize*sizeof(float));
    hipMalloc((void**)&(Pold), volsize*sizeof(float));
    hipMalloc((void**)&(Pnew), volsize*sizeof(float));
}

void Scalar::initialization()
{
    hipMemset(P, 0.0f, volsize*sizeof(float));
    hipMemset(Pold, 0.0f, volsize*sizeof(float));
    hipMemset(Pnew, 0.0f, volsize*sizeof(float));
}

void Scalar::set_forward_solver()
{
    for (time_index = 0; time_index < nt; time_index++)
    {
        // display_progression();

        // apply_wavelet<<<1,1>>>();

        // fdm_8E2T_scalar<<<>>>();

        // update_wavefield<<<>>>();    

        // get_snapshots<<<>>>();
        // get_seismogram<<<>>>();
    }   
}