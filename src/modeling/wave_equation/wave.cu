#include "hip/hip_runtime.h"
# include "wave.cuh"

void Wave::set_specifics()
{
    nt = std::stoi(catch_parameter("time_samples", file));
    dt = std::stof(catch_parameter("time_spacing", file));

    fmax = std::stof(catch_parameter("max_frequency", file));

    nabc = std::stoi(catch_parameter("boundary_samples", file));
    pabc = std::stof(catch_parameter("boundary_damper", file));

    total_snaps = std::stoi(catch_parameter("total_snapshots", file));

    nbxl = nabc; nbxr = nabc;
    nbyl = nabc; nbyr = nabc;
    nbzu = nabc; nbzd = nabc;    

    define_cerjan_dampers();
    define_grid_nodes_position();    
}

void Wave::define_cerjan_dampers()
{
    float * d1D = new float[nabc]();
    float * d2D = new float[nabc*nabc]();
    float * d3D = new float[nabc*nabc*nabc]();

    float factor = std::stof(catch_parameter("boundary_damper", file));

    for (int i = 0; i < nabc; i++) 
    {
        d1D[i] = expf(-powf(factor * (nabc - i), 2.0f));
    }

    for(int i = 0; i < nabc; i++) 
    {
        for (int j = 0; j < nabc; j++)
        {   
            d2D[j + i*nabc] += d1D[i]; // up to bottom
            d2D[i + j*nabc] += d1D[i]; // left to right
        }
    }

    for (int i  = 0; i < nabc; i++)
    {
        for(int j = 0; j < nabc; j++)
        {
            for(int k = 0; k < nabc; k++)
            {
                d3D[i + j*nabc + k*nabc*nabc] += d2D[i + j*nabc]; // XY plane
                d3D[i + j*nabc + k*nabc*nabc] += d2D[j + k*nabc]; // ZX plane
                d3D[i + j*nabc + k*nabc*nabc] += d2D[i + k*nabc]; // ZY plane
            }
        }
    }    

    for (int index = 0; index < nabc*nabc; index++)
        d2D[index] -= 1.0f;

    for (int index = 0; index < nabc*nabc*nabc; index++)
        d3D[index] -= 5.0f;    

	hipMalloc((void**)&(damp1D), nabc*sizeof(float));
	hipMalloc((void**)&(damp2D), nabc*nabc*sizeof(float));
	hipMalloc((void**)&(damp3D), nabc*nabc*nabc*sizeof(float));

	hipMemcpy(damp1D, d1D, nabc*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(damp2D, d2D, nabc*nabc*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(damp3D, d3D, nabc*nabc*nabc*sizeof(float), hipMemcpyHostToDevice);

    delete[] d1D;
    delete[] d2D;
    delete[] d3D;    
}

void Wave::define_grid_nodes_position()
{
    int * rx = new int[total_nodes]();
    int * ry = new int[total_nodes]();
    int * rz = new int[total_nodes]();

    for (int index = 0; index < total_nodes; index++)
    {
        rx[index] = (int)(geometry->nodes.x[index] / dx) + nbxl;
        ry[index] = (int)(geometry->nodes.y[index] / dy) + nbyl;
        rz[index] = (int)(geometry->nodes.z[index] / dz) + nbzu;
    }

	hipMalloc((void**)&(grid_node_x), total_nodes*sizeof(int));
	hipMalloc((void**)&(grid_node_y), total_nodes*sizeof(int));
	hipMalloc((void**)&(grid_node_z), total_nodes*sizeof(int));

    hipMemcpy(grid_node_x, rx, total_nodes*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(grid_node_y, ry, total_nodes*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(grid_node_z, rz, total_nodes*sizeof(int), hipMemcpyHostToDevice);

    delete[] rx;
    delete[] ry;
    delete[] rz;
}

void Wave::set_outputs()
{   
    receiver_output_samples = nt*total_nodes;
    wavefield_output_samples = nPoints*total_snaps;

    if (export_receiver_output)
        receiver_output = new float[receiver_output_samples]();
    
    if (export_wavefield_output)
        wavefield_output = new float[wavefield_output_samples]();

    snapshot = new float[volsize]();

    hipMalloc((void**)&(seismogram), receiver_output_samples*sizeof(float));
}

void Wave::define_common_wavelet()
{
    float * signal = new float[nt]();

    float pi = 4.0f*atanf(1.0f);

    float t0 = 2.0f*sqrtf(pi)/fmax;
    float fc = fmax/(3.0f * sqrtf(pi));

    for (int n = 0; n < nt; n++)
    {
        float td = n*dt - t0;

        float arg = pi*pi*pi*fmax*fmax*td*td;

        signal[n] = (1.0f - 2.0f*arg)*expf(-arg);
    }
    
    hipMalloc((void**)&(wavelet), nt*sizeof(float));

    hipMemcpy(wavelet, signal, nt*sizeof(float), hipMemcpyHostToDevice);

    delete[] signal;
}

void Wave::define_staggered_wavelet()
{
    float * signal = new float[nt]();

    float pi = 4.0f*atanf(1.0f);

    float t0 = 2.0f*sqrtf(pi)/fmax;
    float fc = fmax/(3.0f * sqrtf(pi));

    float summation = 0;

    for (int n = 0; n < nt; n++)
    {
        float td = n*dt - t0;

        float arg = pi*pi*pi*fmax*fmax*td*td;

        summation += (1.0f - 2.0f*arg)*expf(-arg);

        signal[n] = summation;
    }
    
    hipMalloc((void**)&(wavelet), nt*sizeof(float));

    hipMemcpy(wavelet, signal, nt*sizeof(float), hipMemcpyHostToDevice);

    delete[] signal;
}

void Wave::display_progress()
{
    if (time_index % (nt / 10) == 0)
    {
        get_information();

        std::cout<<"Time progress: " << floorf(100.0f * (float)(time_index+1) / (float)(nt)) <<" %\n\n";
    }
}

void Wave::get_wavefield_output()
{
    if (export_wavefield_output)
    {
        wavefield_output_file = wavefield_output_folder + type_name + "_snapshot_" + std::to_string(nz) + "x" + std::to_string(nx) + "x" + std::to_string(ny) + "_shot_" + std::to_string(shot_index+1) + "_Nsnaps" + std::to_string(total_snaps) + ".bin";
        
        if (snap_index < total_snaps)
        {
            if (time_index % (int)((float)(nt) / (float)(total_snaps)) == 0)
            {
                hipMemcpy(snapshot, P, volsize*sizeof(float), hipMemcpyDeviceToHost);

                for (int index = 0; index < nPoints; index++)
                {
                    int y = (int) (index / (nx*nz));         
                    int x = (int) (index - y*nx*nz) / nz;    
                    int z = (int) (index - x*nz - y*nx*nz);  

                    wavefield_output[z + x*nz + y*nx*nz + snap_index*nPoints] = snapshot[(z + nbzu) + (x + nbxl)*nzz + (y + nbyl)*nxx*nzz];
                }

                snap_index += 1;
            }
        }
    }
}

void Wave::get_receiver_output()
{
    if (export_receiver_output)
    {
        hipMemcpy(receiver_output, seismogram, nt*total_nodes*sizeof(float), hipMemcpyDeviceToHost);

        receiver_output_file = receiver_output_folder + type_name + "_seismogram_Nsamples" + std::to_string(nt) + "_nRec" + std::to_string(total_nodes) + "_shot_" + std::to_string(shot_index+1) + ".bin";
    }
}

void Wave::get_seismogram()
{
    if (export_receiver_output)
    {
        int seismBlocks = (int)(total_nodes / threadsPerBlock) + 1;

        compute_seismogram<<<seismBlocks,threadsPerBlock>>>(seismogram,P,grid_node_x,grid_node_y,grid_node_z,total_nodes,nxx,nzz,nt,time_index);
    }
}

__global__ void compute_seismogram(float * seismogram, float * P, int * rx, int * ry, int * rz, int total_nodes, int nxx, int nzz, int nt, int time_id)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; 

    if (index < total_nodes) 
        seismogram[time_id + index*nt] = P[rz[index] + rx[index]*nzz + ry[index]*nxx*nzz];
}

__device__ float get_boundary_damper(float * damp1D, float * damp2D, float * damp3D, int i, int j, int k, int nxx, int nyy, int nzz, int nabc)
{
    float damper = 1.0f;

    // 1D damping
    if((i < nabc) && (j >= nabc) && (j < nxx-nabc) && (k >= nabc) && (k < nyy-nabc)) 
    {
        damper = damp1D[i];
    }         
    else if((i >= nzz-nabc) && (i < nzz) && (j >= nabc) && (j < nxx-nabc) && (k >= nabc) && (k < nyy-nabc)) 
    {
        damper = damp1D[nabc-(i-(nzz-nabc))-1];
    }         
    else if((i >= nabc) && (i < nzz-nabc) && (j >= 0) && (j < nabc) && (k >= nabc) && (k < nyy-nabc)) 
    {
        damper = damp1D[j];
    }
    else if((i >= nabc) && (i < nzz-nabc) && (j >= nxx-nabc) && (j < nxx) && (k >= nabc) && (k < nyy-nabc)) 
    {
        damper = damp1D[nabc-(j-(nxx-nabc))-1];
    }
    else if((i >= nabc) && (i < nzz-nabc) && (j >= nabc) && (j < nxx-nabc) && (k >= 0) && (k < nabc)) 
    {
        damper = damp1D[k];
    }
    else if((i >= nabc) && (i < nzz-nabc) && (j >= nabc) && (j < nxx-nabc) && (k >= nyy-nabc) && (k < nyy)) 
    {
        damper = damp1D[nabc-(k-(nyy-nabc))-1];
    }

    // 2D damping 
    else if((i >= nabc) && (i < nzz-nabc) && (j >= 0) && (j < nabc) && (k >= 0) && (k < nabc))
    {
        damper = damp2D[j + k*nabc];
    }
    else if((i >= nabc) && (i < nzz-nabc) && (j >= nxx-nabc) && (j < nxx) && (k >= 0) && (k < nabc))
    {
        damper = damp2D[nabc-(j-(nxx-nabc))-1 + k*nabc];
    }
    else if((i >= nabc) && (i < nzz-nabc) && (j >= 0) && (j < nabc) && (k >= nyy-nabc) && (k < nyy))
    {
        damper = damp2D[j + (nabc-(k-(nyy-nabc))-1)*nabc];
    }
    else if((i >= nabc) && (i < nzz-nabc) && (j >= nxx-nabc) && (j < nxx) && (k >= nyy-nabc) && (k < nyy))
    {
        damper = damp2D[nabc-(j-(nxx-nabc))-1 + (nabc-(k-(nyy-nabc))-1)*nabc];
    }

    else if((i >= 0) && (i < nabc) && (j >= nabc) && (j < nxx-nabc) && (k >= 0) && (k < nabc))
    {
        damper = damp2D[i + k*nabc];
    }
    else if((i >= nzz-nabc) && (i < nzz) && (j >= nabc) && (j < nxx-nabc) && (k >= 0) && (k < nabc))
    {
        damper = damp2D[nabc-(i-(nzz-nabc))-1 + k*nabc];
    }
    else if((i >= 0) && (i < nabc) && (j >= nabc) && (j < nxx-nabc) && (k >= nyy-nabc) && (k < nyy))
    {
        damper = damp2D[i + (nabc-(k-(nyy-nabc))-1)*nabc];
    }
    else if((i >= nzz-nabc) && (i < nzz) && (j >= nabc) && (j < nxx-nabc) && (k >= nyy-nabc) && (k < nyy))
    {
        damper = damp2D[nabc-(i-(nzz-nabc))-1 + (nabc-(k-(nyy-nabc))-1)*nabc];
    }

    else if((i >= 0) && (i < nabc) && (j >= 0) && (j < nabc) && (k >= nabc) && (k < nyy-nabc))
    {
        damper = damp2D[i + j*nabc];
    }
    else if((i >= nzz-nabc) && (i < nzz) && (j >= 0) && (j < nabc) && (k >= nabc) && (k < nyy-nabc))
    {
        damper = damp2D[nabc-(i-(nzz-nabc))-1 + j*nabc];
    }
    else if((i >= 0) && (i < nabc) && (j >= nxx-nabc) && (j < nxx) && (k >= nabc) && (k < nyy-nabc))
    {
        damper = damp2D[i + (nabc-(j-(nxx-nabc))-1)*nabc];
    }
    else if((i >= nzz-nabc) && (i < nzz) && (j >= nxx-nabc) && (j < nxx) && (k >= nabc) && (k < nyy-nabc))
    {
        damper = damp2D[nabc-(i-(nzz-nabc))-1 + (nabc-(j-(nxx-nabc))-1)*nabc];
    }

    // 3D damping
    else if((i >= 0) && (i < nabc) && (j >= 0) && (j < nabc) && (k >= 0) && (k < nabc))
    {
        damper = damp3D[i + j*nabc + k*nabc*nabc];
    }
    else if((i >= nzz-nabc) && (i < nzz) && (j >= 0) && (j < nabc) && (k >= 0) && (k < nabc))
    {
        damper = damp3D[nabc-(i-(nzz-nabc))-1 + j*nabc + k*nabc*nabc];
    }
    else if((i >= 0) && (i < nabc) && (j >= nxx-nabc) && (j < nxx) && (k >= 0) && (k < nabc))
    {
        damper = damp3D[i + (nabc-(j-(nxx-nabc))-1)*nabc + k*nabc*nabc];
    }
    else if((i >= 0) && (i < nabc) && (j >= 0) && (j < nabc) && (k >= nyy-nabc) && (k < nyy))
    {
        damper = damp3D[i + j*nabc + (nabc-(k-(nyy-nabc))-1)*nabc*nabc];
    }
    else if((i >= nzz-nabc) && (i < nzz) && (j >= nxx-nabc) && (j < nxx) && (k >= 0) && (k < nabc))
    {
        damper = damp3D[nabc-(i-(nzz-nabc))-1 + (nabc-(j-(nxx-nabc))-1)*nabc + k*nabc*nabc];
    }
    else if((i >= nzz-nabc) && (i < nzz) && (j >= 0) && (j < nabc) && (k >= nyy-nabc) && (k < nyy))
    {
        damper = damp3D[nabc-(i-(nzz-nabc))-1 + j*nabc + (nabc-(k-(nyy-nabc))-1)*nabc*nabc];
    }
    else if((i >= 0) && (i < nabc) && (j >= nxx-nabc) && (j < nxx) && (k >= nyy-nabc) && (k < nyy))
    {
        damper = damp3D[i + (nabc-(j-(nxx-nabc))-1)*nabc + (nabc-(k-(nyy-nabc))-1)*nabc*nabc];
    }
    else if((i >= nzz-nabc) && (i < nzz) && (j >= nxx-nabc) && (j < nxx) && (k >= nyy-nabc) && (k < nyy))
    {
        damper = damp3D[nabc-(i-(nzz-nabc))-1 + (nabc-(j-(nxx-nabc))-1)*nabc + (nabc-(k-(nyy-nabc))-1)*nabc*nabc];
    }

    return damper;
}