#include "hip/hip_runtime.h"
# include "scalar.cuh"

void Scalar::set_parameters()
{
    general_modeling_parameters();
    wave_modeling_parameters();

    set_acquisition_geometry();
    set_gridded_geometry();

    set_velocity_model();

    set_boundaries();
    set_model_boundaries();

    set_modeling_volumes();

    set_wavelet();
    set_dampers();
    set_outputs();
}

void Scalar::set_model_boundaries()
{
    float * vp = new float[volsize]();

    expand_boundary(Vp, vp);

    for (int index = 0; index < volsize; index++) 
    {
        vp[index] = dt*dt*vp[index]*vp[index];
    }

	hipMalloc((void**)&(dtVp2), volsize*sizeof(float));

	hipMemcpy(dtVp2, vp, volsize*sizeof(float), hipMemcpyHostToDevice);
    
    delete[] vp;
}

void Scalar::set_modeling_volumes()
{
    modeling_method = std::string("scalar");
    modeling_message = std::string("[3] - Constant density acoustic isotropic media\n\n");

	hipMalloc((void**)&(U_pre), volsize*sizeof(float));
	hipMalloc((void**)&(U_pas), volsize*sizeof(float));
    hipMalloc((void**)&(Pressure), volsize*sizeof(float));
}

void Scalar::set_wavelet()
{
    float * ricker = new float[nt]();

    for (int n = 0; n < nt; n++)
    {        
        float arg = pi*((n*dt - tlag)*fc*pi)*((n*dt - tlag)*fc*pi);
        
        ricker[n] = amp*(1 - 2*arg)*expf(-arg);    
    }

    if (import_wavelet) 
        import_binary_float(wavelet_file, ricker, nt);

	hipMalloc((void**)&(wavelet), nt*sizeof(float));
	hipMemcpy(wavelet, ricker, nt*sizeof(float), hipMemcpyHostToDevice);

    delete[] ricker;
}

void Scalar::info_message()
{
    general_modeling_message();
    
    set_modeling_message();
}

void Scalar::initial_setup()
{
    hipMemset(U_pre, 0.0f, volsize*sizeof(float));
    hipMemset(U_pas, 0.0f, volsize*sizeof(float));
    hipMemset(Pressure, 0.0f, volsize*sizeof(float));
    hipMemset(seismogram, 0.0f, nt*total_nodes*sizeof(float));

    int sidx = (int)(geometry->shots.x[shot_id] / dx) + nbxl;
    int sidy = (int)(geometry->shots.y[shot_id] / dy) + nbyl;
    int sidz = (int)(geometry->shots.z[shot_id] / dz) + nbzu;

    source_id = sidz + sidx*nzz + sidy*nxx*nzz;

    isnap = 0;
}

void Scalar::forward_solver()
{
    for (time_id = 0; time_id < nt; time_id++)
    {
        show_progress();
        
        compute_pressure<<<blocksPerGrid,threadsPerBlock>>>(Pressure,U_pre,U_pas,dtVp2,damp1D,damp2D,damp3D,wavelet,source_id,time_id,dx,dy,dz,nxx,nyy,nzz,nb,nbzu);
        hipDeviceSynchronize();

        update_pressure<<<blocksPerGrid,threadsPerBlock>>>(Pressure,U_pre,U_pas,volsize);
        hipDeviceSynchronize();
    
        get_snapshots();
        get_seismogram();
    }
}

void Scalar::free_space()
{
    hipFree(dtVp2);
    hipFree(U_pre);
    hipFree(U_pas);

    hipFree(damp1D);
    hipFree(damp2D);
    hipFree(damp3D);
    
    hipFree(wavelet);
    hipFree(Pressure);
    hipFree(seismogram);
}

__global__ void compute_pressure(float * Pressure, float * U_pre, float * U_pas, float * dtVp2, float * damp1D, float * damp2D, float * damp3D, float * wavelet, int source_id, int time_id, float dx, float dy, float dz, int nxx, int nyy, int nzz, int nb, int nbzu)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int k = (int) (index / (nxx*nzz));         // y direction
    int j = (int) (index - k*nxx*nzz) / nzz;   // x direction
    int i = (int) (index - j*nzz - k*nxx*nzz); // z direction
    
    if (index == 0) U_pre[source_id] += wavelet[time_id] / (dx*dy*dz);

    if((i >= 4) && (i < nzz-4) && (j >= 4) && (j < nxx-4) && (k >= 4) && (k < nyy-4)) 
    {
        float d2_Px2 = (- 9.0f*(U_pre[i + (j-4)*nzz + k*nxx*nzz] + U_pre[i + (j+4)*nzz + k*nxx*nzz])
                    +   128.0f*(U_pre[i + (j-3)*nzz + k*nxx*nzz] + U_pre[i + (j+3)*nzz + k*nxx*nzz])
                    -  1008.0f*(U_pre[i + (j-2)*nzz + k*nxx*nzz] + U_pre[i + (j+2)*nzz + k*nxx*nzz])
                    +  8064.0f*(U_pre[i + (j-1)*nzz + k*nxx*nzz] + U_pre[i + (j+1)*nzz + k*nxx*nzz])
                    - 14350.0f*(U_pre[i + j*nzz + k*nxx*nzz]))/(5040.0f*powf(dx, 2.0f));

        float d2_Py2 = (- 9.0f*(U_pre[i + j*nzz + (k-4)*nxx*nzz] + U_pre[i + j*nzz + (k+4)*nxx*nzz])
                    +   128.0f*(U_pre[i + j*nzz + (k-3)*nxx*nzz] + U_pre[i + j*nzz + (k+3)*nxx*nzz])
                    -  1008.0f*(U_pre[i + j*nzz + (k-2)*nxx*nzz] + U_pre[i + j*nzz + (k+2)*nxx*nzz])
                    +  8064.0f*(U_pre[i + j*nzz + (k-1)*nxx*nzz] + U_pre[i + j*nzz + (k+1)*nxx*nzz])
                    - 14350.0f*(U_pre[i + j*nzz + k*nxx*nzz]))/(5040.0f*powf(dy,2.0f));

        float d2_Pz2 = (- 9.0f*(U_pre[(i-4) + j*nzz + k*nxx*nzz] + U_pre[(i+4) + j*nzz + k*nxx*nzz])
                    +   128.0f*(U_pre[(i-3) + j*nzz + k*nxx*nzz] + U_pre[(i+3) + j*nzz + k*nxx*nzz])
                    -  1008.0f*(U_pre[(i-2) + j*nzz + k*nxx*nzz] + U_pre[(i+2) + j*nzz + k*nxx*nzz])
                    +  8064.0f*(U_pre[(i-1) + j*nzz + k*nxx*nzz] + U_pre[(i+1) + j*nzz + k*nxx*nzz])
                    - 14350.0f*(U_pre[i + j*nzz + k*nxx*nzz]))/(5040.0f*powf(dz,2.0f));
    
        
        Pressure[index] = dtVp2[index] * (d2_Px2 + d2_Py2 + d2_Pz2) + 2.0f*U_pre[index] - U_pas[index]; 
    }

    float damper = get_boundary_damper(damp1D,damp2D,damp3D,i,j,k,nxx,nyy,nzz,nb,nbzu);
    
    if (index < nxx*nyy*nzz)
    {
        U_pas[index] *= damper;    
        U_pre[index] *= damper;
        Pressure[index] *= damper;
    }
}

__global__ void update_pressure(float * Pressure, float * U_pre, float * U_pas, int volsize)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index < volsize)
    {
        U_pas[index] = U_pre[index];        
        U_pre[index] = Pressure[index];
    }
}