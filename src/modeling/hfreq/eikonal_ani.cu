#include "hip/hip_runtime.h"
# include "eikonal_ani.cuh"

void Eikonal_ANI::set_properties()
{
    Vp = new float[nPoints]();
    Ro = new float[nPoints]();

    S = new float[volsize]();
    qS = new float[volsize]();

    std::string vp_file = catch_parameter("vp_model_file", parameters);
    std::string ro_file = catch_parameter("ro_model_file", parameters);

    import_binary_float(vp_file, Vp, nPoints);
    import_binary_float(ro_file, Ro, nPoints);

    float * slowness = new float[nPoints]();

    # pragma omp parallel for
    for (int index = 0; index < nPoints; index++)
        slowness[index] = 1.0f / Vp[index];

    expand_boundary(slowness, S);

    # pragma omp parallel for
    for (int index = 0; index < volsize; index++)
        qS[index] = S[index];

    delete[] slowness;
}

void Eikonal_ANI::set_conditions()
{
    modeling_type = "eikonal_ani";
    modeling_name = "Modeling type: Anisotropic eikonal solver";

    n = 3;
    v = 6;

    p = new float[n]();
    C = new float[v*v]();
    G = new float[n*n]();
    Gv = new float[n]();

    float * Cij = new float[nPoints]();

    std::string Cijkl_folder = catch_parameter("Cijkl_folder", parameters);

    C11 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C11.bin", Cij, nPoints);
    compression(Cij, C11, nPoints, maxC11, minC11, compress);

    C12 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C12.bin", Cij, nPoints);
    compression(Cij, C12, nPoints, maxC12, minC12, compress);

    C13 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C13.bin", Cij, nPoints);
    compression(Cij, C13, nPoints, maxC13, minC13, compress);

    C14 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C14.bin", Cij, nPoints);
    compression(Cij, C14, nPoints, maxC14, minC14, compress);

    C15 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C15.bin", Cij, nPoints);
    compression(Cij, C15, nPoints, maxC15, minC15, compress);

    C16 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C16.bin", Cij, nPoints);
    compression(Cij, C16, nPoints, maxC16, minC16, compress);

    C22 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C22.bin", Cij, nPoints);
    compression(Cij, C22, nPoints, maxC22, minC22, compress);

    C23 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C23.bin", Cij, nPoints);
    compression(Cij, C23, nPoints, maxC23, minC23, compress);
    
    C24 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C24.bin", Cij, nPoints);
    compression(Cij, C24, nPoints, maxC24, minC24, compress);

    C25 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C25.bin", Cij, nPoints);
    compression(Cij, C25, nPoints, maxC25, minC25, compress);

    C26 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C26.bin", Cij, nPoints);
    compression(Cij, C26, nPoints, maxC26, minC26, compress);

    C33 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C33.bin", Cij, nPoints);
    compression(Cij, C33, nPoints, maxC33, minC33, compress);
    
    C34 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C34.bin", Cij, nPoints);
    compression(Cij, C34, nPoints, maxC34, minC34, compress);

    C35 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C35.bin", Cij, nPoints);
    compression(Cij, C35, nPoints, maxC35, minC35, compress);

    C36 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C36.bin", Cij, nPoints);
    compression(Cij, C36, nPoints, maxC36, minC36, compress);

    C44 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C44.bin", Cij, nPoints);
    compression(Cij, C44, nPoints, maxC44, minC44, compress);

    C45 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C45.bin", Cij, nPoints);
    compression(Cij, C45, nPoints, maxC45, minC45, compress);

    C46 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C46.bin", Cij, nPoints);
    compression(Cij, C46, nPoints, maxC46, minC46, compress);

    C55 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C55.bin", Cij, nPoints);
    compression(Cij, C55, nPoints, maxC55, minC55, compress);

    C56 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C56.bin", Cij, nPoints);
    compression(Cij, C56, nPoints, maxC56, minC56, compress);

    C66 = new uintc[nPoints]();
    import_binary_float(Cijkl_folder + "C66.bin", Cij, nPoints);
    compression(Cij, C66, nPoints, maxC66, minC66, compress);
}

void Eikonal_ANI::forward_solver()
{
    initialization();
    
    propagation();

    for (int index = 0; index < volsize; index++)
    {
        int k = (int) (index / (nxx*nzz));         
        int j = (int) (index - k*nxx*nzz) / nzz;    
        int i = (int) (index - j*nzz - k*nxx*nzz);  

        if ((i == sIdz) && (j == sIdx) && (k == sIdy))    
            continue;

        if ((i >= nb) && (i < nzz-nb) && (j >= nb) && (j < nxx-nb) && (k >= nb) && (k < nyy-nb))
        {
            aId = (i - nb) + (j - nb)*nz + (k - nb)*nx*nz;

            float dTz = 0.5f*(T[(i+1) + j*nzz + k*nxx*nzz] - T[(i-1) + j*nzz + k*nxx*nzz]) / dz;
            float dTx = 0.5f*(T[i + (j+1)*nzz + k*nxx*nzz] - T[i + (j-1)*nzz + k*nxx*nzz]) / dx;
            float dTy = 0.5f*(T[i + j*nzz + (k+1)*nxx*nzz] - T[i + j*nzz + (k-1)*nxx*nzz]) / dy;

            float norm = sqrtf(dTx*dTx + dTy*dTy + dTz*dTz);

            p[0] = dTx / norm;
            p[1] = dTy / norm;
            p[2] = dTz / norm;

            get_stiffness();
            get_christoffel();
            get_eigen_values();

            S[index] = 1.0f / sqrtf(Gv[0] * Ro[aId]);
        }
    }

    initialization();

    propagation();

    compute_seismogram();

    # pragma omp parallel for
    for (int index = 0; index < volsize; index++)
        S[index] = qS[index];
}

void Eikonal_ANI::get_stiffness()
{
    float c11 = (minC11 + (static_cast<float>(C11[aId]) - 1.0f) * (maxC11 - minC11) / ((float)(compress) - 1.0f))*conversion;
    float c12 = (minC12 + (static_cast<float>(C12[aId]) - 1.0f) * (maxC12 - minC12) / ((float)(compress) - 1.0f))*conversion;
    float c13 = (minC13 + (static_cast<float>(C13[aId]) - 1.0f) * (maxC13 - minC13) / ((float)(compress) - 1.0f))*conversion;
    float c14 = (minC14 + (static_cast<float>(C14[aId]) - 1.0f) * (maxC14 - minC14) / ((float)(compress) - 1.0f))*conversion;
    float c15 = (minC15 + (static_cast<float>(C15[aId]) - 1.0f) * (maxC15 - minC15) / ((float)(compress) - 1.0f))*conversion;
    float c16 = (minC16 + (static_cast<float>(C16[aId]) - 1.0f) * (maxC16 - minC16) / ((float)(compress) - 1.0f))*conversion;

    float c22 = (minC22 + (static_cast<float>(C22[aId]) - 1.0f) * (maxC22 - minC22) / ((float)(compress) - 1.0f))*conversion;
    float c23 = (minC23 + (static_cast<float>(C23[aId]) - 1.0f) * (maxC23 - minC23) / ((float)(compress) - 1.0f))*conversion;
    float c24 = (minC24 + (static_cast<float>(C24[aId]) - 1.0f) * (maxC24 - minC24) / ((float)(compress) - 1.0f))*conversion;
    float c25 = (minC25 + (static_cast<float>(C25[aId]) - 1.0f) * (maxC25 - minC25) / ((float)(compress) - 1.0f))*conversion;
    float c26 = (minC26 + (static_cast<float>(C26[aId]) - 1.0f) * (maxC26 - minC26) / ((float)(compress) - 1.0f))*conversion;

    float c33 = (minC33 + (static_cast<float>(C33[aId]) - 1.0f) * (maxC33 - minC33) / ((float)(compress) - 1.0f))*conversion;
    float c34 = (minC34 + (static_cast<float>(C34[aId]) - 1.0f) * (maxC34 - minC34) / ((float)(compress) - 1.0f))*conversion;
    float c35 = (minC35 + (static_cast<float>(C35[aId]) - 1.0f) * (maxC35 - minC35) / ((float)(compress) - 1.0f))*conversion;
    float c36 = (minC36 + (static_cast<float>(C36[aId]) - 1.0f) * (maxC36 - minC36) / ((float)(compress) - 1.0f))*conversion;

    float c44 = (minC44 + (static_cast<float>(C44[aId]) - 1.0f) * (maxC44 - minC44) / ((float)(compress) - 1.0f))*conversion;
    float c45 = (minC45 + (static_cast<float>(C45[aId]) - 1.0f) * (maxC45 - minC45) / ((float)(compress) - 1.0f))*conversion;
    float c46 = (minC46 + (static_cast<float>(C46[aId]) - 1.0f) * (maxC46 - minC46) / ((float)(compress) - 1.0f))*conversion;

    float c55 = (minC55 + (static_cast<float>(C55[aId]) - 1.0f) * (maxC55 - minC55) / ((float)(compress) - 1.0f))*conversion;
    float c56 = (minC56 + (static_cast<float>(C56[aId]) - 1.0f) * (maxC56 - minC56) / ((float)(compress) - 1.0f))*conversion;

    float c66 = (minC66 + (static_cast<float>(C66[aId]) - 1.0f) * (maxC66 - minC66) / ((float)(compress) - 1.0f))*conversion;

    C[0+0*v] = c11; C[0+1*v] = c12; C[0+2*v] = c13; C[0+3*v] = c14; C[0+4*v] = c15; C[0+5*v] = c16;
    C[1+0*v] = c12; C[1+1*v] = c22; C[1+2*v] = c23; C[1+3*v] = c24; C[1+4*v] = c25; C[1+5*v] = c26;
    C[2+0*v] = c13; C[2+1*v] = c23; C[2+2*v] = c33; C[2+3*v] = c34; C[2+4*v] = c35; C[2+5*v] = c36;
    C[3+0*v] = c14; C[3+1*v] = c24; C[3+2*v] = c34; C[3+3*v] = c44; C[3+4*v] = c45; C[3+5*v] = c46;
    C[4+0*v] = c15; C[4+1*v] = c25; C[4+2*v] = c35; C[4+3*v] = c45; C[4+4*v] = c55; C[4+5*v] = c56;
    C[5+0*v] = c16; C[5+1*v] = c26; C[5+2*v] = c36; C[5+3*v] = c46; C[5+4*v] = c56; C[5+5*v] = c66;

    for (int i = 0; i < v*v; i++) C[i] *= 1.0f / Ro[aId] / Ro[aId];         
}

void Eikonal_ANI::get_christoffel()
{
    for (int index = 0; index < n*n; index++) 
        G[index] = 0.0f; 

    for (int i = 0; i < n; i++) 
    {
        for (int j = 0; j < n; j++) 
        {
            for (int k = 0; k < n; k++) 
            {
                for (int l = 0; l < n; l++) 
                {
                    int I = voigt_map(i, k);
                    int J = voigt_map(j, l);

                    G[i + j*n] += C[I + J*v]*p[k]*p[l];
                }
            }
        }
    }
}

void Eikonal_ANI::get_eigen_values()
{
    float a = -(G[0] + G[4] + G[8]);
    
    float b = G[0]*G[4] + G[4]*G[8] + 
              G[0]*G[8] - G[3]*G[1] - 
              G[6]*G[6] - G[7]*G[5];
    
    float c = -(G[0]*(G[4]*G[8] - G[7]*G[5]) -
                G[3]*(G[1]*G[8] - G[7]*G[6]) +
                G[6]*(G[1]*G[5] - G[4]*G[6]));

    float p = b - (a*a)/3.0f;
    float q = (2.0f*a*a*a)/27.0f - (a*b)/3.0f + c;

    float detG = 0.25f*(q*q) + (p*p*p)/27.0f;

    if (detG > 0) 
    {
        float u = cbrtf(-0.5f*q + sqrtf(detG));
        float v = cbrtf(-0.5f*q - sqrtf(detG));
        
        Gv[0] = u + v - a/3.0f;
    } 
    else if (detG == 0) 
    {       
        float u = cbrt(-0.5f*q);

        Gv[0] = 2.0f*u - a/3.0f;
        Gv[1] =-1.0f*u - a/3.0f;         
    } 
    else  
    {
        float r = sqrtf(-p*p*p/27.0f);
        float phi = acosf(-0.5f*q/r);
        
        r = 2.0f*cbrtf(r);

        Gv[0] = r*cosf(phi/3.0f) - a/3.0f;
        Gv[1] = r*cosf((phi + 2.0f*M_PI)/3.0f) - a/3.0f;  
        Gv[2] = r*cosf((phi + 4.0f*M_PI)/3.0f) - a/3.0f;      
    }
    
    if (Gv[0] < Gv[1]) std::swap(Gv[0],Gv[1]);
    if (Gv[1] < Gv[2]) std::swap(Gv[1],Gv[2]);
    if (Gv[0] < Gv[1]) std::swap(Gv[0],Gv[1]);    
}

int Eikonal_ANI::voigt_map(int i, int j)
{
    if (i == j)
        return i;
    if (((i == 1) && (j == 2)) || ((i == 2) && (j == 1)))
        return 3;
    if (((i == 2) && (j == 0)) || ((i == 0) && (j == 2)))
        return 4;
    if (((i == 0) && (j == 1)) || ((i == 1) && (j == 0)))
        return 5;
    
    return -1;
}
