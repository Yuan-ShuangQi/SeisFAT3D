# include "eikonal_iso.cuh"

void Eikonal_ISO::set_properties()
{
    Vp = new float[nPoints]();

    std::string model_file = catch_parameter("vp_model_file", parameters);

    import_binary_float(model_file, Vp, nPoints);

    for (int index = 0; index < nPoints; index++)
        Vp[index] = 1.0f / Vp[index];

    S = new float[volsize]();

    expand_boundary(Vp, S);
}

void Eikonal_ISO::set_conditions()
{
    modeling_type = "eikonal_iso";
    modeling_name = "Modeling type: Eikonal isotropic time propagation";
}

void Eikonal_ISO::forward_solver()
{
    hipMemcpy(d_S, S, volsize*sizeof(float), hipMemcpyHostToDevice);

    initialization();

    hipMemcpy(d_T, T, volsize*sizeof(float), hipMemcpyHostToDevice);

    fast_sweeping_method();

    hipMemcpy(T, d_T, volsize*sizeof(float), hipMemcpyDeviceToHost);

    compute_seismogram();
}
