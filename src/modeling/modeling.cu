#include "hip/hip_runtime.h"
# include "modeling.cuh"

void Modeling::set_parameters()
{    
    get_GPU_initMem();

    nx = std::stoi(catch_parameter("x_samples", file));
    ny = std::stoi(catch_parameter("y_samples", file));
    nz = std::stoi(catch_parameter("z_samples", file));

    padb = 1;
    nSweeps = 8;
    meshDim = 3;
        
    nxx = nx + 2*padb;
    nyy = ny + 2*padb;
    nzz = nz + 2*padb;

    nPoints = nx*ny*nz;
    volsize = nxx*nyy*nzz;

    dx = std::stof(catch_parameter("x_spacing", file));
    dy = std::stof(catch_parameter("y_spacing", file));
    dz = std::stof(catch_parameter("z_spacing", file));

    export_receiver_output = str2bool(catch_parameter("export_receiver_output", file));
    export_wavefield_output = str2bool(catch_parameter("export_wavefield_output", file));

    receiver_output_folder = catch_parameter("receiver_output_folder", file); 
    wavefield_output_folder = catch_parameter("wavefield_output_folder", file);

    V = new float[nPoints]();
    S = new float[volsize]();
    T = new float[volsize]();

    import_binary_float(catch_parameter("vp_model_file", file), V, nPoints);

    Geometry * types[] = {new Regular(), new Circular()};

    geometry = types[std::stoi(catch_parameter("geometry_type", file))];

    geometry->file = file;

    geometry->set_geometry();

    total_shots = geometry->shots.total;
    total_nodes = geometry->nodes.total;

    check_geometry_overflow();

    wavefield_output_samples = nPoints;
    receiver_output_samples = geometry->nodes.total;

    receiver_output = new float[receiver_output_samples]();
    wavefield_output = new float[wavefield_output_samples]();

    dz2i = 1.0f / (dz*dz);
    dx2i = 1.0f / (dx*dx);
    dy2i = 1.0f / (dy*dy);

    dz2dx2 = dz2i * dx2i;
    dz2dy2 = dz2i * dy2i;
    dx2dy2 = dx2i * dy2i;

    dsum = dz2i + dx2i + dy2i;

    threadsPerBlock = 256;

	totalLevels = (nxx - 1) + (nyy - 1) + (nzz - 1);

    int sgnv[nSweeps][meshDim] = {{1,1,1}, {0,1,1}, {1,1,0}, {0,1,0}, {1,0,1}, {0,0,1}, {1,0,0}, {0,0,0}};
    int sgnt[nSweeps][meshDim] = {{1,1,1}, {-1,1,1}, {1,1,-1}, {-1,1,-1}, {1,-1,1}, {-1,-1,1}, {1,-1,-1}, {-1,-1,-1}};

	int * h_sgnv = new int [nSweeps * meshDim]();
	int * h_sgnt = new int [nSweeps * meshDim](); 

	for (int index = 0; index < nSweeps * meshDim; index++)
	{
		int j = index / nSweeps;
		int i = index % nSweeps;				

		h_sgnv[i + j * nSweeps] = sgnv[i][j];
		h_sgnt[i + j * nSweeps] = sgnt[i][j];
	}

	hipMalloc((void**)&(d_sgnv), nSweeps*meshDim*sizeof(int));
	hipMalloc((void**)&(d_sgnt), nSweeps*meshDim*sizeof(int));

	hipMemcpy(d_sgnv, h_sgnv, nSweeps*meshDim*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sgnt, h_sgnt, nSweeps*meshDim*sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&(d_T), volsize*sizeof(float));
	hipMalloc((void**)&(d_S), volsize*sizeof(float));

    delete[] h_sgnt;
    delete[] h_sgnv;
}

void Modeling::set_slowness()
{
    for (int z = padb; z < nzz - padb; z++)
    {
        for (int y = padb; y < nyy - padb; y++)
        {
            for (int x = padb; x < nxx - padb; x++)
            {
                S[z + x*nzz + y*nxx*nzz] = 1.0f / V[(z - padb) + (x - padb)*nz + (y - padb)*nx*nz];
            }
        }
    }

    for (int z = 0; z < padb; z++)
    {
        for (int y = padb; y < nyy - padb; y++)
        {
            for (int x = padb; x < nxx - padb; x++)
            {
                S[z + x*nzz + y*nxx*nzz] = 1.0f / V[0 + (x - padb)*nz + (y - padb)*nx*nz];
                S[(nzz - z - 1) + x*nzz + y*nxx*nzz] = 1.0f / V[(nz - 1) + (x - padb)*nz + (y - padb)*nx*nz];
            }
        }
    }

    for (int x = 0; x < padb; x++)
    {
        for (int z = 0; z < nzz; z++)
        {
            for (int y = padb; y < nyy - padb; y++)
            {
                S[z + x*nzz + y*nxx*nzz] = S[z + padb*nzz + y*nxx*nzz];
                S[z + (nxx - x - 1)*nzz + y*nxx*nzz] = S[z + (nxx - padb - 1)*nzz + y*nxx*nzz];
            }
        }
    }

    for (int y = 0; y < padb; y++)
    {
        for (int z = 0; z < nzz; z++)
        {
            for (int x = 0; x < nxx; x++)
            {
                S[z + x*nzz + y*nxx*nzz] = S[z + x*nzz + padb*nxx*nzz];
                S[z + x*nzz + (nyy - y - 1)*nxx*nzz] = S[z + x*nzz + (nyy - padb - 1)*nxx*nzz];
            }
        }
    }

	hipMemcpy(d_S, S, volsize*sizeof(float), hipMemcpyHostToDevice);
}

void Modeling::initial_setup()
{
    int sidx = (int)(geometry->shots.x[shot_id] / dx) + padb;
    int sidy = (int)(geometry->shots.y[shot_id] / dy) + padb;
    int sidz = (int)(geometry->shots.z[shot_id] / dz) + padb;

    int sId = sidz + sidx*nzz + sidy*nxx*nzz;

    for (int index = 0; index < volsize; index++) T[index] = 1e6f;

    T[sId] = S[sId] * sqrtf(powf((sidx-padb)*dx - geometry->shots.x[shot_id], 2.0f) + powf((sidy-padb)*dy - geometry->shots.y[shot_id], 2.0f) + powf((sidz-padb)*dz - geometry->shots.z[shot_id], 2.0f));

    T[sId + 1] = S[sId] * sqrtf(powf((sidx-padb)*dx - geometry->shots.x[shot_id], 2.0f) + powf((sidy-padb)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)+1)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId - 1] = S[sId] * sqrtf(powf((sidx-padb)*dx - geometry->shots.x[shot_id], 2.0f) + powf((sidy-padb)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)-1)*dz - geometry->shots.z[shot_id], 2.0f));

    T[sId + nzz] = S[sId] * sqrtf(powf(((sidx-padb)+1)*dx - geometry->shots.x[shot_id], 2.0f) + powf((sidy-padb)*dy - geometry->shots.y[shot_id], 2.0f) + powf((sidz-padb)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId - nzz] = S[sId] * sqrtf(powf(((sidx-padb)-1)*dx - geometry->shots.x[shot_id], 2.0f) + powf((sidy-padb)*dy - geometry->shots.y[shot_id], 2.0f) + powf((sidz-padb)*dz - geometry->shots.z[shot_id], 2.0f));
    
    T[sId + nxx*nzz] = S[sId] * sqrtf(powf((sidx-padb)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)+1)*dy - geometry->shots.y[shot_id], 2.0f) + powf((sidz-padb)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId - nxx*nzz] = S[sId] * sqrtf(powf((sidx-padb)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)-1)*dy - geometry->shots.y[shot_id], 2.0f) + powf((sidz-padb)*dz - geometry->shots.z[shot_id], 2.0f));
    
    T[sId + 1 + nzz] = S[sId] * sqrtf(powf(((sidx-padb)+1)*dx - geometry->shots.x[shot_id], 2.0f) + powf((sidy-padb)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)+1)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId + 1 - nzz] = S[sId] * sqrtf(powf(((sidx-padb)+1)*dx - geometry->shots.x[shot_id], 2.0f) + powf((sidy-padb)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)-1)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId - 1 + nzz] = S[sId] * sqrtf(powf(((sidx-padb)-1)*dx - geometry->shots.x[shot_id], 2.0f) + powf((sidy-padb)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)+1)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId - 1 - nzz] = S[sId] * sqrtf(powf(((sidx-padb)-1)*dx - geometry->shots.x[shot_id], 2.0f) + powf((sidy-padb)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)-1)*dz - geometry->shots.z[shot_id], 2.0f));
    
    T[sId + 1 + nxx*nzz] = S[sId] * sqrtf(powf((sidx-padb)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)+1)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)+1)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId + 1 - nxx*nzz] = S[sId] * sqrtf(powf((sidx-padb)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)-1)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)+1)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId - 1 + nxx*nzz] = S[sId] * sqrtf(powf((sidx-padb)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)+1)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)-1)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId - 1 - nxx*nzz] = S[sId] * sqrtf(powf((sidx-padb)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)-1)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)-1)*dz - geometry->shots.z[shot_id], 2.0f));
    
    T[sId + nzz + nxx*nzz] = S[sId] * sqrtf(powf(((sidx-padb)+1)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)+1)*dy - geometry->shots.y[shot_id], 2.0f) + powf((sidz-padb)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId + nzz - nxx*nzz] = S[sId] * sqrtf(powf(((sidx-padb)+1)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)-1)*dy - geometry->shots.y[shot_id], 2.0f) + powf((sidz-padb)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId - nzz + nxx*nzz] = S[sId] * sqrtf(powf(((sidx-padb)-1)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)+1)*dy - geometry->shots.y[shot_id], 2.0f) + powf((sidz-padb)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId - nzz - nxx*nzz] = S[sId] * sqrtf(powf(((sidx-padb)-1)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)-1)*dy - geometry->shots.y[shot_id], 2.0f) + powf((sidz-padb)*dz - geometry->shots.z[shot_id], 2.0f));
    
    T[sId + 1 + nzz + nxx*nzz] = S[sId] * sqrtf(powf(((sidx-padb)+1)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)+1)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)+1)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId + 1 - nzz + nxx*nzz] = S[sId] * sqrtf(powf(((sidx-padb)-1)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)+1)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)+1)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId + 1 + nzz - nxx*nzz] = S[sId] * sqrtf(powf(((sidx-padb)+1)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)-1)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)+1)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId + 1 - nzz - nxx*nzz] = S[sId] * sqrtf(powf(((sidx-padb)-1)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)-1)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)+1)*dz - geometry->shots.z[shot_id], 2.0f));

    T[sId - 1 + nzz + nxx*nzz] = S[sId] * sqrtf(powf(((sidx-padb)+1)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)+1)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)-1)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId - 1 - nzz + nxx*nzz] = S[sId] * sqrtf(powf(((sidx-padb)-1)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)+1)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)-1)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId - 1 + nzz - nxx*nzz] = S[sId] * sqrtf(powf(((sidx-padb)+1)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)-1)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)-1)*dz - geometry->shots.z[shot_id], 2.0f));
    T[sId - 1 - nzz - nxx*nzz] = S[sId] * sqrtf(powf(((sidx-padb)-1)*dx - geometry->shots.x[shot_id], 2.0f) + powf(((sidy-padb)-1)*dy - geometry->shots.y[shot_id], 2.0f) + powf(((sidz-padb)-1)*dz - geometry->shots.z[shot_id], 2.0f));

    t0 = T[sId];

	hipMemcpy(d_T, T, volsize*sizeof(float), hipMemcpyHostToDevice);
}

void Modeling::forward_solver()
{
    for (int sweep = 0; sweep < nSweeps; sweep++)
	{ 
		int start = (sweep == 3 || sweep == 5 || sweep == 6 || sweep == 7) ? totalLevels : meshDim;
		int end = (start == meshDim) ? totalLevels + 1 : meshDim - 1;
		int incr = (start == meshDim) ? true : false;

		int xSweepOff = (sweep == 3 || sweep == 4) ? nxx : 0;
		int ySweepOff = (sweep == 2 || sweep == 5) ? nyy : 0;
		int zSweepOff = (sweep == 1 || sweep == 6) ? nzz : 0;
		
		for (int level = start; level != end; level = (incr) ? level + 1 : level - 1)
		{			
			int xs = max(1, level - (nyy + nzz));	
			int ys = max(1, level - (nxx + nzz));

			int xe = min(nxx, level - (meshDim - 1));
			int ye = min(nyy, level - (meshDim - 1));	
		
			int xr = xe - xs + 1;
			int yr = ye - ys + 1;

			int nThreads = xr * yr;
				
			dim3 bs(16, 16, 1);

			if (nThreads < threadsPerBlock) { bs.x = xr; bs.y = yr; } 

			dim3 gs(iDivUp(xr, bs.x), iDivUp(yr , bs.y), 1);
			
            int sgni = sweep + 0*nSweeps;
            int sgnj = sweep + 1*nSweeps;
            int sgnk = sweep + 2*nSweeps;

			fast_sweeping_kernel<<<gs,bs>>>(d_S, d_T, d_sgnt, d_sgnv, sgni, sgnj, sgnk, level, xs, ys, 
                                            xSweepOff, ySweepOff, zSweepOff, nxx, nyy, nzz, dx, dy, dz, 
                                            dx2i, dy2i, dz2i, dz2dx2, dz2dy2, dx2dy2, dsum);
			
            hipDeviceSynchronize();
		}
	}

    hipMemcpy(T, d_T, volsize*sizeof(float), hipMemcpyDeviceToHost);
}

void Modeling::build_outputs()
{
    get_travelTimes();
    get_firstArrivals();
}

void Modeling::get_travelTimes()
{
    for (int index = 0; index < nPoints; index++)
    {
        int y = (int) (index / (nx*nz));         
        int x = (int) (index - y*nx*nz) / nz;    
        int z = (int) (index - x*nz - y*nx*nz);  

        wavefield_output[z + x*nz + y*nx*nz] = T[(z + padb) + (x + padb)*nzz + (y + padb)*nxx*nzz];
    }

    wavefield_output_file = wavefield_output_folder + "time_volume_" + std::to_string(nz) + "x" + std::to_string(nx) + "x" + std::to_string(ny) + "_shot_" + std::to_string(shot_id+1) + ".bin";
}

void Modeling::get_firstArrivals()
{
    for (int r = 0; r < total_nodes; r++)
    {
        float x = geometry->nodes.x[r];
        float y = geometry->nodes.y[r];
        float z = geometry->nodes.z[r];

        float x0 = floorf(x / dx) * dx;
        float y0 = floorf(y / dy) * dy;
        float z0 = floorf(z / dz) * dz;

        float x1 = floorf(x / dx) * dx + dx;
        float y1 = floorf(y / dy) * dy + dy;
        float z1 = floorf(z / dz) * dz + dz;

        int id = ((int)(z / dz)) + ((int)(x / dx))*nz + ((int)(y / dy))*nx*nz;

        float c000 = wavefield_output[id];
        float c001 = wavefield_output[id + 1];
        float c100 = wavefield_output[id + nz]; 
        float c101 = wavefield_output[id + 1 + nz]; 
        float c010 = wavefield_output[id + nx*nz]; 
        float c011 = wavefield_output[id + 1 + nx*nz]; 
        float c110 = wavefield_output[id + nz + nx*nz]; 
        float c111 = wavefield_output[id + 1 + nz + nx*nz];

        float xd = (x - x0) / (x1 - x0);
        float yd = (y - y0) / (y1 - y0);
        float zd = (z - z0) / (z1 - z0);

        float c00 = c000*(1 - xd) + c100*xd;    
        float c01 = c001*(1 - xd) + c101*xd;    
        float c10 = c010*(1 - xd) + c110*xd;    
        float c11 = c011*(1 - xd) + c111*xd;    

        float c0 = c00*(1 - yd) + c10*yd;
        float c1 = c01*(1 - yd) + c11*yd;

        receiver_output[r] = c0*(1 - zd) + c1*zd;
    }

    receiver_output_file = receiver_output_folder + "data_" + std::to_string(geometry->nodes.total) + "_shot_" + std::to_string(shot_id+1) + ".bin";
}

void Modeling::free_space()
{
    hipFree(d_T);
    hipFree(d_S);

    hipFree(d_sgnt);
    hipFree(d_sgnv);

    delete[] T;
    delete[] S;
    delete[] V;
}

void Modeling::info_message()
{
    get_RAM_usage();
    get_GPU_usage();

    auto clear = system("clear");
        
    std::cout<<"Model dimensions (z = "<<(nz-1)*dz<<", x = "<<(nx-1)*dx<<", y = "<<(ny-1)*dy<<") m\n\n";

    std::cout<<"Shot "<<shot_id+1<<" of "<<geometry->shots.total;

    std::cout<<" at position (z = "<<geometry->shots.z[shot_id]<<", x = " 
                                   <<geometry->shots.x[shot_id]<<", y = " 
                                   <<geometry->shots.y[shot_id]<<") m\n\n";

    std::cout<<"Memory usage: \n";
    std::cout<<"RAM = "<<RAM<<" Mb\n";
    std::cout<<"GPU = "<<vRAM<<" Mb\n\n";
}

void Modeling::set_runtime()
{
    ti = std::chrono::system_clock::now();
}

void Modeling::get_runtime()
{
    tf = std::chrono::system_clock::now();

    std::chrono::duration<double> elapsed_seconds = tf - ti;

    std::cout<<"\nRun time: "<<elapsed_seconds.count()<<" s."<<std::endl;
}

void Modeling::get_RAM_usage()
{
    struct rusage usage;
    getrusage(RUSAGE_SELF, &usage);
    RAM = (int) (usage.ru_maxrss / 1024);
}

void Modeling::get_GPU_initMem()
{
	size_t freeMem, totalMem;
	hipMemGetInfo(&freeMem, &totalMem);
    ivRAM = (int) ((totalMem - freeMem) / (1024 * 1024));
}

void Modeling::get_GPU_usage()
{
	size_t freeMem, totalMem;
	hipMemGetInfo(&freeMem, &totalMem);
    vRAM = (int) ((totalMem - freeMem) / (1024 * 1024));
    vRAM -= ivRAM;
}

void Modeling::export_outputs()
{
    if (export_receiver_output) export_binary_float(receiver_output_file, receiver_output, receiver_output_samples);
    if (export_wavefield_output) export_binary_float(wavefield_output_file, wavefield_output, wavefield_output_samples);
}

int Modeling::iDivUp(int a, int b) 
{ 
    return ( (a % b) != 0 ) ? (a / b + 1) : (a / b); 
}

void Modeling::check_geometry_overflow()
{
    for (int shot = 0; shot < total_shots; shot++)
    {
        if ((geometry->shots.x[shot] < 0) && (geometry->shots.x[shot] > (nx-1)*dx) && 
            (geometry->shots.y[shot] < 0) && (geometry->shots.y[shot] > (ny-1)*dy) &&
            (geometry->shots.z[shot] < 0) && (geometry->shots.z[shot] > (nz-1)*dz))       
        throw std::invalid_argument("\033[31mError: shots geometry overflow!\033[0;0m");
    }

    for (int node = 0; node < total_nodes; node++)
    {
        if ((geometry->nodes.x[node] < 0) && (geometry->nodes.x[node] > (nx-1)*dx) && 
            (geometry->nodes.y[node] < 0) && (geometry->nodes.y[node] > (ny-1)*dy) &&
            (geometry->nodes.z[node] < 0) && (geometry->nodes.z[node] > (nz-1)*dz))       
        throw std::invalid_argument("\033[31mError: nodes geometry overflow!\033[0;0m");
    }
}

__global__ void fast_sweeping_kernel(float * S, float * T, int * sgnt, int * sgnv, int sgni, int sgnj, int sgnk, 
                                     int level, int xOffset, int yOffset, int xSweepOffset, int ySweepOffset, int zSweepOffset, 
                                     int nxx, int nyy, int nzz, float dx, float dy, float dz, float dx2i, float dy2i, float dz2i, 
                                     float dz2dx2, float dz2dy2, float dx2dy2, float dsum)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x) + xOffset;
	int y = (blockIdx.y * blockDim.y + threadIdx.y) + yOffset;

    float ta, tb, tc, t1, t2, t3, Sref;
    float t1D1, t1D2, t1D3, t1D, t2D1, t2D2, t2D3, t2D, t3D;

	if ((x <= nxx) && (y <= nyy)) 
	{
		int z = level - (x + y);
		
		if ((z > 0) && (z <= nzz))	
		{
			int i = abs(z - zSweepOffset);
			int j = abs(x - xSweepOffset);
			int k = abs(y - ySweepOffset);

			if ((i > 0) && (i < nzz-1) && (j > 0) && (j < nxx-1) && (k > 0) && (k < nyy-1))
			{		
				int i1 = i - sgnv[sgni];
				int j1 = j - sgnv[sgnj];
				int k1 = k - sgnv[sgnk];

				int ijk = i + j*nzz + k*nxx*nzz;
				
				float tv = T[(i - sgnt[sgni]) + j*nzz + k*nxx*nzz];
				float te = T[i + (j - sgnt[sgnj])*nzz + k*nxx*nzz];
				float tn = T[i + j*nzz + (k - sgnt[sgnk])*nxx*nzz];

				float tev = T[(i - sgnt[sgni]) + (j - sgnt[sgnj])*nzz + k*nxx*nzz];
				float ten = T[i + (j - sgnt[sgnj])*nzz + (k - sgnt[sgnk])*nxx*nzz];
				float tnv = T[(i - sgnt[sgni]) + j*nzz + (k - sgnt[sgnk])*nxx*nzz];
				
				float tnve = T[(i - sgnt[sgni]) + (j - sgnt[sgnj])*nzz + (k - sgnt[sgnk])*nxx*nzz];

				t1D1 = tv + dz * min(S[i1 + max(j-1,1)*nzz   + max(k-1,1)*nxx*nzz], 
								 min(S[i1 + max(j-1,1)*nzz   + min(k,nyy-1)*nxx*nzz], 
								 min(S[i1 + min(j,nxx-1)*nzz + max(k-1,1)*nxx*nzz],
									 S[i1 + min(j,nxx-1)*nzz + min(k,nyy-1)*nxx*nzz])));                                     

				t1D2 = te + dx * min(S[max(i-1,1)   + j1*nzz + max(k-1,1)*nxx*nzz], 
								 min(S[min(i,nzz-1) + j1*nzz + max(k-1,1)*nxx*nzz],
								 min(S[max(i-1,1)   + j1*nzz + min(k,nyy-1)*nxx*nzz], 
									 S[min(i,nzz-1) + j1*nzz + min(k,nyy-1)*nxx*nzz])));                    

				t1D3 = tn + dy * min(S[max(i-1,1)   + max(j-1,1)*nzz   + k1*nxx*nzz], 
								 min(S[max(i-1,1)   + min(j,nxx-1)*nzz + k1*nxx*nzz],
								 min(S[min(i,nzz-1) + max(j-1,1)*nzz   + k1*nxx*nzz], 
									 S[min(i,nzz-1) + min(j,nxx-1)*nzz + k1*nxx*nzz])));

				t1D = min(t1D1, min(t1D2, t1D3));

                //------------------- 2D operators - 4 points operator ---------------------------------------------------------------------------------------------------
                t2D1 = 1e6; t2D2 = 1e6; t2D3 = 1e6;

                // XZ plane ----------------------------------------------------------------------------------------------------------------------------------------------
                Sref = min(S[i1 + j1*nzz + max(k-1,1)*nxx*nzz], S[i1 + j1*nzz + min(k, nyy-1)*nxx*nzz]);
                
                if ((tv < te + dx*Sref) && (te < tv + dz*Sref))
                {
                    ta = tev + te - tv;
                    tb = tev - te + tv;

                    t2D1 = ((tb*dz2i + ta*dx2i) + sqrtf(4.0f*Sref*Sref*(dz2i + dx2i) - dz2i*dx2i*(ta - tb)*(ta - tb))) / (dz2i + dx2i);
                }

                // YZ plane -------------------------------------------------------------------------------------------------------------------------------------------------------------
                Sref = min(S[i1 + max(j-1,1)*nzz + k1*nxx*nzz], S[i1 + min(j,nxx-1)*nzz + k1*nxx*nzz]);

                if((tv < tn + dy*Sref) && (tn < tv + dz*Sref))
                {
                    ta = tv - tn + tnv;
                    tb = tn - tv + tnv;
                    
                    t2D2 = ((ta*dz2i + tb*dy2i) + sqrtf(4.0f*Sref*Sref*(dz2i + dy2i) - dz2i*dy2i*(ta - tb)*(ta - tb))) / (dz2i + dy2i); 
                }

                // XY plane -------------------------------------------------------------------------------------------------------------------------------------------------------------
                Sref = min(S[max(i-1,1) + j1*nzz + k1*nxx*nzz],S[min(i,nzz-1) + j1*nzz + k1*nxx*nzz]);

                if((te < tn + dy*Sref) && (tn < te + dx*Sref))
                {
                    ta = te - tn + ten;
                    tb = tn - te + ten;

                    t2D3 = ((ta*dx2i + tb*dy2i) + sqrtf(4.0f*Sref*Sref*(dx2i + dy2i) - dx2i*dy2i*(ta - tb)*(ta - tb))) / (dx2i + dy2i);
                }

                t2D = min(t2D1, min(t2D2, t2D3));

                //------------------- 3D operators - 8 point operator ---------------------------------------------------------------------------------------------------
                t3D = 1e6;

                Sref = S[i1 + j1*nzz + k1*nxx*nzz];

                ta = te - 0.5f*tn + 0.5f*ten - 0.5f*tv + 0.5f*tev - tnv + tnve;
                tb = tv - 0.5f*tn + 0.5f*tnv - 0.5f*te + 0.5f*tev - ten + tnve;
                tc = tn - 0.5f*te + 0.5f*ten - 0.5f*tv + 0.5f*tnv - tev + tnve;

                if (min(t1D, t2D) > max(tv, max(te, tn)))
                {
                    t2 = 9.0f*Sref*Sref*dsum; 
                    
                    t3 = dz2dx2*(ta - tb)*(ta - tb) + dz2dy2*(tb - tc)*(tb - tc) + dx2dy2*(ta - tc)*(ta - tc);
                    
                    if (t2 >= t3)
                    {
                        t1 = tb*dz2i + ta*dx2i + tc*dy2i;        
                        
                        t3D = (t1 + sqrtf(t2 - t3)) / dsum;
                    }
                }

				T[ijk] = min(T[ijk], min(t1D, min(t2D, t3D)));
            }
        }
    }
}
