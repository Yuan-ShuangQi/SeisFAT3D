#include "hip/hip_runtime.h"
# include "podvin_and_lecomte.cuh"

void Podvin_and_Lecomte::set_parameters()
{
    general_modeling_parameters();

    set_acquisition_geometry();

    set_velocity_model();
    
    set_boundaries();
    set_model_boundaries();
    
    set_slowness_model();
    set_outputs();

    set_modeling_volumes();
}

void Podvin_and_Lecomte::set_boundaries()
{
    nbxl = 1; nbxr = 1;
    nbyl = 1; nbyr = 1;
    nbzu = 1; nbzd = 1;
}

void Podvin_and_Lecomte::set_modeling_volumes()
{
    modeling_method = std::string("pod");

    T = new float[volsize](); 
    K = new float[volsize]();

    check_spatial_spacing();

    hipMalloc((void**)&(d_S), volsize*sizeof(float));     
    hipMalloc((void**)&(d_T), volsize*sizeof(float));     
    hipMalloc((void**)&(d_K), volsize*sizeof(float));     
    hipMalloc((void**)&(d_nK), volsize*sizeof(float));     
    hipMalloc((void**)&(d_nT), volsize*sizeof(float));   
}

void Podvin_and_Lecomte::check_spatial_spacing()
{
    if ((dx != dy) || (dx != dz))
        throw std::invalid_argument("\033[31mError: For Podvin and Lecomte method the model spacing must to be fixed (dx = dy = dz).\033[0;0m");
}

void Podvin_and_Lecomte::info_message()
{
    general_modeling_message();

    std::cout<<"[0] - Podvin & Lecomte (1991)\n\n"; 
}

void Podvin_and_Lecomte::initial_setup()
{
    nit = 0;
    
    int sidx = (int)(geometry->shots.x[shot_id] / dx) + nbxl;
    int sidy = (int)(geometry->shots.y[shot_id] / dy) + nbyl;
    int sidz = (int)(geometry->shots.z[shot_id] / dz) + nbzu;

    source_id = sidz + sidx*nzz + sidy*nxx*nzz;

    t0 = S[source_id] * sqrtf(powf((float)((sidx-nbxl)*dx) - geometry->shots.x[shot_id], 2.0f) +
                              powf((float)((sidy-nbyl)*dy) - geometry->shots.y[shot_id], 2.0f) +
                              powf((float)((sidz-nbzu)*dz) - geometry->shots.z[shot_id], 2.0f));

    for (int index = 0; index < volsize; index++)
    {    
        T[index] = 1e6f;
        K[index] = 0.0f;
    }

    T[source_id] = S[source_id] * sqrtf(powf((sidx-nbxl)*dx - geometry->shots.x[shot_id], 2.0f) + powf((sidy-nbyl)*dy - geometry->shots.y[shot_id], 2.0f) + powf((sidz-nbzu)*dz - geometry->shots.z[shot_id], 2.0f));

    int aux = (int)sqrtf(powf(sidx, 2.0f) + powf(sidy,2.0f) + powf(sidz,2.0f)); 
    if (aux > nit) nit = aux;

    aux = (int)sqrtf(powf(nxx - sidx,2.0f) + powf(sidy,2.0f) + powf(sidz,2.0f));
    if (aux > nit) nit = aux;

    aux = (int)sqrtf(powf(sidx,2.0f) + powf(nyy - sidy,2.0f) + powf(sidz,2.0f)); 
    if (aux > nit) nit = aux;

    aux = (int)sqrtf(powf(sidx,2.0f) + powf(sidy,2.0f) + powf(nzz - sidz,2.0f)); 
    if (aux > nit) nit = aux;

    aux = (int)sqrtf(powf(sidx,2.0f) + powf(nyy - sidy,2.0f) + powf(nzz - sidz,2.0f));
    if (aux > nit) nit = aux;

    aux = (int)sqrtf(powf(nxx - sidx,2.0f) + powf(sidy,2.0f) + powf(nzz - sidz,2.0f));
    if (aux > nit) nit = aux;

    aux = (int)sqrtf(powf(nxx - sidx,2.0f) + powf(nyy - sidy,2.0f) + powf(sidz,2.0f));
    if (aux > nit) nit = aux;

    aux = (int)sqrtf(powf(nxx - sidx,2.0f) + powf(nyy - sidy,2.0f) + powf(nzz - sidz,2.0f));
    if (aux > nit) nit = aux;

    K[source_id - 1] = 1.0f;
    K[source_id + 1] = 1.0f;
    K[source_id - nzz] = 1.0f;
    K[source_id + nzz] = 1.0f;
    K[source_id - nxx*nzz] = 1.0f;
    K[source_id + nxx*nzz] = 1.0f;
    K[source_id + 1 - nzz] = 1.0f;
    K[source_id - 1 - nzz] = 1.0f;
    K[source_id + 1 + nzz] = 1.0f;
    K[source_id - 1 + nzz] = 1.0f;
    K[source_id + 1 + nxx*nzz] = 1.0f;
    K[source_id + 1 - nxx*nzz] = 1.0f;
    K[source_id - 1 + nxx*nzz] = 1.0f;
    K[source_id - 1 - nxx*nzz] = 1.0f;
    K[source_id - nzz - nxx*nzz] = 1.0f;
    K[source_id - nzz + nxx*nzz] = 1.0f;
    K[source_id + nzz - nxx*nzz] = 1.0f;
    K[source_id + nzz + nxx*nzz] = 1.0f;
    K[source_id + 1 + nzz + nxx*nzz] = 1.0f;
    K[source_id + 1 + nzz - nxx*nzz] = 1.0f;
    K[source_id + 1 - nzz + nxx*nzz] = 1.0f;
    K[source_id + 1 - nzz - nxx*nzz] = 1.0f;
    K[source_id - 1 - nzz - nxx*nzz] = 1.0f;
    K[source_id - 1 - nzz + nxx*nzz] = 1.0f;
    K[source_id - 1 + nzz - nxx*nzz] = 1.0f;
    K[source_id - 1 + nzz + nxx*nzz] = 1.0f;
}

void Podvin_and_Lecomte::forward_solver()
{
    hipMemcpy(d_K, K, volsize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_T, T, volsize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_nT, T, volsize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_S, S, volsize*sizeof(float), hipMemcpyHostToDevice);
    
    int blocksPerGrid = volsize / threadsPerBlock;

    float dh = dx;

    for (int it = 0; it < nit; it++)
    {
        fdm_operators<<<blocksPerGrid,threadsPerBlock>>>(d_S, d_T, d_K, d_nT, dh, nxx, nyy, nzz);        
        hipDeviceSynchronize();

        hipMemset(d_nK, 0.0f, volsize*sizeof(float));

        expanding_box<<<blocksPerGrid,threadsPerBlock>>>(d_K, d_nK, nxx, nyy, nzz);
        hipDeviceSynchronize();

        update_volume<<<blocksPerGrid,threadsPerBlock>>>(d_T, d_nT, d_K, d_nK, volsize);
        hipDeviceSynchronize();
    }

    hipMemcpy(T, d_T, volsize*sizeof(float), hipMemcpyDeviceToHost);
}

void Podvin_and_Lecomte::free_space()
{
    delete[] K;

    hipFree(d_K);
    hipFree(d_S);
    hipFree(d_T);
    hipFree(d_nK);
    hipFree(d_nT);
}

__global__ void fdm_operators(float * S, float * T, float * K, float * nT, float h, int nxx, int nyy, int nzz)
{
    float sqrt2 = sqrtf(2.0f);
    float sqrt3 = sqrtf(3.0f);

    float tolerance = 1e-6f;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if ((K[index] == 1.0f) && (index < nxx*nyy*nzz))
    {
        int k = (int) (index / (nxx*nzz));         // y direction
        int j = (int) (index - k*nxx*nzz) / nzz;   // x direction
        int i = (int) (index - j*nzz - k*nxx*nzz); // z direction

        if ((i > 0) && (i < nzz-1) && (j > 0) && (j < nxx-1) && (k > 0) && (k < nyy-1))
        {
            float lowest = T[index];
            float Tijk, T1, T2, Sref, M, N, P, Q, hs2;    

            /* 1D operator head wave: i,j-1,k -> i,j,k (x direction) */
            Tijk = T[index - nzz] + h * min(S[index - nzz], 
                                        min(S[index - 1 - nzz], 
                                        min(S[index - nzz - nxx*nzz], S[index - 1 - nzz - nxx*nzz]))); 
            
            if (Tijk < lowest) lowest = Tijk;

            /* 1D operator head wave: i,j+1,k -> i,j,k (x direction) */
            Tijk = T[index + nzz] + h * min(S[index], 
                                        min(S[index - 1], 
                                        min(S[index - nxx*nzz], S[index - 1 - nxx*nzz])));
            
            if (Tijk < lowest) lowest = Tijk;

            /* 1D operator head wave: i,j,k-1 -> i,j,k (y direction) */
            Tijk = T[index - nxx*nzz] + h * min(S[index - nxx*nzz], 
                                            min(S[index - nzz - nxx*nzz], 
                                            min(S[index - 1 - nxx*nzz], S[index - 1 - nzz - nxx*nzz]))); 
            
            if (Tijk < lowest) lowest = Tijk;

            /* 1D operator head wave: i,j,k+1 -> i,j,k (y direction) */
            Tijk = T[index + nxx*nzz] + h * min(S[index],
                                            min(S[index - 1], 
                                            min(S[index - nzz], S[index - 1 - nzz]))); 
            
            if (Tijk < lowest) lowest = Tijk;

            /* 1D operator head wave: i-1,j,k -> i,j,k (z direction) */
            Tijk = T[index - 1] + h * min(S[index - 1], 
                                      min(S[index - 1 - nzz], 
                                      min(S[index - 1 - nxx*nzz], S[index - 1 - nzz - nxx*nzz]))); 
            
            if (Tijk < lowest) lowest = Tijk;

            /* 1D operator head wave: i+1,j,k -> i,j,k (z direction) */
            Tijk = T[index + 1] + h * min(S[index], 
                                      min(S[index - nzz], 
                                      min(S[index - nxx*nzz], S[index - nzz - nxx*nzz]))); 
            
            if (Tijk < lowest) lowest = Tijk;
                
            /* 1D operator diffraction XZ plane */
            
            // i-1,j-1,k -> i,j,k
            Tijk = T[index - 1 - nzz] + h*sqrt2*min(S[index - 1 - nzz], S[index - 1 - nzz - nxx*nzz]); 
            if (Tijk < lowest) lowest = Tijk;

            // i-1,j+1,k -> i,j,k
            Tijk = T[index - 1 + nzz] + h*sqrt2*min(S[index - 1], S[index - 1 - nxx*nzz]); 
            if (Tijk < lowest) lowest = Tijk;
            
            // i+1,j-1,k -> i,j,k
            Tijk = T[index + 1 - nzz] + h*sqrt2*min(S[index - nzz], S[index - nzz - nxx*nzz]); 
            if (Tijk < lowest) lowest = Tijk;
            
            // i+1,j+1,k -> i,j,k
            Tijk = T[index + 1 + nzz] + h*sqrt2*min(S[index], S[index - nxx*nzz]); 
            if (Tijk < lowest) lowest = Tijk;

            /* 1D operator diffraction YZ plane */

            // i-1,j,k-1 -> i,j,k
            Tijk = T[index - 1 - nxx*nzz] + h*sqrt2*min(S[index - 1 - nxx*nzz], S[index - 1 - nzz - nxx*nzz]); 
            if (Tijk < lowest) lowest = Tijk;

            // i-1,j,k+1 -> i,j,k
            Tijk = T[index - 1 + nxx*nzz] + h*sqrt2*min(S[index - 1], S[index - 1 - nzz]); 
            if (Tijk < lowest) lowest = Tijk;
            
            // i+1,j,k-1 -> i,j,k
            Tijk = T[index + 1 - nxx*nzz] + h*sqrt2*min(S[index - nxx*nzz], S[index - nzz - nxx*nzz]); 
            if (Tijk < lowest) lowest = Tijk;
            
            // i+1,j,k+1 -> i,j,k
            Tijk = T[index + 1 + nxx*nzz] + h*sqrt2*min(S[index], S[index - nzz]); 
            if (Tijk < lowest) lowest = Tijk;

            /* 1D operator diffraction XY plane */
            
            // i,j-1,k-1 -> i,j,k
            Tijk = T[index - nzz - nxx*nzz] + h*sqrt2*min(S[index - nzz - nxx*nzz], S[index - 1 - nzz - nxx*nzz]); 
            if (Tijk < lowest) lowest = Tijk;

            // i,j-1,k+1 -> i,j,k
            Tijk = T[index - nzz + nxx*nzz] + h*sqrt2*min(S[index - nzz], S[index - 1 - nzz]); 
            if (Tijk < lowest) lowest = Tijk;

            // i,j+1,k-1 -> i,j,k
            Tijk = T[index + nzz - nxx*nzz] + h*sqrt2*min(S[index - nxx*nzz], S[index - 1 - nxx*nzz]); 
            if (Tijk < lowest) lowest = Tijk;

            // i,j+1,k+1 -> i,j,k
            Tijk = T[index + nzz + nxx*nzz] + h*sqrt2*min(S[index], S[index - 1]); 
            if (Tijk < lowest) lowest = Tijk;

            /* 1D operator corner diffractions */

            // i-1,j-1,k-1 -> i,j,k
            Tijk = T[index - 1 - nzz - nxx*nzz] + h*sqrt3*S[index - 1 - nzz - nxx*nzz]; 
            if (Tijk < lowest) lowest = Tijk;

            // i-1,j-1,k+1 -> i,j,k
            Tijk = T[index - 1 - nzz + nxx*nzz] + h*sqrt3*S[index - 1 - nzz]; 
            if (Tijk < lowest) lowest = Tijk;

            // i+1,j-1,k-1 -> i,j,k
            Tijk = T[index + 1 - nzz - nxx*nzz] + h*sqrt3*S[index - nzz - nxx*nzz]; 
            if (Tijk < lowest) lowest = Tijk;

            // i+1,j-1,k+1 -> i,j,k
            Tijk = T[index + 1 - nzz + nxx*nzz] + h*sqrt3*S[index - nzz]; 
            if (Tijk < lowest) lowest = Tijk;

            // i-1,j+1,k-1 -> i,j,k
            Tijk = T[index - 1 + nzz - nxx*nzz] + h*sqrt3*S[index - 1 - nxx*nzz]; 
            if (Tijk < lowest) lowest = Tijk;

            // i-1,j+1,k+1 -> i,j,k
            Tijk = T[index - 1 + nzz + nxx*nzz] + h*sqrt3*S[index - 1]; 
            if (Tijk < lowest) lowest = Tijk;

            // i+1,j+1,k-1 -> i,j,k
            Tijk = T[index + 1 + nzz - nxx*nzz] + h*sqrt3*S[index - nxx*nzz]; 
            if (Tijk < lowest) lowest = Tijk;

            // i+1,j+1,k+1 -> i,j,k
            Tijk = T[index + 1 + nzz + nxx*nzz] + h*sqrt3*S[index]; 
            if (Tijk < lowest) lowest = Tijk;

            /* 2D operator XZ plane: First Quadrant*/

            Sref = min(S[index - 1 - nzz], S[index - 1 - nzz - nxx*nzz]);

            // i,j-1,k - i-1,j-1,k -> i,j,k
            T1 = T[index - nzz];
            T2 = T[index - 1 - nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i-1,j,k - i-1,j-1,k -> i,j,k
            T1 = T[index - 1];
            T2 = T[index - 1 - nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator XZ plane: Second Quadrant*/                        

            Sref = min(S[index - nzz], S[index - nzz - nxx*nzz]);

            // i,j-1,k - i+1,j-1,k -> i,j,k
            T1 = T[index - nzz];
            T2 = T[index + 1 - nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i+1,j,k - i+1,j-1,k -> i,j,k
            T1 = T[index + 1];
            T2 = T[index + 1 - nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator XZ plane: Third Quadrant*/                        

            Sref = min(S[index], S[index - nxx*nzz]);

            // i+1,j,k - i+1,j+1,k -> i,j,k
            T1 = T[index + 1];
            T2 = T[index + 1 + nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i,j+1,k - i+1,j+1,k -> i,j,k
            T1 = T[index + nzz];
            T2 = T[index + 1 + nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator XZ plane: Fourth Quadrant*/                        

            Sref = min(S[index - 1], S[index - 1 - nxx*nzz]);

            // i,j+1,k - i-1,j+1,k -> i,j,k
            T1 = T[index + nzz];
            T2 = T[index - 1 + nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i-1,j,k - i-1,j+1,k -> i,j,k
            T1 = T[index - 1];
            T2 = T[index - 1 + nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator YZ plane: First Quadrant */                        

            Sref = min(S[index - 1 - nxx*nzz], S[index - 1 - nzz - nxx*nzz]);

            // i,j,k-1 - i-1,j,k-1 -> i,j,k
            T1 = T[index - nxx*nzz];
            T2 = T[index - 1 - nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i-1,j,k - i-1,j,k-1 -> i,j,k
            T1 = T[index - 1];
            T2 = T[index - 1 - nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator YZ plane: Second Quadrant */                        

            Sref = min(S[index - nxx*nzz], S[index - nzz - nxx*nzz]);

            // i,j,k-1 - i+1,j,k-1 -> i,j,k
            T1 = T[index - nxx*nzz];
            T2 = T[index + 1 - nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i+1,j,k - i+1,j,k-1 -> i,j,k
            T1 = T[index + 1];
            T2 = T[index + 1 - nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator YZ plane: Third Quadrant*/                        

            Sref = min(S[index], S[index - nzz]);

            // i+1,j,k - i+1,j,k+1 -> i,j,k
            T1 = T[index + 1];
            T2 = T[index + 1 + nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i,j,k+1 - i+1,j,k+1 -> i,j,k
            T1 = T[index + nxx*nzz];
            T2 = T[index + 1 + nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator YZ plane: Fourth Quadrant*/                        

            Sref = min(S[index - 1], S[index - 1 - nzz]);

            // i,j,k+1 - i-1,j,k+1 -> i,j,k
            T1 = T[index + nxx*nzz];
            T2 = T[index - 1 + nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i-1,j,k - i-1,j,k+1 -> i,j,k
            T1 = T[index - 1];
            T2 = T[index - 1 + nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator XY plane: First Quadrant*/                        

            Sref = min(S[index - nzz - nxx*nzz], S[index - 1 - nzz - nxx*nzz]);

            // i,j-1,k - i,j-1,k-1 -> i,j,k
            T1 = T[index - nzz];
            T2 = T[index - nzz - nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i,j,k-1 - i,j-1,k-1 -> i,j,k
            T1 = T[index - nxx*nzz];
            T2 = T[index - nzz - nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator XY plane: Second Quadrant*/                        

            Sref = min(S[index - nzz], S[index - 1 - nzz]);

            // i,j-1,k - i,j-1,k+1 -> i,j,k
            T1 = T[index - nzz];
            T2 = T[index - nzz + nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i,j,k+1 - i,j-1,k+1 -> i,j,k
            T1 = T[index + nxx*nzz];
            T2 = T[index - nzz + nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator XY plane: Third Quadrant*/                        

            Sref = min(S[index], S[index - 1]);

            // i,j,k+1 - i,j+1,k+1 -> i,j,k
            T1 = T[index + nxx*nzz];
            T2 = T[index + nzz + nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i,j+1,k - i,j+1,k+1 -> i,j,k
            T1 = T[index + nzz];
            T2 = T[index + nzz + nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 2D operator XY plane: Fourth Quadrant*/                        

            Sref = min(S[index - nxx*nzz], S[index - 1 - nxx*nzz]);

            // i,j+1,k - i,j+1,k-1 -> i,j,k
            T1 = T[index + nzz];
            T2 = T[index + nzz - nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            // i,j,k-1 - i,j+1,k-1 -> i,j,k
            T1 = T[index - nxx*nzz];
            T2 = T[index + nzz - nxx*nzz];
            if ((T1 - T2) > 0.0f)
            {
                if ((T1 - T2) < h*Sref/sqrt2)
                {
                    Tijk = T1 + sqrtf(h*h*Sref*Sref - (T1 - T2)*(T1 - T2));
                    if (Tijk < lowest) lowest = Tijk;
                }
            }

            /* 3D operator - First octant: XY plane */

            Sref = S[index - 1 - nzz - nxx*nzz];
            hs2 = h*h*Sref*Sref;

            M = T[index - 1 - nzz - nxx*nzz]; /* i-1,j-1,k-1 */  
            N = T[index - 1 - nzz];           /* i-1,j-1, k  */  
            P = T[index - 1 - nxx*nzz];       /* i-1, j ,k-1 */
            Q = T[index - 1];                 /* i-1, j , k  */

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - First octant: YZ plane */

            M = T[index - 1 - nzz - nxx*nzz]; /* i-1,j-1,k-1 */   
            N = T[index - 1 - nzz];           /* i-1,j-1, k  */   
            P = T[index - nzz - nxx*nzz];     /*  i ,j-1,k-1 */   
            Q = T[index - nzz];               /*  i ,j-1, k  */   

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - First octant: XZ plane */

            M = T[index - 1 - nzz - nxx*nzz]; /* i-1,j-1,k-1 */   
            N = T[index - nzz - nxx*nzz];     /*  i ,j-1,k-1 */         
            P = T[index - 1 - nxx*nzz];       /* i-1, j ,k-1 */ 
            Q = T[index - nxx*nzz];           /*  i , j ,k-1 */       

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
            ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
            ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
            ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
            ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Second octant: XY plane */

            Sref = S[index - 1 - nxx*nzz];
            hs2 = h*h*Sref*Sref;

            M = T[index - 1 + nzz - nxx*nzz];  /* i-1,j+1,k-1 */ 
            N = T[index - 1 - nxx*nzz];        /* i-1, j ,k-1 */     
            P = T[index - 1 + nzz];            /* i-1,j+1, k  */
            Q = T[index - 1];                  /* i-1, j , k  */

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Second octant: YZ plane */

            M = T[index - 1 + nzz - nxx*nzz]; /* i-1,j+1,k-1 */   
            N = T[index - 1 + nzz];           /* i-1,j+1, k  */   
            P = T[index + nzz - nxx*nzz];     /*  i ,j+1,k-1 */   
            Q = T[index + nzz];               /*  i ,j+1, k  */   

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Second octant: XZ plane */

            M = T[index - 1 + nzz - nxx*nzz]; /* i-1,j+1,k-1 */   
            N = T[index - 1 - nxx*nzz];       /* i-1, j ,k-1 */       
            P = T[index + nzz - nxx*nzz];     /*  i ,j+1,k-1 */   
            Q = T[index - nxx*nzz];           /*  i , j ,k-1 */       

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Third octant: XY plane */

            Sref = S[index - 1];
            hs2 = h*h*Sref*Sref;

            M = T[index - 1 + nzz + nxx*nzz]; /* i-1,j+1,k+1 */   
            N = T[index - 1 + nzz];           /* i-1,j+1, k  */   
            P = T[index - 1 + nxx*nzz];       /* i-1, j ,k+1 */ 
            Q = T[index - 1];                 /* i-1, j , k  */ 

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Third octant: YZ plane */

            M = T[index - 1 + nzz + nxx*nzz]; /* i-1,j+1,k+1 */   
            N = T[index + nzz + nxx*nzz];     /*  i ,j+1,k+1 */         
            P = T[index - 1 + nzz];           /* i-1,j+1, k  */ 
            Q = T[index + nzz];               /*  i ,j+1, k  */   

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Third octant: XZ plane */

            M = T[index - 1 + nzz + nxx*nzz]; /* i-1,j+1,k+1 */   
            N = T[index - 1 + nxx*nzz];       /* i-1, j ,k+1 */       
            P = T[index + nzz + nxx*nzz];     /*  i ,j+1,k+1 */   
            Q = T[index + nxx*nzz];           /*  i , j ,k+1 */       

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Fourth octant: XY plane */

            Sref = S[index - 1 - nzz];
            hs2 = h*h*Sref*Sref;

            M = T[index - 1 - nzz + nxx*nzz]; /* i-1,j-1,k+1 */  
            N = T[index - 1 + nxx*nzz];       /* i-1, j ,k+1 */      
            P = T[index - 1 - nzz];           /* i-1,j-1, k  */
            Q = T[index - 1];                 /* i-1, j , k  */

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Fourth octant: YZ plane */

            M = T[index - 1 - nzz + nxx*nzz]; /* i-1,j-1,k+1 */  
            N = T[index - 1 - nzz];           /* i-1,j-1, k  */  
            P = T[index - nzz + nxx*nzz];     /*  i ,j-1,k+1 */  
            Q = T[index - nzz];               /*  i ,j-1, k  */  

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Fourth octant: XZ plane */

            M = T[index - 1 - nzz + nxx*nzz]; /* i-1,j-1,k+1 */   
            N = T[index - nzz + nxx*nzz];     /*  i ,j-1,k+1 */         
            P = T[index - 1 + nxx*nzz];       /* i-1, j ,k+1 */ 
            Q = T[index + nxx*nzz];           /*  i , j ,k+1 */       

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Fifth octant: XY plane */

            Sref = S[index - nzz - nxx*nzz];
            hs2 = h*h*Sref*Sref;

            M = T[index + 1 - nzz - nxx*nzz]; /* i+1,j-1,k-1 */  
            N = T[index + 1 - nxx*nzz];       /* i+1, j ,k-1 */      
            P = T[index + 1 - nzz];           /* i+1,j-1, k  */
            Q = T[index + 1];                 /* i+1, j , k  */

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Fifth octant: YZ plane */

            M = T[index + 1 - nzz - nxx*nzz]; /* i+1,j-1,k-1 */   
            N = T[index + 1 - nzz];           /* i+1,j-1, k  */   
            P = T[index - nzz - nxx*nzz];     /*  i ,j-1,k-1 */   
            Q = T[index - nzz];               /*  i ,j-1, k  */   

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
            ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
            ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
            ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
            ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Fifth octant: XZ plane */

            M = T[index + 1 - nzz - nxx*nzz]; /* i+1,j-1,k-1 */   
            N = T[index - nzz - nxx*nzz];     /*  i ,j-1,k-1 */         
            P = T[index + 1 - nxx*nzz];       /* i+1, j ,k-1 */ 
            Q = T[index - nxx*nzz];           /*  i , j ,k-1 */       

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Sixth octant: XY plane */

            Sref = S[index - nxx*nzz];
            hs2 = h*h*Sref*Sref;

            M = T[index + 1 + nzz - nxx*nzz]; /* i+1,j+1,k-1 */   
            N = T[index + 1 + nzz];           /* i+1,j+1, k  */   
            P = T[index + 1 - nxx*nzz];       /* i+1, j ,k-1 */ 
            Q = T[index + 1];                 /* i+1, j , k  */ 

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Sixth octant: YZ plane */

            M = T[index + 1 + nzz - nxx*nzz]; /* i+1,j+1,k-1 */   
            N = T[index + nzz - nxx*nzz];     /*  i ,j+1,k-1 */        
            P = T[index + 1 + nzz];           /* i+1,j+1, k  */
            Q = T[index + nzz];               /*  i ,j+1, k  */  

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Sixth octant: XZ plane */

            M = T[index + 1 + nzz - nxx*nzz]; /* i+1,j+1,k-1 */   
            N = T[index + 1 - nxx*nzz];       /* i+1, j ,k-1 */       
            P = T[index + nzz - nxx*nzz];     /*  i ,j+1,k-1 */   
            Q = T[index - nxx*nzz];           /*  i , j ,k-1 */       

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Seventh octant: XY plane */
            
            Sref = S[index - nzz];
            hs2 = h*h*Sref*Sref;

            M = T[index + 1 - nzz + nxx*nzz]; /* i+1,j-1,k+1 */   
            N = T[index + 1 - nzz];           /* i+1,j-1, k  */   
            P = T[index + 1 + nxx*nzz];       /* i+1, j ,k+1 */ 
            Q = T[index + 1];                 /* i+1, j , k  */ 

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Seventh octant: YZ plane */

            M = T[index + 1 - nzz + nxx*nzz]; /* i+1,j-1,k+1 */  
            N = T[index - nzz + nxx*nzz];     /*  i ,j-1,k+1 */        
            P = T[index + 1 - nzz];           /* i+1,j-1, k  */
            Q = T[index - nzz];               /*  i ,j-1, k  */  

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Seventh octant: XZ plane */

            M = T[index + 1 - nzz + nxx*nzz]; /* i+1,j-1,k+1 */    
            N = T[index + 1 + nxx*nzz];       /* i+1, j ,k+1 */        
            P = T[index - nzz + nxx*nzz];     /*  i ,j-1,k+1 */    
            Q = T[index + nxx*nzz];           /*  i , j ,k+1 */        

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Eighth octant: XY plane */

            Sref = S[index];
            hs2 = h*h*Sref*Sref;

            M = T[index + 1 + nzz + nxx*nzz]; /* i+1,j+1,k+1 */  
            N = T[index + 1 + nxx*nzz];       /* i+1, j ,k+1 */      
            P = T[index + 1 + nzz];           /* i+1,j+1, k  */
            Q = T[index + 1];                 /* i+1, j , k  */

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Eighth octant: YZ plane */

            M = T[index + 1 + nzz + nxx*nzz]; /* i+1,j+1,k+1 */   
            N = T[index + 1 + nzz];           /* i+1,j+1, k  */   
            P = T[index + nzz + nxx*nzz];     /*  i ,j+1,k+1 */   
            Q = T[index + nzz];               /*  i ,j+1, k  */   

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* 3D operator - Eighth octant: XZ plane */

            M = T[index + 1 + nzz + nxx*nzz]; /* i+1,j+1,k+1 */   
            N = T[index + nzz + nxx*nzz];     /*  i ,j+1,k+1 */         
            P = T[index + 1 + nxx*nzz];       /* i+1, j ,k+1 */ 
            Q = T[index + nxx*nzz];           /*  i , j ,k+1 */       

            // MNP -> R 
            if ((M <= N) && (M <= P) && 
                ((2.0f*(P-M)*(P-M) + (N-M)*(N-M)) <= hs2) && 
                ((2.0f*(N-M)*(N-M) + (P-M)*(P-M)) <= hs2) && 
                ((N-M)*(N-M) + (P-M)*(P-M) + (N-M)*(P-M) >= 0.5f*hs2))
            {
                Tijk = N + P - M + sqrtf(hs2 - (N-M)*(N-M) - (P-M)*(P-M));
                if (Tijk < lowest) lowest = Tijk;
            }   

            // QNP -> R    
            if ((N <= Q) && (P <= Q) && 
                ((Q-N)*(Q-N) + (Q-P)*(Q-P) + (Q-N)*(Q-P) <= 0.5f*hs2))    
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (Q-P)*(Q-P));    
                if (Tijk < lowest) lowest = Tijk;
            }

            // NMQ -> R
            if ((N-M >= 0) && (N-M <= Q-N) && 
                (2*(Q-N)*(Q-N) + (N-M)*(N-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-N)*(Q-N) - (N-M)*(N-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            // PMQ -> R
            if ((P-M >= 0) && (P-M <= Q-P) && 
                (2*(Q-P)*(Q-P) + (P-M)*(P-M) <= hs2))
            {
                Tijk = Q + sqrtf(hs2 - (Q-P)*(Q-P) - (P-M)*(P-M));    
                if (Tijk < lowest) lowest = Tijk;
            }        

            /* Time atualization */
            if (fabsf(lowest - T[index]) <= tolerance) K[index] = 0.0f;

            nT[index] = lowest;
        }
    }
}

__global__ void expanding_box(float * K, float * nK, int nxx, int nyy, int nzz)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if ((K[index] == 1.0f) && (index < nxx*nyy*nzz))
    {
        int k = (int) (index / (nxx*nzz));         // y direction
        int j = (int) (index - k*nxx*nzz) / nzz;   // x direction
        int i = (int) (index - j*nzz - k*nxx*nzz); // z direction

        if ((i > 0) && (i < nzz-1) && (j > 0) && (j < nxx-1) && (k > 0) && (k < nyy-1))
        {
            nK[index - 1] = 1.0f;
            nK[index + 1] = 1.0f;
            nK[index - nzz] = 1.0f;
            nK[index + nzz] = 1.0f;
            nK[index - nxx*nzz] = 1.0f;
            nK[index + nxx*nzz] = 1.0f;
            nK[index + 1 - nzz] = 1.0f;
            nK[index - 1 - nzz] = 1.0f;
            nK[index + 1 + nzz] = 1.0f;
            nK[index - 1 + nzz] = 1.0f;
            nK[index + 1 + nxx*nzz] = 1.0f;
            nK[index + 1 - nxx*nzz] = 1.0f;
            nK[index - 1 + nxx*nzz] = 1.0f;
            nK[index - 1 - nxx*nzz] = 1.0f;
            nK[index - nzz - nxx*nzz] = 1.0f;
            nK[index - nzz + nxx*nzz] = 1.0f;
            nK[index + nzz - nxx*nzz] = 1.0f;
            nK[index + nzz + nxx*nzz] = 1.0f;
            nK[index + 1 + nzz + nxx*nzz] = 1.0f;
            nK[index + 1 + nzz - nxx*nzz] = 1.0f;
            nK[index + 1 - nzz + nxx*nzz] = 1.0f;
            nK[index + 1 - nzz - nxx*nzz] = 1.0f;
            nK[index - 1 - nzz - nxx*nzz] = 1.0f;
            nK[index - 1 - nzz + nxx*nzz] = 1.0f;
            nK[index - 1 + nzz - nxx*nzz] = 1.0f;
            nK[index - 1 + nzz + nxx*nzz] = 1.0f;
        }
    }
}

__global__ void update_volume(float * T, float * nT, float * K, float * nK, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
    {
        T[index] = nT[index];
        K[index] = nK[index];
    }
}